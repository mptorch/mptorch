#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cmath>
#include <chrono>
#include "common.h"

/* Helper function for tensor striding */
// ---------------------------------------------------------------------------------------
void dim_striding(const int *norm_dims, int n_norm, const int *dims, int n_dims, int &B, int &T, int &C){
	int real_dims[n_norm];
	for (int i = 0; i < n_norm; i++){
		real_dims[i] = (n_dims + (norm_dims[i] % n_dims)) % n_dims;
	}

	C = 1;
	for (int i : real_dims){
		C *= dims[i];
	}

	int min_dim = real_dims[n_norm - 1];
	int max_dim = real_dims[0];

	B = 1;
	for (int i = 0; i < min_dim; i++){
		B *= dims[i];
	}

	T = 1;
	for (int i = max_dim + 1; i < n_dims; i++){
		T *= dims[i];
	}
}


/* Quantization function and wrapper */
// ---------------------------------------------------------------------------------------

#define FLOAT_TO_BITS(x) (*reinterpret_cast<uint32_t *>(x))
#define BITS_TO_FLOAT(x) (*reinterpret_cast<float *>(x))

__host__ __device__ __forceinline__ uint32_t round_bitwise_nearest(uint32_t target, int man_bits)
{
    uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
    uint32_t machine_eps = 1 << (22 - man_bits);
    // tie breaking rule offset
    int offset = (down == machine_eps);
    uint32_t add_r = target + machine_eps;
    // apply the mask
    // this is the analogue of how you would do round
    // to nearest integer using the floor function:
    // round(x) = floor(x + 0.5)
    return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}

__host__ __device__ uint32_t clip_exponent_with_subnormals(int exp_bits, int man_bits, uint32_t old_num,
                                                        uint32_t quantized_num, bool saturate = false)
{
    if (quantized_num == 0)
        return quantized_num;

    int quantized_exponent_store = quantized_num << 1 >> 24;
    int min_exponent_store = -((1 << (exp_bits - 1)) - 2) - man_bits + 127;

    uint32_t old_sign = old_num >> 31 << 31;
    // underflow or round to smallest non zero subnormal value
    if (quantized_exponent_store < min_exponent_store)
    {
        int offset = (quantized_exponent_store == (min_exponent_store - 1));
        quantized_num += offset * (1u << 23);
        quantized_num = quantized_num | old_sign;
        quantized_num = offset * quantized_num;
    }
    return quantized_num;
}

__host__ __device__ uint32_t clip_exponent_without_subnormals(int exp_bits, int man_bits, uint32_t old_num,
                                                           uint32_t quantized_num, bool saturate = false)
{
    if (quantized_num == 0)
        return quantized_num;

    int quantized_exponent_store = quantized_num << 1 >> 24;
    int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127;
    int min_exponent_store = -((1 << (exp_bits - 1)) - 2) + 127;

    uint32_t old_sign = old_num >> 31 << 31;
    // saturate or overflow
    if (quantized_exponent_store > max_exponent_store)
    {
        if (saturate)
        {
            uint32_t max_man =
                (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits);
            uint32_t max_num = ((uint32_t)max_exponent_store << 23) | max_man;
            quantized_num = old_sign | max_num;
        }
        else
        {
            quantized_num = ((((uint32_t)1 << 31) - 1) ^ (((uint32_t)1 << 23) - 1));
            quantized_num = quantized_num | old_sign;
        }
    } // underflow or round to smallest nonzero normal value
    else if (quantized_exponent_store < min_exponent_store)
    {
        uint32_t offset = (quantized_exponent_store == (min_exponent_store - 1)) && ((old_num << 9 >> 9) > (1 << 22));
        quantized_num = offset * (min_exponent_store << 23);
        quantized_num |= old_sign;
    }
    return quantized_num;
}

__host__ __device__ float cast_fp_nearest(float origin_float, int man_bits, int exp_bits,
                                       bool subnormal_support = true,
                                       bool saturate = false)
{
    uint32_t target, quantize_bits;
    target = FLOAT_TO_BITS(&origin_float);
    float quantized;

    int target_exp = (target << 1 >> 1 >> 23) - 127;
    int min_exp = -((1 << (exp_bits - 1)) - 2);
    bool subnormal = (target_exp < min_exp);
    bool noquantize = (man_bits >= 23);

    if (noquantize)
    {
        quantized = origin_float;
    }
    else
    {
        // handle subnormal inputs (if subnormal mode is active)
        if (subnormal && subnormal_support)
        {
            int exp_diff = man_bits - (min_exp - target_exp);
            int not_uflow = exp_diff > -1 || ((exp_diff == -1) && ((target << 9) > 0));
            quantize_bits = not_uflow * round_bitwise_nearest(target, exp_diff);
            quantize_bits =
                clip_exponent_with_subnormals(exp_bits, man_bits, target, quantize_bits, saturate);
            quantized = BITS_TO_FLOAT(&quantize_bits);
        }
        // handle NaN/inf inputs
        else if (target_exp == 128)
        {
            quantized = origin_float;
        }
        // normal value range or overflow
        else
        {
            quantize_bits = round_bitwise_nearest(target, man_bits);
            quantize_bits =
                clip_exponent_without_subnormals(exp_bits, man_bits, target, quantize_bits, saturate);
            quantized = BITS_TO_FLOAT(&quantize_bits);
        }
    }

    return quantized;
}

__host__ __device__ float quant_acc(float origin_float) {
    return cast_fp_nearest(origin_float, 10, 8, true, false);
}

__host__ __device__ float quant_mul(float origin_float) {
    return cast_fp_nearest(origin_float, 10, 8, true, false);
}

__host__ __device__ float quant_div(float origin_float) {
    return cast_fp_nearest(origin_float, 10, 8, true, false);
}

// ---------------------------------------------------------------------------------------
// CPU version
static void layernorm_backward_cpu(const float* in_arr, const float* out_grad, 
                                const float* w_array, const float* b_array,
                                const float* m_array, const float* r_array,
                                float* grad_gamma, float* grad_beta, float* out_arr,
                                int B, int T, int C){
    for (int i = 0; i < B * T; i++){
        int b = i / T;
        int t = i % T;

        int base_index = (b * C * T) + t;
        const float* input = in_arr + base_index;
        const float* gradient = out_grad + base_index;
        float* output = out_arr + base_index;

        float m = m_array[b * T + t];
        float r = r_array[b * T + t];

        // two reduce operations
        float grad_sum = 0.0f;
        float grad_sum_xhat = 0.0f;
        for (int k = 0; k < C; k++){
            int idx = k * T;
            float in_m = quant_acc(input[idx] - m);
            float xhat = quant_mul(in_m * r);
            float grad_xhat = quant_mul(w_array[k] * gradient[idx]);
            float dot_xhat = quant_mul(xhat * grad_xhat);
            grad_sum = quant_acc(grad_sum + grad_xhat);
            grad_sum_xhat = quant_acc(grad_sum_xhat + dot_xhat);
            }
            grad_sum = quant_div(grad_sum/C);
            grad_sum_xhat = quant_div(grad_sum_xhat/C);

        // iterate and accumulate 
        for (int k = 0; k < C; k++){
            int idx = k * T;
            float in_m = quant_acc(input[idx] - m);
            float xhat = quant_mul(in_m * r);
            float xhat_gradient = quant_mul(xhat * gradient[idx]);
            float grad_xhat = quant_mul(w_array[k] * gradient[idx]);
            
            grad_gamma[k] = quant_acc(grad_gamma[k] + xhat_gradient);
            grad_beta[k] = quant_acc(grad_beta[k] + gradient[idx]);
            
            float weighted_grad_sum = quant_mul(xhat * grad_sum_xhat);
            float grad_input = grad_xhat;
            grad_input = quant_acc(grad_input - grad_sum);
            grad_input = quant_acc(grad_input - weighted_grad_sum);
            grad_input = quant_mul(grad_input * r);

            output[idx] = grad_input;
        }
    }
}

// ---------------------------------------------------------------------------------------
// GPU kernels

__global__ void layernorm_backward_first_pass_kernel1(const float* __restrict__ in_arr, const float* __restrict__ out_grad, 
                                const float* w_array, const float* b_array,
                                const float* m_array, const float* r_array,  
                                float* out_arr, float* xhat_gradient,
                                int B, int T, int C){
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= B*T) return;

    int b = i / T;
    int t = i % T;

    int base_index = (b * C * T) + t;
    const float* input = in_arr + base_index;
    const float* gradient = out_grad + base_index;
    float* output = out_arr + base_index;
    float* xhat_grad = xhat_gradient + base_index;

    float m = m_array[b * T + t];
    float r = r_array[b * T + t];

    // two reduce operations
    float grad_sum = 0.0f;
    float grad_sum_xhat = 0.0f;
    for (int k = 0; k < C; k++){
        int idx = k * T;
        float in_m = quant_acc(input[idx] - m);
        float xhat = quant_mul(in_m * r);
        float norm_grad = quant_mul(w_array[k] * gradient[idx]);
        float dot_xhat = quant_mul(xhat * norm_grad);
        grad_sum = quant_acc(grad_sum + norm_grad);
        grad_sum_xhat = quant_acc(grad_sum_xhat + dot_xhat);
    }
    grad_sum = quant_div(grad_sum/C);
    grad_sum_xhat = quant_div(grad_sum_xhat/C);

    // iterate and accumulate 
    for (int k = 0; k < C; k++){
        int idx = k * T;
        float in_m = quant_acc(input[idx] - m);
        float xhat = quant_mul(in_m * r);
        xhat_grad[idx] = quant_mul(xhat * gradient[idx]);
        
        float norm_grad = quant_mul(w_array[k] * gradient[idx]);
        float weighted_grad_sum = quant_mul(xhat * grad_sum_xhat);
        float grad_input = norm_grad;
        grad_input = quant_acc(grad_input - grad_sum);
        grad_input = quant_acc(grad_input - weighted_grad_sum);
        grad_input = quant_mul(grad_input * r);

        output[idx] = grad_input;
    }
}

__global__ void layernorm_backward_second_pass_kernel1(const float* __restrict__ xhat_gradient, const float* __restrict__ out_grad,
                                                    float* grad_gamma, float* grad_beta,
                                                    int B, int T, int C){
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k >= C) return;

    int idx = k * T;

    float grad_gamma_sum = 0.0f;
    float grad_beta_sum = 0.0f;

    for (int i = 0; i < B * T; i++){
        int b = i / T;
        int t = i % T;
        int base_index = (b * C * T) + t;
        const float* gradient = out_grad + base_index;
        const float* xhat_grad = xhat_gradient + base_index;

        grad_gamma_sum = quant_acc(grad_gamma_sum + xhat_grad[idx]);
        grad_beta_sum = quant_acc(grad_beta_sum + gradient[idx]);
    }

    grad_gamma[k] = grad_gamma_sum;
    grad_beta[k] = grad_beta_sum;
}

__global__ void layernorm_backward_first_pass_kernel2(const float* __restrict__ in_arr, const float* __restrict__ out_grad, 
                                const float* w_array, const float* b_array,
                                const float* m_array, const float* r_array,  
                                float* out_arr, float* xhat_gradient,
                                int B, int T, int C)
{
    extern __shared__ float shared[];

    int tid = threadIdx.x;
    int warp = threadIdx.x / warpSize; // groups of 32 threads (which warp the thread belongs to)
    int lane = threadIdx.x % warpSize; // a warp has 32 lanes (id of the thread in a warp)

    int warpsPerBlock = blockDim.x / warpSize;

    int b = blockIdx.x / T;
    int t = blockIdx.x % T;

    int base_index = (b * C * T) + t;
    const float* input = in_arr + base_index;
    const float* gradient = out_grad + base_index;
    float* output = out_arr + base_index;
    float* xhat_grad = xhat_gradient + base_index;

    float m = m_array[b * T + t];
    float r = r_array[b * T + t];

    // two reduce operations
    // grad_sum
    float grad_sum = 0.0f;
    float grad_sum_xhat = 0.0f;
    for (int k = tid; k < C; k += blockDim.x){
        int idx = k * T;
        float in_m = quant_acc(input[idx] - m);
        float xhat = quant_mul(in_m * r);
        float norm_grad = quant_mul(w_array[k] * gradient[idx]);
        float dot_xhat = quant_mul(xhat * norm_grad);
        grad_sum = quant_acc(grad_sum + norm_grad);
        grad_sum_xhat = quant_acc(grad_sum_xhat + dot_xhat); 
    }
    for (int offset = warpSize/2; offset > 0; offset /= 2){
        grad_sum = quant_acc(grad_sum + __shfl_down_sync(0xffffffff, grad_sum, offset));
    }
    if (lane == 0){
        shared[warp] = grad_sum;
    }
    __syncthreads();
    if (tid == 0){
        grad_sum = shared[0];
        for (int i = 1; i < warpsPerBlock; i++){
            grad_sum = quant_acc(grad_sum + shared[i]);
        }
        shared[0] = quant_div(grad_sum/C);
    }
    __syncthreads();
    grad_sum = shared[0];

    // grad_sum_xhat
    for (int offset = warpSize/2; offset > 0; offset /= 2){
        grad_sum_xhat = quant_acc(grad_sum_xhat + __shfl_down_sync(0xffffffff, grad_sum_xhat, offset));
    }
    if (lane == 0){
        shared[warp] = grad_sum_xhat;
    }
    __syncthreads();
    if (tid == 0){
        grad_sum_xhat = shared[0];
        for (int i = 1; i < warpsPerBlock; i++){
            grad_sum_xhat = quant_acc(grad_sum_xhat + shared[i]);
        }
        shared[0] = quant_div(grad_sum_xhat/C);
    }
    __syncthreads();
    grad_sum_xhat = shared[0];

    for (int k = tid; k < C; k += blockDim.x){
        int idx = k * T;
        float in_m = quant_acc(input[idx] - m);
        float xhat = quant_mul(in_m * r);
        float norm_grad = quant_mul(w_array[k] * gradient[idx]);
        xhat_grad[idx] = quant_mul(xhat * gradient[idx]);

        float weighted_grad_sum = quant_mul(xhat * grad_sum_xhat);
        float grad_input = norm_grad;
        grad_input = quant_acc(grad_input - grad_sum);
        grad_input = quant_acc(grad_input - weighted_grad_sum);
        grad_input = quant_mul(grad_input * r);

        output[idx] = grad_input;
    }
}

__global__ void layernorm_backward_second_pass_kernel2(const float* __restrict__ xhat_gradient, const float* __restrict__ out_grad,
                                                    float* grad_gamma, float* grad_beta,
                                                    int B, int T, int C){
    int k = blockIdx.x * blockDim.x + threadIdx.x;

    if (k >= C) return;

    int idx = k * T;

    float grad_gamma_sum = 0.0f;
    float grad_beta_sum = 0.0f;

    for (int i = 0; i < B * T; i++){
        int b = i / T;
        int t = i % T;
        int base_index = (b * C * T) + t;
        const float* gradient = out_grad + base_index;
        const float* xhat_grad = xhat_gradient + base_index;

        grad_gamma_sum = quant_acc(grad_gamma_sum + xhat_grad[idx]);
        grad_beta_sum = quant_acc(grad_beta_sum + gradient[idx]);
    }

    grad_gamma[k] = grad_gamma_sum;
    grad_beta[k] = grad_beta_sum;
}



// reference kernel using atomicAdd w/o quant
__global__ void layernorm_backward_kernel0(const float* __restrict__ in_arr, const float* out_grad, 
                                const float* w_array, const float* b_array,
                                const float* m_array, const float* r_array,
                                float* grad_gamma, float* grad_beta, float* out_arr,
                                int B, int T, int C){
    extern __shared__ float shared[];

    int tid = threadIdx.x;
    int warp = threadIdx.x / warpSize; // groups of 32 threads (which warp the thread belongs to)
    int lane = threadIdx.x % warpSize; // a warp has 32 lanes (id of the thread in a warp)

    int warpsPerBlock = blockDim.x / warpSize;

    int b = blockIdx.x / T;
    int t = blockIdx.x % T;

    int base_index = (b * C * T) + t;
    const float* input = in_arr + base_index;
    const float* gradient = out_grad + base_index;
    float* output = out_arr + base_index;

    float m = m_array[b * T + t];
    float r = r_array[b * T + t];

    // two reduce operations
    // grad_sum
    float grad_sum = 0.0f;
    float grad_sum_xhat = 0.0f;
    for (int k = tid; k < C; k += blockDim.x){
        int idx = k * T;
        float in_m = quant_acc(input[idx] - m);
        float xhat = quant_mul(in_m * r);
        float norm_grad = quant_mul(w_array[k] * gradient[idx]);
        float dot_xhat = quant_mul(xhat * norm_grad);
        grad_sum = quant_acc(grad_sum + norm_grad);
        grad_sum_xhat = quant_acc(grad_sum_xhat + dot_xhat); 
    }
    for (int offset = warpSize/2; offset > 0; offset /= 2){
        grad_sum = quant_acc(grad_sum + __shfl_down_sync(0xffffffff, grad_sum, offset));
    }
    if (lane == 0){
        shared[warp] = grad_sum;
    }
    __syncthreads();
    if (tid == 0){
        grad_sum = shared[0];
        for (int i = 1; i < warpsPerBlock; i++){
            grad_sum = quant_acc(grad_sum + shared[i]);
        }
        shared[0] = quant_div(grad_sum/C);
    }
    __syncthreads();
    grad_sum = shared[0];

    // grad_sum_xhat
    for (int offset = warpSize/2; offset > 0; offset /= 2){
        grad_sum_xhat = quant_acc(grad_sum_xhat + __shfl_down_sync(0xffffffff, grad_sum_xhat, offset));
    }
    if (lane == 0){
        shared[warp] = grad_sum_xhat;
    }
    __syncthreads();
    if (tid == 0){
        grad_sum_xhat = shared[0];
        for (int i = 1; i < warpsPerBlock; i++){
            grad_sum_xhat = quant_acc(grad_sum_xhat + shared[i]);
        }
        shared[0] = quant_div(grad_sum_xhat/C);
    }
    __syncthreads();
    grad_sum_xhat = shared[0];

    // iterate and accumulate 
    for (int k = tid; k < C; k += blockDim.x){
        int idx = k * T;
        float in_m = quant_acc(input[idx] - m);
        float xhat = quant_mul(in_m * r);
        float xhat_gradient = quant_mul(xhat * gradient[idx]);
        float norm_grad = quant_mul(w_array[k] * gradient[idx]);

        atomicAdd(&grad_gamma[k], xhat_gradient);
        atomicAdd(&grad_beta[k], gradient[idx]);

        float weighted_grad_sum = quant_mul(xhat * grad_sum_xhat);
        float grad_input = norm_grad;
        grad_input = quant_acc(grad_input - grad_sum);
        grad_input = quant_acc(grad_input - weighted_grad_sum);
        grad_input = quant_mul(grad_input * r);

        output[idx] = grad_input;
    }
}

// ---------------------------------------------------------------------------------------
// Kernel launchers
void layernorm_backward_cuda1(const float* in_arr, const float* out_grad, 
                            const float* w_array, const float* b_array,
                            const float* m_array, const float* r_array,
                            float* grad_gamma, float* grad_beta, float* out_arr,
                            int B, int T, int C, int block_size)
{
    int N = B * T;
    int blocks = N / block_size + (N % block_size != 0);
    float* xhat_gradient;
    cudaCheck(hipMalloc(&xhat_gradient, sizeof(float) * (B * T * C)));
    layernorm_backward_first_pass_kernel1<<<blocks, block_size>>>(in_arr, out_grad, w_array, b_array, m_array, r_array, out_arr, xhat_gradient, B, T, C);
    blocks = C / block_size + (C % block_size != 0);
    layernorm_backward_second_pass_kernel1<<<blocks, block_size>>>(xhat_gradient, out_grad, grad_gamma, grad_beta, B, T, C);
    cudaCheck(hipFree(xhat_gradient));
}

void layernorm_backward_cuda2(const float* in_arr, const float* out_grad, 
                            const float* w_array, const float* b_array,
                            const float* m_array, const float* r_array,
                            float* grad_gamma, float* grad_beta, float* out_arr,
                            int B, int T, int C, int block_size)
{
    int blocks = B * T;
    size_t shared_mem_size = (block_size / 32) * sizeof(float);
    float* xhat_gradient;
    cudaCheck(hipMalloc(&xhat_gradient, sizeof(float) * (B * T * C)));
    layernorm_backward_first_pass_kernel2<<<blocks, block_size, shared_mem_size>>>(in_arr, out_grad, w_array, b_array, m_array, r_array, out_arr, xhat_gradient, B, T, C);
    blocks = C / block_size + (C % block_size != 0);
    layernorm_backward_second_pass_kernel2<<<blocks, block_size>>>(xhat_gradient, out_grad, grad_gamma, grad_beta, B, T, C);
    cudaCheck(hipFree(xhat_gradient));
}

// reference using atomic add
void layernorm_backward_cuda0(const float* in_arr, const float* out_grad, 
                            const float* w_array, const float* b_array,
                            const float* m_array, const float* r_array,
                            float* grad_gamma, float* grad_beta, float* out_arr,
                            int B, int T, int C, int block_size)
{
    int blocks = B * T;
    size_t shared_mem_size = (block_size / 32) * sizeof(float);
    layernorm_backward_kernel0<<<blocks, block_size, shared_mem_size>>>(in_arr, out_grad, w_array, b_array, m_array, r_array, grad_gamma, grad_beta, out_arr, B, T, C);
}

void layernorm_backward_cuda(int kernel_num, const float* in_arr, const float* out_grad, 
                            const float* w_array, const float* b_array,
                            const float* m_array, const float* r_array,
                            float* grad_gamma, float* grad_beta, float* out_arr,
                            int B, int T, int C, int block_size){
    switch (kernel_num){
        case 0:
            layernorm_backward_cuda0(in_arr, out_grad, 
                            w_array, b_array,
                            m_array, r_array,
                            grad_gamma, grad_beta, out_arr,
                            B, T, C, block_size);
            break;
        case 1:
            layernorm_backward_cuda1(in_arr, out_grad, 
                            w_array, b_array,
                            m_array, r_array,
                            grad_gamma, grad_beta, out_arr,
                            B, T, C, block_size);
            break;
        case 2:
            layernorm_backward_cuda2(in_arr, out_grad, 
                            w_array, b_array,
                            m_array, r_array,
                            grad_gamma, grad_beta, out_arr,
                            B, T, C, block_size);
            break;
        default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ---------------------------------------------------------------------------------------
int main(int argc, const char **argv) {
    setup_main();

    const int norm_dims[] = {-1, -2};
    const int n_norm = sizeof(norm_dims)/sizeof(norm_dims[0]);
    const int dims[] = {40, 60, 300};
    const int n_dims = sizeof(dims)/sizeof(dims[0]);

    int B, T, C;
    dim_striding(norm_dims, n_norm, dims, n_dims, B, T, C);

    // which kernel to use
    int version = 1;
    if (argc > 1){
        version = atoi(argv[1]);
    }

    // host tensors
    int numel = 1;
    for (int i = 0; i < n_dims; i++){
    	numel *= dims[i];
    }
    float* h_input = make_random_float(numel);
    float* h_grad = make_random_float(numel);
    float* h_output = make_zeros_float(numel);
    float* h_weight = make_ones_float(C);
    float* h_bias = make_zeros_float(C);
    float* h_gg = make_zeros_float(C);
    float* h_gb = make_zeros_float(C);
    float* h_mean = make_random_float(B * T);
    float* h_rstd = make_random_float(B * T);

    // compute cpu reference
    layernorm_backward_cpu(h_input, h_grad, 
                        h_weight, h_bias,
                        h_mean, h_rstd,
                        h_gg, h_gb, h_output,
                        B, T, C);

    // device tensors (move data to gpu)
    float *d_input, *d_grad, *d_output, *d_weight, *d_bias, *d_gg, *d_gb, *d_mean, *d_rstd;
    cudaCheck(hipMalloc(&d_input, numel * sizeof(float)));
    cudaCheck(hipMalloc(&d_grad, numel * sizeof(float)));
    cudaCheck(hipMalloc(&d_weight, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_bias, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_gg, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_gb, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_output, numel * sizeof(float)));
    cudaCheck(hipMalloc(&d_mean, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_rstd, B * T * sizeof(float)));
    cudaCheck(hipMemcpy(d_input, h_input, numel * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_grad, h_grad, numel * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weight, h_weight, C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_bias, h_bias, C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_mean, h_mean, B * T * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_rstd, h_rstd, B * T * sizeof(float), hipMemcpyHostToDevice));


    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j) {
        hipMemset(d_gg, 0, C * sizeof(float));
        hipMemset(d_gb, 0, C * sizeof(float));

        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        layernorm_backward_cuda(version, d_input, d_grad, 
                            d_weight, d_bias,
                            d_mean, d_rstd,
                            d_gg, d_gb, d_output,
                            B, T, C, block_size);

        float tol = 2e-0f;
        printf("validating input_grad\n");
        validate_result(d_output, h_output, "input_grad", numel, tol);
        printf("validating weight_grad\n");
        validate_result(d_gg, h_gg, "weight_grad", C, tol);
        printf("validating bias_grad\n");
        validate_result(d_gb, h_gb, "bias_grad", C, tol);
    }

    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j) {
        int block_size = block_sizes[j];
        int repeat_times = 100;
        float elapsed_time = benchmark_kernel(repeat_times, layernorm_backward_cuda, 
                                            version, d_input, d_grad, 
                                            d_weight, d_bias,
                                            d_mean, d_rstd,
                                            d_gg, d_gb, d_output,
                                            B, T, C, block_size);
        printf("block_size %4d | time %.4f ms\n", block_size, elapsed_time);
    }

    printf("\nBenchmarking CPU version.");
    int repeat_times = 10;
    namespace chr = std::chrono;
    chr::steady_clock::time_point begin = chr::steady_clock::now();
    for(int i = 0; i < repeat_times; i++) {
        layernorm_backward_cpu(h_input, h_grad, 
                        h_weight, h_bias,
                        h_mean, h_rstd,
                        h_gg, h_gb, h_output,
                        B, T, C);
    }
    chr::steady_clock::time_point end = chr::steady_clock::now();
    auto elapsed_time_us = chr::duration_cast<chr::microseconds>(end - begin).count();
    float average_time_ms = ((float)elapsed_time_us / (float)repeat_times) / 1000.f;
    printf(" %.4f ms\n ", average_time_ms);


    free(h_input);
    free(h_grad);
    free(h_output);
    free(h_weight);
    free(h_bias);
    free(h_gg);
    free(h_gb);
    free(h_mean);
    free(h_rstd);

    cudaCheck(hipFree(d_input));
    cudaCheck(hipFree(d_grad));
    cudaCheck(hipFree(d_output));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_bias));
    cudaCheck(hipFree(d_gg));
    cudaCheck(hipFree(d_gb));
    cudaCheck(hipFree(d_mean));
    cudaCheck(hipFree(d_rstd));

    return 0;
}