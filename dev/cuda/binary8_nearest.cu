#include "hip/hip_runtime.h"
/*
Kernels for IEEE-754 down casting from binary32 to a lower precision format.
Payload is still a binary32 value.

Compile example:
nvcc -O3 binary8_nearest.cu -o binary8_nearest -std=c++17 -lcublas

version 1 attempted to make the code as compact as possible, while also 
maintaining readability; bit shifts and masking are used aplenty
./binary8_nearest 1

*/

#include <hip/hip_runtime.h>
#include "common.h"

enum class SaturateMode {
    SATURATE,
    NO_OVERFLOW,
    OVERFLOWS
};


// --------------------------------------------------------------------------------------
// I/O pairs to sanity check the CPU reference code (E5M2 without subnormals, RNE)
uint32_t test_inputs[] = {
    0b00111000000000000000000000000000, // min normal
    0b00110111000000000000000000000000, // subnormals 1
    0b00110111100000000000000000000000, // subnormals 2
    0b00110111110000000000000000000000, // subnormals 3
    0b00110110100000000000000000000000,
    0b00110110100000000000000000000001,
    0b01000111010000000000000000000000,
    0b01000111011000000000000000000000,
    0b00110110000000000000000000000000,
    0b10111111110000000000000000000000}; // -1.5

uint32_t test_outputs[] = {
    0b00111000000000000000000000000000, // min normal
    0b00110111000000000000000000000000, // subnormals 1
    0b00110111100000000000000000000000, // subnormals 2
    0b00110111110000000000000000000000, // subnormals 3
    0b00000000000000000000000000000000,
    0b00110111000000000000000000000000,
    0b01000111010000000000000000000000,
    0b01000111010000000000000000000000,
    0b00000000000000000000000000000000,
    0b10111111110000000000000000000000}; // -1.5

uint32_t test_outputs_no_sub[] = {
    0b00111000000000000000000000000000, // min normal
    0b00000000000000000000000000000000, // min normal 
    0b00000000000000000000000000000000, // min normal
    0b00111000000000000000000000000000, // min normal
    0b00000000000000000000000000000000,
    0b00000000000000000000000000000000  ,
    0b01000111010000000000000000000000,
    0b01000111010000000000000000000000,
    0b00000000000000000000000000000000,
    0b10111111110000000000000000000000}; // -1.5

uint32_t test_inputs_unsigned[] = {
    0b00110000000000000000000000000000, // min normal
    0b00101111000000000000000000000000, // subnormals 1
    0b00101110100000000000000000000000, // round to 0
    0b00101110100000000000000000000001, // round to min
    0b00110110100000000000000000000000,
    0b00110110100000000000000000000001,
    0b01000111010000000000000000000000,
    0b01000111011000000000000000000000,
    0b00110110000000000000000000000000,
    0b10111111110000000000000000000000}; // -1.5

uint32_t test_outputs_unsigned[] = {
    0b00110000000000000000000000000000, // min normal
    0b00101111000000000000000000000000, // min subnormals
    0b00000000000000000000000000000000, // round to 0
    0b00101111000000000000000000000000, 
    0b00110110100000000000000000000000,
    0b00110110100000000000000000000000,
    0b01000111010000000000000000000000,
    0b01000111011000000000000000000000,
    0b00110110000000000000000000000000,
    0x7FC00000}; // NAN

uint32_t test_outputs_unsigned_w[] = {
    0b00110000000000000000000000000000, // min normal
    0b00000000000000000000000000000000, // subnormals round to 0
    0b00000000000000000000000000000000, // round to 0
    0b00000000000000000000000000000000, // round to 0
    0b00110110100000000000000000000000,
    0b00110110100000000000000000000000,
    0b01000111010000000000000000000000,
    0b01000111011000000000000000000000,
    0b00110110000000000000000000000000,
    0x7FC00000}; // NAN


uint32_t round_bitwise_nearest_cpu(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}

uint32_t round_bitwise_nearest_p1_cpu(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  // tie breaking rule offset
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}

uint32_t binary8_clip_exponent_cpu(int exp_bits, int man_bits, uint32_t old_num, uint32_t quantized_num, SaturateMode saturation_mode, bool subnormal) {

  if (quantized_num == 0){
    return quantized_num;
  }
  
  uint32_t man_val = quantized_num & 0x7FFFFF;
  uint32_t old_sign = old_num >> 31 << 31;
  uint32_t max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  
  int spec_exp = (man_bits == 0) ? 1 : 0; // if P = 1
  int special_p1 = 0;
  
  if(exp_bits == 8 && saturation_mode != SaturateMode::NO_OVERFLOW){ // unsigned and p=1
      special_p1 = 1; // 0 bit of mantissa so the max value 0xfd = max_exp - 1 | mantissa = 0
  }else if (exp_bits == 7 &&  man_bits == 1 && saturation_mode != SaturateMode::NO_OVERFLOW){ // unsigned and p=2 
      special_p1 = 1;
  }else if(exp_bits + man_bits == 8){ // unsigned
      max_man = ((1u << man_bits) - 3u) << (23 - man_bits); // 2+ bit of mantissa so the max value 0xfd = mACax_exp | max_mantissa - 1 
  }

  // Special because in unsigned we want our min to be 1 less because the space is taken by the Nan
  int quantized_exponent_store = (quantized_num << 1 >> 24);
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127 - special_p1; 
  int min_exponent_store = -((1 << (exp_bits - 1)) - 1) + 127 + spec_exp;

  if (saturation_mode == SaturateMode::NO_OVERFLOW) { // Saturate to max without infinity
    max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  }

  if (quantized_exponent_store > max_exponent_store || ((quantized_exponent_store == max_exponent_store) && (man_val > max_man))) 
  {
    if (saturation_mode == SaturateMode::OVERFLOWS){ // Overflow to infinity
        return quantized_num = old_sign | 0x7F800000; // INF
    } 
    return quantized_num = old_sign | ((uint32_t)max_exponent_store << 23) | max_man;
  }
  if (quantized_exponent_store < min_exponent_store) {
    if (subnormal) {
        int subnormal_shift = min_exponent_store - quantized_exponent_store;
        int min_subnormals_exp = min_exponent_store - man_bits;
        uint32_t min_num = ((uint32_t)min_subnormals_exp << 23);
        uint32_t middle_num = ((uint32_t)(min_subnormals_exp - 1) << 23);
      if (subnormal_shift <= man_bits) {
        // quantized_num stays the same in this case
      } else if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    } 
    if(!subnormal) {
      uint32_t min_num = (uint32_t)min_exponent_store << 23;
      uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23);
      if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    }
  }
  return quantized_num;
}

float cast_binary8_signed_nearest_cpu(float origin_float, int P, SaturateMode saturation_mode, bool subnormals) {

    const int exp_bits = 8 - P;
    const int man_bits = P - 1;
    const uint32_t uval32 = FLOAT_TO_BITS(&origin_float);
    const int exp_val = (uval32 << 1 >> 24) - 127;
    const uint32_t man_val = uval32 & 0x7FFFFF;

    // Early return for inf/NaN case
    if (exp_val == 128 && !(saturation_mode == SaturateMode::NO_OVERFLOW && man_val == 0)) {
        return origin_float;
    }

    int subnormal_shift = 0;
    if (subnormals) {
        const int spec_exp = (P == 1) ? 1 : 0;
        const int max_exp = (1 << (exp_bits - 1)) - 1;
        const int min_exp = spec_exp - max_exp;

        if (((min_exp - exp_val) <= man_bits) && (exp_val < min_exp)) {
            subnormal_shift = min_exp - exp_val;
        }
    }

    uint32_t uval8 = (P == 1) ? round_bitwise_nearest_p1_cpu(uval32, man_bits - subnormal_shift)
                              : round_bitwise_nearest_cpu(uval32, man_bits - subnormal_shift);

    uval8 = binary8_clip_exponent_cpu(exp_bits, man_bits, uval32, uval8, saturation_mode, subnormals);
    return BITS_TO_FLOAT(&uval8);
}

float cast_binary8_unsigned_nearest_cpu(float origin_float, int P, SaturateMode saturation_mode, bool subnormals) {

    if (origin_float < 0) return NAN;   

    uint32_t uval32 = FLOAT_TO_BITS(&origin_float);
    const int exp_val = (uval32 << 1 >> 24) - 127;
    uint32_t man_val = uval32 & 0x7FFFFF;

    if (exp_val == 128 && !(saturation_mode == SaturateMode::NO_OVERFLOW && man_val == 0)) {
        return origin_float;
    }

    const int exp_bits = 9 - P;
    const int man_bits = P - 1;
    int subnormal_shift = 0;

    if (subnormals) {
        const int max_exp = (1 << (exp_bits - 1)) - 1;
        const int min_exp = (P == 1) - max_exp;
        
        if ((min_exp - exp_val) <= man_bits && exp_val < min_exp){ 
            subnormal_shift = min_exp - exp_val;
        }
    }

    uint32_t uval8 = (P == 1) ? round_bitwise_nearest_p1_cpu(uval32, man_bits - subnormal_shift)
                               : round_bitwise_nearest_cpu(uval32, man_bits - subnormal_shift);
    
    uval8 = binary8_clip_exponent_cpu(exp_bits, man_bits, uval32, uval8, saturation_mode, subnormals);
    
    return BITS_TO_FLOAT(&uval8);
}

void binary8_signed_nearest_cpu(float *o, float *a, int N, int P, bool is_signed, SaturateMode saturation_mode, bool subnormals) {
  for (int i = 0; i < N; ++i){
    if(is_signed){
      o[i] = cast_binary8_signed_nearest_cpu(a[i], P, saturation_mode, subnormals);
    }
    else{
      o[i] = cast_binary8_unsigned_nearest_cpu(a[i], P, saturation_mode, subnormals);
    }
  }
}

// ---------------------------------------------------------------------------------------
// GPU kernels
__device__ __forceinline__ uint32_t
round_bitwise_nearest(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}

__device__ __forceinline__ uint32_t
round_bitwise_nearest_p1(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  // tie breaking rule offset
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}


__device__ __forceinline__ uint32_t
binary8_clip_exponent(int exp_bits, int man_bits, uint32_t old_num, uint32_t quantized_num, SaturateMode saturation_mode, bool subnormal) {

  if (quantized_num == 0){
    return quantized_num;
  }
  
  uint32_t man_val = quantized_num & 0x7FFFFF;
  uint32_t old_sign = old_num >> 31 << 31;
  uint32_t max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  
  int spec_exp = (man_bits == 0) ? 1 : 0; // if P = 1
  int special_p1 = 0;
  
  if(exp_bits == 8 && saturation_mode != SaturateMode::NO_OVERFLOW){ // unsigned and p=1
      special_p1 = 1; // 0 bit of mantissa so the max value 0xfd = max_exp - 1 | mantissa = 0
  }else if (exp_bits == 7 &&  man_bits == 1 && saturation_mode != SaturateMode::NO_OVERFLOW){ // unsigned and p=2 
      special_p1 = 1;
  }else if(exp_bits + man_bits == 8){ // unsigned
      max_man = ((1u << man_bits) - 3u) << (23 - man_bits); // 2+ bit of mantissa so the max value 0xfd = mACax_exp | max_mantissa - 1 
  }

  // Special because in unsigned we want our min to be 1 less because the space is taken by the Nan
  int quantized_exponent_store = (quantized_num << 1 >> 24);
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127 - special_p1; 
  int min_exponent_store = -((1 << (exp_bits - 1)) - 1) + 127 + spec_exp;

  if (saturation_mode == SaturateMode::NO_OVERFLOW) { // Saturate to max without infinity
    max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  }

  if (quantized_exponent_store > max_exponent_store || ((quantized_exponent_store == max_exponent_store) && (man_val > max_man))) 
  {
    if (saturation_mode == SaturateMode::OVERFLOWS){ // Overflow to infinity
        return quantized_num = old_sign | 0x7F800000; // INF
    } 
    return quantized_num = old_sign | ((uint32_t)max_exponent_store << 23) | max_man;
  }
  if (quantized_exponent_store < min_exponent_store) {
    if (subnormal) {
        int subnormal_shift = min_exponent_store - quantized_exponent_store;
        int min_subnormals_exp = min_exponent_store - man_bits;
        uint32_t min_num = ((uint32_t)min_subnormals_exp << 23);
        uint32_t middle_num = ((uint32_t)(min_subnormals_exp - 1) << 23);
      if (subnormal_shift <= man_bits) {
        // quantized_num stays the same in this case
      } else if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    } 
    if(!subnormal) {
      uint32_t min_num = (uint32_t)min_exponent_store << 23;
      uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23);
      if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    }
  }
  return quantized_num;
}

__device__ float cast_binary8_signed_nearest(float origin_float, int P, SaturateMode saturation_mode, bool subnormals) {

    const int exp_bits = 8 - P;
    const int man_bits = P - 1;
    const uint32_t uval32 = FLOAT_TO_BITS(&origin_float);
    const int exp_val = (uval32 << 1 >> 24) - 127;
    const uint32_t man_val = uval32 & 0x7FFFFF;

    // Early return for inf/NaN case
    if (exp_val == 128 && !(saturation_mode == SaturateMode::NO_OVERFLOW && man_val == 0)) {
        return origin_float;
    }

    int subnormal_shift = 0;
    if (subnormals) {
        const int spec_exp = (P == 1) ? 1 : 0;
        const int max_exp = (1 << (exp_bits - 1)) - 1;
        const int min_exp = spec_exp - max_exp;

        if (((min_exp - exp_val) <= man_bits) && (exp_val < min_exp)) {
            subnormal_shift = min_exp - exp_val;
        }
    }

    uint32_t uval8 = (P == 1) ? round_bitwise_nearest_p1(uval32, man_bits - subnormal_shift)
                              : round_bitwise_nearest(uval32, man_bits - subnormal_shift);

    uval8 = binary8_clip_exponent(exp_bits, man_bits, uval32, uval8, saturation_mode, subnormals);
    return BITS_TO_FLOAT(&uval8);
}

__device__ float cast_binary8_unsigned_nearest(float origin_float, int P, SaturateMode saturation_mode, bool subnormals) {
    if (origin_float < 0) return NAN;   

    uint32_t uval32 = FLOAT_TO_BITS(&origin_float);
    const int exp_val = (uval32 << 1 >> 24) - 127;
    uint32_t man_val = uval32 & 0x7FFFFF;

    if (exp_val == 128 && !(saturation_mode == SaturateMode::NO_OVERFLOW && man_val == 0)) {
        return origin_float;
    }

    const int exp_bits = 9 - P;
    const int man_bits = P - 1;
    int subnormal_shift = 0;

    if (subnormals) {
        const int max_exp = (1 << (exp_bits - 1)) - 1;
        const int min_exp = (P == 1) - max_exp;
        
        if ((min_exp - exp_val) <= man_bits && exp_val < min_exp){ 
            subnormal_shift = min_exp - exp_val;
        }
    }

    uint32_t uval8 = (P == 1) ? round_bitwise_nearest_p1(uval32, man_bits - subnormal_shift)
                               : round_bitwise_nearest(uval32, man_bits - subnormal_shift);
    
    uval8 = binary8_clip_exponent(exp_bits, man_bits, uval32, uval8, saturation_mode, subnormals);
    
    return BITS_TO_FLOAT(&uval8);
}

__global__ void binary8_signed_nearest_gpu(float *o, float *__restrict__ a, int N, int P, bool is_signed, SaturateMode saturation_mode, bool subnormals)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
      if(is_signed){
        o[index] = cast_binary8_signed_nearest(a[index], P, saturation_mode, subnormals);
      }else{
        o[index] = cast_binary8_unsigned_nearest(a[index], P, saturation_mode, subnormals);
      }
    }
}

// ---------------------------------------------------------------------------------------
// Kernel launchers
void binary8_signed_nearest1(float *o, float *a, int N, int P, const int block_size, bool is_signed, SaturateMode saturation_mode, bool subnormals)
{
    const int grid_size = ceil_div(N, block_size);
    binary8_signed_nearest_gpu<<<grid_size, block_size>>>(o, a, N, P, is_signed, saturation_mode, subnormals);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void binary8_signed_nearest(int kernel_num, float *o, float *a, int N, int P, const int block_size, bool is_signed, SaturateMode saturation_mode, bool subnormals)
{
    switch (kernel_num)
    {
    case 1:
        binary8_signed_nearest1(o, a, N, P, block_size, is_signed, saturation_mode, subnormals);
        break;
    default:
        printf("Invalid kernel number\n");
        exit(EXIT_FAILURE);
    }
}


// ---------------------------------------------------------------------------------------
int main(int argc, const char **argv)
{
    setup_main();

    // // Get signed or unsigned from user input
    // bool is_signed;
    // std::cout << "signed or unsigned, enter true (1) for signed, false (0) for unsigned: ";
    // int is_signed_input;
    // std::cin >> is_signed_input;
    // is_signed = static_cast<bool>(is_signed_input);

    // // Get subnormals from user input
    // bool subnormals;
    // std::cout << "subnormals or not, enter true (1) for subnormals, false (0) for not: ";
    // int subnormals_input;
    // std::cin >> subnormals_input;
    // subnormals = static_cast<bool>(subnormals_input);

    // SaturateMode saturation_mode = SaturateMode::SATURATE;

    SaturateMode saturation_mode = SaturateMode::SATURATE;
    bool subnormals = true;
    bool is_signed = true;
    
    // read the kernel number from the command line
    int kernel_num = 1;
    if (argc > 1)
    {
        kernel_num = atoi(argv[1]);
    }

    // sanity check the CPU reference code - signed and with subnormals
    for (int j = 0; j < sizeof(test_inputs) / sizeof(uint32_t); ++j)
    {
        float fres = cast_binary8_signed_nearest_cpu(BITS_TO_FLOAT(&test_inputs[j]), 3, saturation_mode, subnormals);
        uint32_t res = FLOAT_TO_BITS(&fres);
        if (res != test_outputs[j])
        {
            printf("1. index = %d\n", j);
            print_float(res);
            printf("\nvs\n");
            print_uint32(test_outputs[j]);
            printf("\n");
            //exit(EXIT_FAILURE);
        }
    }

    // sanity check the CPU reference code - signed without subnormals
    for (int j = 0; j < sizeof(test_inputs) / sizeof(uint32_t); ++j)
    {
        float fres = cast_binary8_signed_nearest_cpu(BITS_TO_FLOAT(&test_inputs[j]), 3, saturation_mode, !subnormals);
        uint32_t res = FLOAT_TO_BITS(&fres);
        if (res != test_outputs_no_sub[j])
        {
            printf("2. index = %d\n", j);
            print_float(res);
            printf("\nvs\n");
            print_uint32(test_outputs_no_sub[j]);
            printf("\n");
            //exit(EXIT_FAILURE);
        }
    }

    // sanity check the CPU reference code - unsigned with subnormals
    for (int j = 0; j < sizeof(test_inputs) / sizeof(uint32_t); ++j)
    {
        float fres = cast_binary8_unsigned_nearest_cpu(BITS_TO_FLOAT(&test_inputs_unsigned[j]), 3, saturation_mode, subnormals);
        uint32_t res = FLOAT_TO_BITS(&fres);
        if (res != test_outputs_unsigned[j] && !std::isnan(fres))
        {
            printf("3. index = %d\n", j);
            print_float(res);
            printf("\nvs\n");
            print_uint32(test_outputs_unsigned[j]);
            printf("\n");
            //exit(EXIT_FAILURE);
        }
    }

    // sanity check the CPU reference code - unsigned without subnormals
    for (int j = 0; j < sizeof(test_inputs) / sizeof(uint32_t); ++j)
    {
        float fres = cast_binary8_unsigned_nearest_cpu(BITS_TO_FLOAT(&test_inputs_unsigned[j]), 3, saturation_mode, !subnormals);
        uint32_t res = FLOAT_TO_BITS(&fres);
        if (res != test_outputs_unsigned_w[j] && !std::isnan(fres))
        {
            printf("4. index = %d\n", j);
            print_float(res);
            printf("\nvs\n");
            print_uint32(test_outputs_unsigned_w[j]);
            printf("\n");
            //exit(EXIT_FAILURE);
        }
    }

    int N = 1 << 24;
    int P = 3;

    float *x = make_random_float(N);
    float *y = (float *)malloc(N * sizeof(float));
    float *v = (float *)malloc(N * sizeof(float));
    float *w = (float *)malloc(N * sizeof(float));
    float *u = (float *)malloc(N * sizeof(float));

    printf("Using kernel %d\n", kernel_num);

    // compute reference CPU solution - with subnormals
    binary8_signed_nearest_cpu(y, x, N, P, is_signed, saturation_mode, subnormals);
    // compute reference CPU solution - without subnormals
    binary8_signed_nearest_cpu(v, x, N, P, is_signed, saturation_mode, !subnormals);
    // compute reference CPU solution - with subnormals - unsigned
    binary8_signed_nearest_cpu(w, x, N, P, !is_signed, saturation_mode, subnormals);
    // compute reference CPU solution - without subnormals - unsigned
    binary8_signed_nearest_cpu(u, x, N, P, !is_signed, saturation_mode, !subnormals);

    // move data to the GPU
    float *d_x, *d_y, *d_v, *d_w, *d_u;
    cudaCheck(hipMalloc(&d_x, N * sizeof(float)));

    cudaCheck(hipMalloc(&d_y, N * sizeof(float)));
    cudaCheck(hipMalloc(&d_v, N * sizeof(float)));
    cudaCheck(hipMalloc(&d_w, N * sizeof(float)));
    cudaCheck(hipMalloc(&d_u, N * sizeof(float)));

    cudaCheck(hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice));

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        binary8_signed_nearest(kernel_num, d_y, d_x, N, P, block_size, is_signed, saturation_mode, subnormals);

        float tol = 0.0f;
        validate_result(d_y, y, "y", N, tol);
    }
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        binary8_signed_nearest(kernel_num, d_v, d_x, N, P, block_size, is_signed, saturation_mode, !subnormals);

        float tol = 0.0f;
        validate_result(d_v, v, "v", N, tol);
    }
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        binary8_signed_nearest(kernel_num, d_w, d_x, N, P, block_size, !is_signed, saturation_mode, subnormals);

        float tol = 0.0f;
        validate_result(d_w, w, "w", N, tol);
    }
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        binary8_signed_nearest(kernel_num, d_u, d_x, N, P, block_size, !is_signed, saturation_mode, !subnormals);

        float tol = 0.0f;
        validate_result(d_u, u, "u", N, tol);
    }
    printf("All results match.\n\n");

    printf("\nStarting benchmarks for signed with subnormals.\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];

        int repeat_times = 1000;

        float elapsed_time = benchmark_kernel(repeat_times, binary8_signed_nearest, 
                kernel_num, d_y, d_x, N, P, block_size, is_signed, saturation_mode, subnormals);

        // estimate memory bandwidth achieved
        // for each output element, we do 1 read and 1 write, 4 bytes each
        long memory_ops = N * 2 * (int)sizeof(float);
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;

        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    printf("\nStarting benchmarks for signed without subnormals.\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];

        int repeat_times = 1000;

        float elapsed_time = benchmark_kernel(repeat_times, binary8_signed_nearest, 
                kernel_num, d_v, d_x, N, P, block_size, is_signed, saturation_mode, !subnormals);

        // estimate memory bandwidth achieved
        // for each output element, we do 1 read and 1 write, 4 bytes each
        long memory_ops = N * 2 * (int)sizeof(float);
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;

        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    printf("\nStarting benchmarks for unsigned with subnormals.\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];

        int repeat_times = 1000;

        float elapsed_time = benchmark_kernel(repeat_times, binary8_signed_nearest, 
                kernel_num, d_w, d_x, N, P, block_size, !is_signed, saturation_mode, subnormals);

        // estimate memory bandwidth achieved
        // for each output element, we do 1 read and 1 write, 4 bytes each
        long memory_ops = N * 2 * (int)sizeof(float);
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;

        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    printf("\nStarting benchmarks for unsigned without subnormals.\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];

        int repeat_times = 1000;

        float elapsed_time = benchmark_kernel(repeat_times, binary8_signed_nearest, 
                kernel_num, d_u, d_x, N, P, block_size, !is_signed, saturation_mode, !subnormals);

        // estimate memory bandwidth achieved
        // for each output element, we do 1 read and 1 write, 4 bytes each
        long memory_ops = N * 2 * (int)sizeof(float);
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;

        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    // free memory
    free(x);
    free(y);
    free(v);
    free(w);
    free(u);

    cudaCheck(hipFree(d_x));
    cudaCheck(hipFree(d_y));
    cudaCheck(hipFree(d_v));
    cudaCheck(hipFree(d_w));
    cudaCheck(hipFree(d_u));

    return 0;
}