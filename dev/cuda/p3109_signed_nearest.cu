#include "hip/hip_runtime.h"
/*
Kernels for IEEE-754 down casting from binary32 to a lower precision format.
Payload is still a binary32 value.

Compile example:
nvcc -O3 p3109_signed_nearest.cu -o p3109_signed_nearest -std=c++17 -lcublas

version 1 attempted to make the code as compact as possible, while also 
maintaining readability; bit shifts and masking are used aplenty
./p3109_signed_nearest 1

*/

#include <hip/hip_runtime.h>
#include "common.h"

enum class SaturateMode {
    SATURATE,
    NO_OVERFLOW,
    OVERFLOWS
};

// --------------------------------------------------------------------------------------
// I/O pairs to sanity check the CPU reference code (E5M2 without subnormals, RNE)
uint32_t test_inputs[] = {
    0b00111000000000000000000000000000,
    0b00110111000000000000000000000000,
    0b00110110100000000000000000000000,
    0b00110110100000000000000000000001,
    0b01000111010000000000000000000000,
    0b01000111011000000000000000000000,
    0b00110110000000000000000000000000};
uint32_t test_outputs[] = {
    0b00111000000000000000000000000000,
    0b00110111000000000000000000000000,
    0b00000000000000000000000000000000,
    0b00110111000000000000000000000000,
    0b01000111010000000000000000000000,
    0b01000111010000000000000000000000,
    0b00000000000000000000000000000000};


uint32_t round_bitwise_nearest_cpu(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}

uint32_t p3109_clip_exponent_cpu(int exp_bits, int man_bits, uint32_t old_num, uint32_t quantized_num, SaturateMode saturation_mode, bool subnormal) {

  if (quantized_num == 0){
    return quantized_num;
  }
  
  uint32_t man_val = quantized_num & 0x7FFFFF;
  uint32_t old_sign = old_num >> 31 << 31;
  uint32_t max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  
  int spec_exp = (man_bits == 0) ? 1 : 0; // if P = 1
  int special_p1 = 0;
  
  if(exp_bits == 8 && saturation_mode != SaturateMode::NO_OVERFLOW){ // unsigned and p=1
      special_p1 = 1; // 0 bit of mantissa so the max value 0xfd = max_exp - 1 | mantissa = 0
  }else if (exp_bits == 7 &&  man_bits == 1 && saturation_mode != SaturateMode::NO_OVERFLOW){ // unsigned and p=2 
      special_p1 = 1;
  }else if(exp_bits + man_bits == 8){ // unsigned
      max_man = ((1u << man_bits) - 3u) << (23 - man_bits); // 2+ bit of mantissa so the max value 0xfd = mACax_exp | max_mantissa - 1 
  }

  // Special because in unsigned we want our min to be 1 less because the space is taken by the Nan
  int quantized_exponent_store = (quantized_num << 1 >> 24);
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127 - special_p1; 
  int min_exponent_store = -((1 << (exp_bits - 1)) - 1) + 127 + spec_exp;

  if (saturation_mode == SaturateMode::NO_OVERFLOW) { // Saturate to max without infinity
    max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  }

  if (quantized_exponent_store > max_exponent_store || ((quantized_exponent_store == max_exponent_store) && (man_val > max_man))) 
  {
    if (saturation_mode == SaturateMode::OVERFLOWS){ // Overflow to infinity
        return quantized_num = old_sign | 0x7F800000; // INF
    } 
    return quantized_num = old_sign | ((uint32_t)max_exponent_store << 23) | max_man;
  }
  if (quantized_exponent_store < min_exponent_store) {
    if (subnormal) {
        int subnormal_shift = min_exponent_store - quantized_exponent_store;
        int min_subnormals_exp = min_exponent_store - man_bits;
        uint32_t min_num = ((uint32_t)min_subnormals_exp << 23);
        uint32_t middle_num = ((uint32_t)(min_subnormals_exp - 1) << 23);
      if (subnormal_shift <= man_bits) {
        // quantized_num stays the same in this case
      } else if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    } 
    if(!subnormal) {
      uint32_t min_num = (uint32_t)min_exponent_store << 23;
      uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23);
      if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    }
  }
  return quantized_num;
}

float cast_p3109_signed_nearest_cpu(float origin_float, int P, SaturateMode saturation_mode, bool subnormals) {

    int exp_bits = 8-P;
    int man_bits = P-1;    
    int subnormal_shift = 0;
    uint32_t uval32, uval8;
    float fval8;

    uval32 = FLOAT_TO_BITS(&origin_float);

    int exp_val = (uval32 << 1 >> 24) - 127;
    uint32_t man_val = uval32 & 0x7FFFFF;

    if (exp_val == 128 && !(saturation_mode == SaturateMode::NO_OVERFLOW && man_val == 0)) {             // inf/Nan case
        return origin_float;
    }

    if (subnormals){
        int spec_exp = (P == 1) ? 1 : 0;
        int max_exp = (1 << (exp_bits -1)) - 1;    // minimal and maximal exponent value in binary8
        int min_exp = spec_exp - max_exp;

        if(((min_exp - exp_val) <= man_bits) && (exp_val < min_exp) && (subnormals)){ 
            subnormal_shift = min_exp - exp_val;
        }
    }

    uval8 = round_bitwise_nearest_cpu(uval32, man_bits - subnormal_shift);
    uval8 = p3109_clip_exponent_cpu(exp_bits, man_bits, uval32, uval8, saturation_mode, subnormals);
    fval8 = BITS_TO_FLOAT(&uval8);

    return fval8;
}

float cast_p3109_unsigned_nearest_cpu(float origin_float, int P, SaturateMode saturation_mode, bool subnormals) {
    // we had talks abt the following for unsigned, P = 1:
    // 0: 0000 0000
    // NaN: FE or FF (currently. it is 1000 0000 in this revision)
    // inf: FE of FF (currently, it is FF)
    // max float: FD (currently, it is FE)
    // we have a variation that allows for this in the special condition where 
    
    if (origin_float < 0){
      return NAN;
    }
 
    int exp_bits = 8 - P + 1;
    int man_bits = P - 1; 
    int subnormal_shift = 0;
    uint32_t uval32, uval8;
    float fval8;

    uval32 = FLOAT_TO_BITS(&origin_float);

    int exp_val = (uval32 << 1 >> 24) - 127;
    uint32_t man_val = uval32 & 0x7FFFFF;
    
    if (exp_val == 128 && !(saturation_mode == SaturateMode::NO_OVERFLOW && man_val == 0)) {  // return inf/Nan expect in the case of no_overflow && inf
        return origin_float;
    }

    if (subnormals){
        int spec_exp = (P == 1) ? 1 : 0;
        int max_exp = (1 << (exp_bits -1)) - 1;
        int min_exp = spec_exp - max_exp;
           
        if((min_exp - exp_val) <= man_bits && exp_val < min_exp && subnormals){ 
            subnormal_shift = min_exp - exp_val;
        }
    }

    uval8 = round_bitwise_nearest_cpu(uval32, man_bits - subnormal_shift);
    uval8 = p3109_clip_exponent_cpu(exp_bits, man_bits, uval32, uval8, saturation_mode, subnormals);
    fval8 = BITS_TO_FLOAT(&uval8);

    return fval8;
}

void p3109_signed_nearest_cpu(float *o, float *a, int N, int P, bool is_signed, SaturateMode saturation_mode, bool subnormals) {
  for (int i = 0; i < N; ++i){
    if(is_signed){
      o[i] = cast_p3109_signed_nearest_cpu(a[i], P, saturation_mode, subnormals);
    }
    else{
      o[i] = cast_p3109_unsigned_nearest_cpu(a[i], P, saturation_mode, subnormals);
    }
  }
}

// ---------------------------------------------------------------------------------------
// GPU kernels
__device__ __forceinline__ uint32_t
round_bitwise_nearest(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}

__device__ __forceinline__ uint32_t
p3109_clip_exponent(int exp_bits, int man_bits, uint32_t old_num, uint32_t quantized_num, SaturateMode saturation_mode, bool subnormal) {

  if (quantized_num == 0){
    return quantized_num;
  }
  
  uint32_t man_val = quantized_num & 0x7FFFFF;
  uint32_t old_sign = old_num >> 31 << 31;
  uint32_t max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  
  int spec_exp = (man_bits == 0) ? 1 : 0; // if P = 1
  int special_p1 = 0;
  
  if(exp_bits == 8 && saturation_mode != SaturateMode::NO_OVERFLOW){ // unsigned and p=1
      special_p1 = 1; // 0 bit of mantissa so the max value 0xfd = max_exp - 1 | mantissa = 0
  }else if (exp_bits == 7 &&  man_bits == 1 && saturation_mode != SaturateMode::NO_OVERFLOW){ // unsigned and p=2 
      special_p1 = 1;
  }else if(exp_bits + man_bits == 8){ // unsigned
      max_man = ((1u << man_bits) - 3u) << (23 - man_bits); // 2+ bit of mantissa so the max value 0xfd = mACax_exp | max_mantissa - 1 
  }

  // Special because in unsigned we want our min to be 1 less because the space is taken by the Nan
  int quantized_exponent_store = (quantized_num << 1 >> 24);
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127 - special_p1; 
  int min_exponent_store = -((1 << (exp_bits - 1)) - 1) + 127 + spec_exp;

  if (saturation_mode == SaturateMode::NO_OVERFLOW) { // Saturate to max without infinity
    max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  }

  if (quantized_exponent_store > max_exponent_store || ((quantized_exponent_store == max_exponent_store) && (man_val > max_man))) 
  {
    if (saturation_mode == SaturateMode::OVERFLOWS){ // Overflow to infinity
        return quantized_num = old_sign | 0x7F800000; // INF
    } 
    return quantized_num = old_sign | ((uint32_t)max_exponent_store << 23) | max_man;
  }
  if (quantized_exponent_store < min_exponent_store) {
    if (subnormal) {
        int subnormal_shift = min_exponent_store - quantized_exponent_store;
        int min_subnormals_exp = min_exponent_store - man_bits;
        uint32_t min_num = ((uint32_t)min_subnormals_exp << 23);
        uint32_t middle_num = ((uint32_t)(min_subnormals_exp - 1) << 23);
      if (subnormal_shift <= man_bits) {
        // quantized_num stays the same in this case
      } else if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    } 
    if(!subnormal) {
      uint32_t min_num = (uint32_t)min_exponent_store << 23;
      uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23);
      if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    }
  }
  return quantized_num;
}

__device__ float cast_p3109_signed_nearest(float origin_float, int P, SaturateMode saturation_mode, bool subnormals) {

    int exp_bits = 8-P;
    int man_bits = P-1;    
    int subnormal_shift = 0;
    uint32_t uval32, uval8;
    float fval8;

    uval32 = FLOAT_TO_BITS(&origin_float);

    int exp_val = (uval32 << 1 >> 24) - 127;
    uint32_t man_val = uval32 & 0x7FFFFF;

    if (exp_val == 128 && !(saturation_mode == SaturateMode::NO_OVERFLOW && man_val == 0)) {             // inf/Nan case
        return origin_float;
    }

    if (subnormals){
        int spec_exp = (P == 1) ? 1 : 0;
        int max_exp = (1 << (exp_bits -1)) - 1;    // minimal and maximal exponent value in binary8
        int min_exp = spec_exp - max_exp;

        if(((min_exp - exp_val) <= man_bits) && (exp_val < min_exp) && (subnormals)){ 
            subnormal_shift = min_exp - exp_val;
        }
    }

    uval8 = round_bitwise_nearest(uval32, man_bits - subnormal_shift);
    uval8 = p3109_clip_exponent(exp_bits, man_bits, uval32, uval8, saturation_mode, subnormals);
    fval8 = BITS_TO_FLOAT(&uval8);

    return fval8;
}

__device__ float cast_p3109_unsigned_nearest(float origin_float, int P, SaturateMode saturation_mode, bool subnormals) {
    // we had talks abt the following for unsigned, P = 1:
    // 0: 0000 0000
    // NaN: FE or FF (currently. it is 1000 0000 in this revision)
    // inf: FE of FF (currently, it is FF)
    // max float: FD (currently, it is FE)
    // we have a variation that allows for this in the special condition where 
    
    if (origin_float < 0){
      return NAN;
    }
 
    int exp_bits = 8 - P + 1;
    int man_bits = P - 1; 
    int subnormal_shift = 0;
    uint32_t uval32, uval8;
    float fval8;

    uval32 = FLOAT_TO_BITS(&origin_float);

    int exp_val = (uval32 << 1 >> 24) - 127;
    uint32_t man_val = uval32 & 0x7FFFFF;
    
    if (exp_val == 128 && !(saturation_mode == SaturateMode::NO_OVERFLOW && man_val == 0)) {  // return inf/Nan expect in the case of no_overflow && inf
        return origin_float;
    }

    if (subnormals){
        int spec_exp = (P == 1) ? 1 : 0;
        int max_exp = (1 << (exp_bits -1)) - 1;
        int min_exp = spec_exp - max_exp;
           
        if((min_exp - exp_val) <= man_bits && exp_val < min_exp && subnormals){ 
            subnormal_shift = min_exp - exp_val;
        }
    }

    uval8 = round_bitwise_nearest(uval32, man_bits - subnormal_shift);
    uval8 = p3109_clip_exponent(exp_bits, man_bits, uval32, uval8, saturation_mode, subnormals);
    fval8 = BITS_TO_FLOAT(&uval8);

    return fval8;
}

__global__ void p3109_signed_nearest_gpu(float *o, float *__restrict__ a, int N, int P, bool is_signed, SaturateMode saturation_mode, bool subnormals)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
      if(is_signed){
        o[index] = cast_p3109_signed_nearest(a[index], P, saturation_mode, subnormals);
      }else{
        o[index] = cast_p3109_unsigned_nearest(a[index], P, saturation_mode, subnormals);
      }
    }
}

// ---------------------------------------------------------------------------------------
// Kernel launchers
void p3109_signed_nearest1(float *o, float *a, int N, int P, const int block_size, bool is_signed, SaturateMode saturation_mode, bool subnormals)
{
    const int grid_size = ceil_div(N, block_size);
    p3109_signed_nearest_gpu<<<grid_size, block_size>>>(o, a, N, P, is_signed, saturation_mode, subnormals);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void p3109_signed_nearest(int kernel_num, float *o, float *a, int N, int P, const int block_size, bool is_signed, SaturateMode saturation_mode, bool subnormals)
{
    switch (kernel_num)
    {
    case 1:
        p3109_signed_nearest1(o, a, N, P, block_size, is_signed, saturation_mode, subnormals);
        break;
    default:
        printf("Invalid kernel number\n");
        exit(EXIT_FAILURE);
    }
}


// ---------------------------------------------------------------------------------------
int main(int argc, const char **argv)
{
    setup_main();

    // Get signed or unsigned from user input
    bool is_signed;
    std::cout << "signed or unsigned, enter true (1) for signed, false (0) for unsigned: ";
    int is_signed_input;
    std::cin >> is_signed_input;
    is_signed = static_cast<bool>(is_signed_input);

    // Get subnormals from user input
    bool subnormals;
    std::cout << "subnormals or not, enter true (1) for subnormals, false (0) for not: ";
    int subnormals_input;
    std::cin >> subnormals_input;
    subnormals = static_cast<bool>(subnormals_input);

    SaturateMode saturation_mode = SaturateMode::SATURATE;


    // read the kernel number from the command line
    int kernel_num = 1;
    if (argc > 1)
    {
        kernel_num = atoi(argv[1]);
    }

    // sanity check the CPU reference code
    for (int j = 0; j < sizeof(test_inputs) / sizeof(uint32_t); ++j)
    {
        float fres = cast_p3109_signed_nearest_cpu(BITS_TO_FLOAT(&test_inputs[j]), 3, saturation_mode, subnormals);
        uint32_t res = FLOAT_TO_BITS(&fres);
        if (res != test_outputs[j])
        {
            printf("index = %d\n", j);
            print_float(res);
            printf("\nvs\n");
            print_uint32(test_outputs[j]);
            printf("\n");
            //exit(EXIT_FAILURE);
        }
    }

    int N = 1 << 24;
    int P = 3;

    float *x = make_random_float(N);
    float *y = (float *)malloc(N * sizeof(float));

    printf("Using kernel %d\n", kernel_num);

    // compute reference CPU solution
    p3109_signed_nearest_cpu(y, x, N, P, is_signed, saturation_mode, subnormals);

    // move data to the GPU
    float *d_x, *d_y;
    cudaCheck(hipMalloc(&d_x, N * sizeof(float)));
    cudaCheck(hipMalloc(&d_y, N * sizeof(float)));

    cudaCheck(hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice));

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        p3109_signed_nearest(kernel_num, d_y, d_x, N, P, block_size, is_signed, saturation_mode, subnormals);

        float tol = 0.0f;
        validate_result(d_y, y, "y", N, tol);
    }

    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];

        int repeat_times = 1000;

        float elapsed_time = benchmark_kernel(repeat_times, p3109_signed_nearest, 
                kernel_num, d_y, d_x, N, P, block_size, is_signed, saturation_mode, subnormals);

        // estimate memory bandwidth achieved
        // for each output element, we do 1 read and 1 write, 4 bytes each
        long memory_ops = N * 2 * (int)sizeof(float);
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;

        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    // free memory
    free(x);
    free(y);

    cudaCheck(hipFree(d_x));
    cudaCheck(hipFree(d_y));

    return 0;
}