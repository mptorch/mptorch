/*
Batched matrix-matrix multiply using hipblasGemmBatchedEx function.

Compile example:
nvcc -O3 cublas_bgemm.cu -o cublas_bgemm -lcublas

Version 1 uses separate memory allocation for each matrices A[i], B[i] and C[i]
./cublas_bgemm 1

Version 2 stores batch matrcies A, B and C in a single array and passes the pointer
to the beginning of each matrix A[i], B[i] and C[i] to hipblasGemmBatchedEx
./cublas_bgemm 2
*/

#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <stdio.h>
#include "common.h"

// ---------------------------------------------------------------------------------------
/* Host (CPU) implementation of a simple version of sgemm */
static void simple_bsgemm(int P, int M, int N, int K, float alpha, const float **A, const float **B,
                          float beta, float **C)
{
    for (int p = 0; p < P; p++)
    {
        for (int i = 0; i < M; ++i)
        {
            for (int j = 0; j < N; ++j)
            {
                float prod = 0.0f;
                for (int k = 0; k < K; ++k)
                {
                    prod = prod + A[p][k * M + i] * B[p][j * K + k];
                }
                C[p][j * M + i] = alpha * prod + beta * C[p][j * M + i];
            }
        }
    }
}

// ---------------------------------------------------------------------------------------
/*This function allocates distinct memory locations for each matrices A[i], B[i] and C[i] */
void main_separate_arrays(hipblasHandle_t handle, int P, int M, int N, int K)
{
    printf("CUBLAS GEMM %dx%dx%d (MxNxK) using separate arrays...\n", M, N, K);

    float **h_A, **h_B, **h_C;
    float **h_dA, **h_dB, **h_dC;
    float **d_dA, **d_dB, **d_dC;
    float alpha = 1.0f;
    float beta = 0.0f;

    h_A = new float *[P];
    h_B = new float *[P];
    h_C = new float *[P];
    h_dA = new float *[P];
    h_dB = new float *[P];
    h_dC = new float *[P];

    for (int i = 0; i < P; i++)
    {
        h_A[i] = make_random_float(M * K);
        h_B[i] = make_random_float(K * N);
        h_C[i] = make_zeros_float(M * N);
    }

    /* Allocate device memory for the matrices */
    for (int i = 0; i < P; i++)
    {
        cudaCheck(hipMalloc(&h_dA[i], M * K * sizeof(float)));
        cudaCheck(hipMalloc(&h_dB[i], K * N * sizeof(float)));
        cudaCheck(hipMalloc(&h_dC[i], M * N * sizeof(float)));

        cudaCheck(hipMemcpy(h_dA[i], h_A[i], M * K * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(h_dB[i], h_B[i], K * N * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(h_dC[i], h_C[i], M * N * sizeof(float), hipMemcpyHostToDevice));
    }

    cudaCheck(hipMalloc(&d_dA, P * sizeof(float *)));
    cudaCheck(hipMalloc(&d_dB, P * sizeof(float *)));
    cudaCheck(hipMalloc(&d_dC, P * sizeof(float *)));

    cudaCheck(hipMemcpy(d_dA, h_dA, P * sizeof(float *), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_dB, h_dB, P * sizeof(float *), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_dC, h_dC, P * sizeof(float *), hipMemcpyHostToDevice));

    int lda = M;
    int ldb = K;
    int ldc = M;

    /* Performs operation using cublas */
    auto cublas_bgemm = [&]()
    {
        cublasCheck(hipblasGemmBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha,
                                        (void **)d_dA, HIP_R_32F, lda,
                                        (void **)d_dB, HIP_R_32F, ldb, &beta,
                                        (void **)d_dC, HIP_R_32F, ldc, P,
                                        HIPBLAS_COMPUTE_32F,
                                        HIPBLAS_GEMM_DEFAULT));
    };
    cublas_bgemm();

    /* Performs operation using reference CPU C code */
    simple_bsgemm(P, M, N, K, alpha, (const float **)h_A, (const float **)h_B, beta, h_C);

    /* Check CUBLAS batched GEMM result against reference */
    for (int i = 0; i < P; i++)
    {
        validate_result(h_dC[i], h_C[i], "C", M * N, 1.0e-3f);
        printf("\n");
    }
    printf("All results match. Starting benchmarks.\n\n");

    /* Benchmark */
    int repeat_times = 1000;
    float elapsed_time = benchmark_gpu_kernel(repeat_times, cublas_bgemm);
    printf("time %.4f ms\n", elapsed_time);

    /* Memory clean up */
    for (int i = 0; i < P; i++)
    {
        free(h_A[i]);
        free(h_B[i]);
        free(h_C[i]);

        cudaCheck(hipFree(h_dA[i]));
        cudaCheck(hipFree(h_dB[i]));
        cudaCheck(hipFree(h_dC[i]));
    }

    cudaCheck(hipFree(d_dA));
    cudaCheck(hipFree(d_dB));
    cudaCheck(hipFree(d_dC));

    delete[] h_dA;
    delete[] h_dB;
    delete[] h_dC;
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
}

/*
 * This version uses a single contiguous array to store the batches of matrices A, B and C,
 * and the beginning address of each matrix is computed and passed to hipblasGemmBatchedEx.
 */
void main_shared_arrays(hipblasHandle_t handle, int P, int M, int N, int K)
{
    printf("CUBLAS GEMM %dx%dx%d (MxNxK) using shared arrays...\n", M, N, K);

    float **h_A, **h_B, **h_C;
    float *d_A, *d_B, *d_C;
    float **h_dA, **h_dB, **h_dC;
    float **d_dA, **d_dB, **d_dC;
    float alpha = 1.0f;
    float beta = 0.0f;

    h_A = new float *[P];
    h_B = new float *[P];
    h_C = new float *[P];
    h_dA = new float *[P];
    h_dB = new float *[P];
    h_dC = new float *[P];

    for (int i = 0; i < P; i++)
    {
        h_A[i] = make_random_float(M * K);
        h_B[i] = make_random_float(K * N);
        h_C[i] = make_zeros_float(M * N);
    }

    /* Allocate device memory for the matrices */
    cudaCheck(hipMalloc(&d_A, P * M * K * sizeof(float)));
    cudaCheck(hipMalloc(&d_B, P * K * N * sizeof(float)));
    cudaCheck(hipMalloc(&d_C, P * M * N * sizeof(float)));

    for (int i = 0; i < P; i++)
    {
        cudaCheck(hipMemcpy(d_A + i * M * K, h_A[i], M * K * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(d_B + i * K * N, h_B[i], K * N * sizeof(float), hipMemcpyHostToDevice));
        cudaCheck(hipMemcpy(d_C + i * M * N, h_C[i], M * N * sizeof(float), hipMemcpyHostToDevice));
    }

    // Retrieve each pointers to each matrices
    for (int i = 0; i < P; i++)
    {
        h_dA[i] = d_A + i * M * K;
        h_dB[i] = d_B + i * K * N;
        h_dC[i] = d_C + i * M * N;
    }

    cudaCheck(hipMalloc(&d_dA, P * sizeof(float *)));
    cudaCheck(hipMalloc(&d_dB, P * sizeof(float *)));
    cudaCheck(hipMalloc(&d_dC, P * sizeof(float *)));

    cudaCheck(hipMemcpy(d_dA, h_dA, P * sizeof(float *), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_dB, h_dB, P * sizeof(float *), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_dC, h_dC, P * sizeof(float *), hipMemcpyHostToDevice));

    int lda = M;
    int ldb = K;
    int ldc = M;

    /* Performs operation using cublas */
    auto cublas_bgemm = [&]()
    {
        cublasCheck(hipblasGemmBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha,
                                        (void **)d_dA, HIP_R_32F, lda,
                                        (void **)d_dB, HIP_R_32F, ldb, &beta,
                                        (void **)d_dC, HIP_R_32F, ldc, P,
                                        HIPBLAS_COMPUTE_32F,
                                        HIPBLAS_GEMM_DEFAULT));
    };
    cublas_bgemm();

    // /* Performs operation using reference CPU C code */
    simple_bsgemm(P, M, N, K, alpha, (const float **)h_A, (const float **)h_B, beta, h_C);

    /* Check result against reference */
    for (int i = 0; i < P; i++)
    {
        validate_result(d_C + i * M * N, h_C[i], "C", M * N, 1.0e-3f);
        printf("\n");
    }
    printf("All results match. Starting benchmarks.\n\n");

    /* Benchmark */
    int repeat_times = 1000;
    float elapsed_time = benchmark_gpu_kernel(repeat_times, cublas_bgemm);
    printf("time %.4f ms\n", elapsed_time);

    /* Memory clean up */
    for (int i = 0; i < P; i++)
    {
        free(h_A[i]);
        free(h_B[i]);
        free(h_C[i]);
    }

    cudaCheck(hipFree(d_A));
    cudaCheck(hipFree(d_B));
    cudaCheck(hipFree(d_C));

    cudaCheck(hipFree(d_dA));
    cudaCheck(hipFree(d_dB));
    cudaCheck(hipFree(d_dC));

    delete[] h_dA;
    delete[] h_dB;
    delete[] h_dC;
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
}

// ---------------------------------------------------------------------------------------
int main(int argc, const char **argv)
{
    setup_main();

    const int M = 512;
    const int N = 512;
    const int K = 1024;
    const int P = 3; // batch count
    hipblasHandle_t handle;

    /* Initialize CUBLAS */
    cublasCheck(hipblasCreate(&handle));

    int version = 1;
    if (argc > 1)
    {
        version = atoi(argv[1]);
    }

    switch (version)
    {
    case 1:
        main_separate_arrays(handle, P, M, N, K);
        break;
    case 2:
        main_shared_arrays(handle, P, M, N, K);
        break;
    default:
        printf("Invalid version number\n");
        exit(1);
    }

    return 0;
}
