#include "hip/hip_runtime.h"
/*
Low-precision float softmax forward along any dimension.

Compile example:
nvcc -O3 softmax_backward.cu -o softmax_backward -lcublas

Simple implementation parallelizing over the rows that were softmaxed, one thread per row
./softmax_backward 1 [dim=-3..2]

Efficient version using intra-warp and inter-warp reductions, block_size % 32 = 0
./softmax_backward 2 [dim=-3..2]

*/

#include <hip/hip_runtime.h>
#include <cmath>
#include <chrono>
#include "common.h"

struct DimSizes
{
    int outer;
    int inner;
    int channel;
};

/* Helper function for tensor striding */
// ---------------------------------------------------------------------------------------

DimSizes partition_tensor(const int *dims, int n_dims, int dim)
{
    DimSizes sizes;
    int real_dim = (n_dims + (dim % n_dims)) % n_dims;
    sizes.outer = 1;
    sizes.channel = dims[real_dim];
    sizes.inner = 1;
    for (int i = 0; i < real_dim; ++i)
    {
        sizes.outer *= dims[i];
    }
    for (int i = real_dim + 1; i < n_dims; ++i)
    {
        sizes.inner *= dims[i];
    }
    return sizes;
}

/* Quantization function and wrapper */
// ---------------------------------------------------------------------------------------

#define FLOAT_TO_BITS(x) (*reinterpret_cast<uint32_t *>(x))
#define BITS_TO_FLOAT(x) (*reinterpret_cast<float *>(x))

__host__ __device__ __forceinline__ uint32_t round_bitwise_nearest(uint32_t target, int man_bits)
{
    uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
    uint32_t machine_eps = 1 << (22 - man_bits);
    // tie breaking rule offset
    int offset = (down == machine_eps);
    uint32_t add_r = target + machine_eps;
    // apply the mask
    // this is the analogue of how you would do round
    // to nearest integer using the floor function:
    // round(x) = floor(x + 0.5)
    return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}

__host__ __device__ uint32_t clip_exponent_with_subnormals(int exp_bits, int man_bits, uint32_t old_num,
                                                           uint32_t quantized_num, bool saturate = false)
{
    if (quantized_num == 0)
        return quantized_num;

    int quantized_exponent_store = quantized_num << 1 >> 24;
    int min_exponent_store = -((1 << (exp_bits - 1)) - 2) - man_bits + 127;

    uint32_t old_sign = old_num >> 31 << 31;
    // underflow or round to smallest non zero subnormal value
    if (quantized_exponent_store < min_exponent_store)
    {
        int offset = (quantized_exponent_store == (min_exponent_store - 1));
        quantized_num += offset * (1u << 23);
        quantized_num = quantized_num | old_sign;
        quantized_num = offset * quantized_num;
    }
    return quantized_num;
}

__host__ __device__ uint32_t clip_exponent_without_subnormals(int exp_bits, int man_bits, uint32_t old_num,
                                                              uint32_t quantized_num, bool saturate = false)
{
    if (quantized_num == 0)
        return quantized_num;

    int quantized_exponent_store = quantized_num << 1 >> 24;
    int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127;
    int min_exponent_store = -((1 << (exp_bits - 1)) - 2) + 127;

    uint32_t old_sign = old_num >> 31 << 31;
    // saturate or overflow
    if (quantized_exponent_store > max_exponent_store)
    {
        if (saturate)
        {
            uint32_t max_man =
                (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits);
            uint32_t max_num = ((uint32_t)max_exponent_store << 23) | max_man;
            quantized_num = old_sign | max_num;
        }
        else
        {
            quantized_num = ((((uint32_t)1 << 31) - 1) ^ (((uint32_t)1 << 23) - 1));
            quantized_num = quantized_num | old_sign;
        }
    } // underflow or round to smallest nonzero normal value
    else if (quantized_exponent_store < min_exponent_store)
    {
        uint32_t offset = (quantized_exponent_store == (min_exponent_store - 1)) && ((old_num << 9 >> 9) > (1 << 22));
        quantized_num = offset * (min_exponent_store << 23);
        quantized_num |= old_sign;
    }
    return quantized_num;
}

__host__ __device__ float cast_fp_nearest(float origin_float, int man_bits, int exp_bits,
                                          bool subnormal_support = true,
                                          bool saturate = false)
{
    uint32_t target, quantize_bits;
    target = FLOAT_TO_BITS(&origin_float);
    float quantized;

    int target_exp = (target << 1 >> 1 >> 23) - 127;
    int min_exp = -((1 << (exp_bits - 1)) - 2);
    bool subnormal = (target_exp < min_exp);
    bool noquantize = (man_bits >= 23) && (exp_bits >= 8);

    if (noquantize)
    {
        quantized = origin_float;
    }
    else
    {
        // handle subnormal inputs (if subnormal mode is active)
        if (subnormal && subnormal_support)
        {
            int exp_diff = man_bits - (min_exp - target_exp);
            int not_uflow = exp_diff > -1 || ((exp_diff == -1) && ((target << 9) > 0));
            quantize_bits = not_uflow * round_bitwise_nearest(target, exp_diff);
            quantize_bits =
                clip_exponent_with_subnormals(exp_bits, man_bits, target, quantize_bits, saturate);
            quantized = BITS_TO_FLOAT(&quantize_bits);
        }
        // handle NaN/inf inputs
        else if (target_exp == 128)
        {
            quantized = origin_float;
        }
        // normal value range or overflow
        else
        {
            quantize_bits = round_bitwise_nearest(target, man_bits);
            quantize_bits =
                clip_exponent_without_subnormals(exp_bits, man_bits, target, quantize_bits, saturate);
            quantized = BITS_TO_FLOAT(&quantize_bits);
        }
    }

    return quantized;
}

__host__ __device__ float quant_add(float origin_float)
{
    return cast_fp_nearest(origin_float, 22, 8, true, false);
}

__host__ __device__ float quant_mul(float origin_float)
{
    return cast_fp_nearest(origin_float, 10, 8, true, false);
}

// ---------------------------------------------------------------------------------------
/* Host (CPU) implementation of a simple softmax backward */
static void softmax_backward_cpu(const float *input_array, const float *out_gradient, float *output_array, const int *dims, int n_dims, int dim)
{
    auto sizes = partition_tensor(dims, n_dims, dim);

    for (int i = 0; i < sizes.outer * sizes.inner; ++i)
    {
        int outer_idx = i / sizes.inner;
        int inner_idx = i % sizes.inner;

        int base_index = outer_idx * sizes.channel * sizes.inner + inner_idx;
        const float *input = input_array + base_index;
        const float *grad = out_gradient + base_index;
        float *output = output_array + base_index;

        float weighted_grad_sum = 0.f;
        for (int k = 0; k < sizes.channel; ++k)
        {
            int idx = k * sizes.inner;
            float prod = quant_mul(input[idx] * grad[idx]);
            weighted_grad_sum = quant_add(weighted_grad_sum + prod);
        }

        for (int k = 0; k < sizes.channel; ++k)
        {
            int idx = k * sizes.inner;
            float a = quant_add(grad[idx] - weighted_grad_sum);
            output[idx] = quant_mul(a * input[idx]);
        }
    }
}

// ---------------------------------------------------------------------------------------
/* Device (CUDA) softmax kernels */

__global__ void softmax_backward_kernel1(const float *__restrict__ input_array, const float *__restrict__ out_gradient, float *output_array,
                                         const DimSizes sizes, int N)
{
    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if (id >= N)
        return;

    int outer_idx = id / sizes.inner;
    int inner_idx = id % sizes.inner;

    int base_index = outer_idx * sizes.channel * sizes.inner + inner_idx;
    const float *input = input_array + base_index;
    const float *grad = out_gradient + base_index;
    float *output = output_array + base_index;

    float weighted_grad_sum = 0.f;
    for (int k = 0; k < sizes.channel; ++k)
    {
        int idx = k * sizes.inner;
        float prod = quant_mul(input[idx] * grad[idx]);
        weighted_grad_sum = quant_add(weighted_grad_sum + prod);
    }

    for (int k = 0; k < sizes.channel; ++k)
    {
        int idx = k * sizes.inner;
        float a = quant_add(grad[idx] - weighted_grad_sum);
        output[idx] = quant_mul(a * input[idx]);
    }
}

__global__ void softmax_backward_kernel2(const float *__restrict__ input_array, const float *__restrict__ out_gradient, float *output_array,
                                         const DimSizes sizes)
{
    extern __shared__ float shared[];

    int tid = threadIdx.x;
    int warp = threadIdx.x / warpSize; // which warp within this block the thread belongs to
    int lane = threadIdx.x % warpSize; // the id of the thread in its warp

    int warpsPerBlock = blockDim.x / warpSize;

    int outer_idx = blockIdx.x / sizes.inner;
    int inner_idx = blockIdx.x % sizes.inner;

    int base_index = outer_idx * sizes.channel * sizes.inner + inner_idx;
    const float *input = input_array + base_index;
    const float *grad = out_gradient + base_index;
    float *output = output_array + base_index;

    // Compute the input row sum and weighted sum

    float *shared_weighted_grad_sum = &shared[0];

    float weighted_grad_sum = 0.f;
    for (int k = tid; k < sizes.channel; k += blockDim.x)
    {
        int idx = k * sizes.inner;
        float prod = quant_mul(input[idx] * grad[idx]);
        weighted_grad_sum = quant_add(weighted_grad_sum + prod);
    }
    for (int offset = warpSize / 2; offset > 0; offset /= 2)
    {
        weighted_grad_sum = quant_add(weighted_grad_sum + __shfl_down_sync(0xFFFFFFFF, weighted_grad_sum, offset));
    }
    if (lane == 0)
    {
        shared_weighted_grad_sum[warp] = weighted_grad_sum;
    }
    __syncthreads();
    if (tid == 0)
    {
        weighted_grad_sum = shared_weighted_grad_sum[0];
        for (int i = 1; i < warpsPerBlock; i++)
        {
            weighted_grad_sum = quant_add(weighted_grad_sum + shared_weighted_grad_sum[i]);
        }
        shared_weighted_grad_sum[0] = weighted_grad_sum;
    }
    __syncthreads();
    weighted_grad_sum = shared_weighted_grad_sum[0];

    // Last step, subsrtact the weighted sum from the gradient, and divide by input sum
    for (int k = tid; k < sizes.channel; k += blockDim.x)
    {
        int idx = k * sizes.inner;
        float a = quant_add(grad[idx] - weighted_grad_sum);
        output[idx] = quant_mul(a * input[idx]);
    }
}

// ---------------------------------------------------------------------------------------
/* Kernel launchers */

void softmax_backward_cuda1(float *input, float *out_grad, float *output, const int *dims, int n_dims, int dim, int block_size)
{
    DimSizes sizes = partition_tensor(dims, n_dims, dim);
    // one thread per row that was softmaxed
    int N = sizes.outer * sizes.inner; // number of rows
    int blocks = N / block_size + (N % block_size != 0);
    softmax_backward_kernel1<<<blocks, block_size>>>(input, out_grad, output, sizes, N);
}

void softmax_backward_cuda2(float *input, float *out_grad, float *output, const int *dims, int n_dims, int dim, int block_size)
{
    DimSizes sizes = partition_tensor(dims, n_dims, dim);
    // one block per row that was softmaxed
    int blocks = sizes.outer * sizes.inner; // number of rows
    // block_size must be multiple of 32
    size_t shared_mem_size = (block_size / 32) * sizeof(float);
    softmax_backward_kernel2<<<blocks, block_size, shared_mem_size>>>(input, out_grad, output, sizes);
}

void softmax_backward_cuda(int kernel_num, float *input, float *out_grad, float *output,
                           const int *dims, int n_dims, int dim, int block_size)
{
    switch (kernel_num)
    {
    case 1:
        softmax_backward_cuda1(input, out_grad, output, dims, n_dims, dim, block_size);
        break;
    case 2:
        softmax_backward_cuda2(input, out_grad, output, dims, n_dims, dim, block_size);
        break;
    default:
        printf("Invalid kernel number\n");
        exit(1);
    }
}

// ---------------------------------------------------------------------------------------
int main(int argc, const char **argv)
{
    setup_main();

    const int dims[] = {125, 81, 384};
    const int n_dims = sizeof(dims) / sizeof(dims[0]);

    // which kernel version to use
    int version = 1;
    if (argc > 1)
    {
        version = atoi(argv[1]);
    }

    // which dimension to softmax
    int dim = 1;
    if (argc > 2)
    {
        dim = atoi(argv[2]);
    }
    if (dim <= -n_dims - 1 || dim >= n_dims)
    {
        exit(1);
    }

    // create host tensors
    int numel = 1;
    for (int i = 0; i < n_dims; i++)
    {
        numel *= dims[i];
    }
    float *h_input = make_random_float(numel);
    float *h_grad = make_random_float(numel);
    float *h_output = make_zeros_float(numel);

    // create cuda tensors
    float *d_input, *d_grad, *d_output;
    cudaCheck(hipMalloc(&d_input, numel * sizeof(float)));
    cudaCheck(hipMalloc(&d_grad, numel * sizeof(float)));
    cudaCheck(hipMalloc(&d_output, numel * sizeof(float)));
    cudaCheck(hipMemcpy(d_input, h_input, numel * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_grad, h_grad, numel * sizeof(float), hipMemcpyHostToDevice));

    // cpu reference
    softmax_backward_cpu(h_input, h_grad, h_output, dims, n_dims, dim);

    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        softmax_backward_cuda(version, d_input, d_grad, d_output, dims, n_dims, dim, block_size);
        float tol = 1e-1f;
        float rtol = 1e-1f;
        validate_result(d_output, h_output, "output", numel, tol, rtol);
    }

    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];
        int repeat_times = 1000;
        float elapsed_time = benchmark_gpu_kernel(repeat_times, softmax_backward_cuda, version,
                                                  d_input, d_grad, d_output, dims, n_dims, dim, block_size);

        printf("block_size %4d | time %.4f ms\n", block_size, elapsed_time);
    }

    printf("\nBenchmarking CPU version.");
    int repeat_times = 10;
    namespace chr = std::chrono;
    chr::steady_clock::time_point begin = chr::steady_clock::now();
    for (int i = 0; i < repeat_times; i++)
    {
        softmax_backward_cpu(h_input, h_grad, h_output, dims, n_dims, dim);
    }
    chr::steady_clock::time_point end = chr::steady_clock::now();
    auto elapsed_time_us = chr::duration_cast<chr::microseconds>(end - begin).count();
    float average_time_ms = ((float)elapsed_time_us / (float)repeat_times) / 1000.f;
    printf(" %.4f ms\n ", average_time_ms);

    // cleanup memory
    free(h_input);
    free(h_grad);
    free(h_output);

    cudaCheck(hipFree(d_input));
    cudaCheck(hipFree(d_grad));
    cudaCheck(hipFree(d_output));
}
