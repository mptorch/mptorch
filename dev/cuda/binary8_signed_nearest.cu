#include "hip/hip_runtime.h"


#include <hip/hip_runtime.h>
#include "common.h"

enum class SaturateMode {
    SATURATE,
    NO_OVERFLOW,
    OVERFLOWS
};

// --------------------------------------------------------------------------------------
// I/O pairs to sanity check the CPU reference code (E5M2 without subnormals, RNE)
uint32_t test_inputs[] = {
    0b00111000000000000000000000000000,
    0b00110111000000000000000000000000,
    0b00110110100000000000000000000000,
    0b00110110100000000000000000000001,
    0b01000111010000000000000000000000,
    0b01000111011000000000000000000000,
    0b00110110000000000000000000000000};
uint32_t test_outputs[] = {
    0b00111000000000000000000000000000,
    0b00110111000000000000000000000000,
    0b00000000000000000000000000000000,
    0b00110111000000000000000000000000,
    0b01000111010000000000000000000000,
    0b01000111010000000000000000000000,
    0b00000000000000000000000000000000};


uint32_t round_bitwise_nearest_cpu(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}

uint32_t binary8_clip_exponent_cpu(int exp_bits, int man_bits, uint32_t old_num, uint32_t quantized_num, SaturateMode saturation_mode, bool subnormal) {

  if (quantized_num == 0){
    return quantized_num;
  }
  
  uint32_t man_val = quantized_num & 0x7FFFFF;
  uint32_t old_sign = old_num >> 31 << 31;
  uint32_t max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  
  int spec_exp = (man_bits == 0) ? 1 : 0; // if P = 1
  int special_p1 = 0;
  
  if(exp_bits == 8 && saturation_mode != SaturateMode::NO_OVERFLOW){ // unsigned and p=1
      special_p1 = 1; // 0 bit of mantissa so the max value 0xfd = max_exp - 1 | mantissa = 0
  }else if (exp_bits == 7 &&  man_bits == 1 && saturation_mode != SaturateMode::NO_OVERFLOW){ // unsigned and p=2 
      special_p1 = 1;
  }else if(exp_bits + man_bits == 8){ // unsigned
      max_man = ((1u << man_bits) - 3u) << (23 - man_bits); // 2+ bit of mantissa so the max value 0xfd = mACax_exp | max_mantissa - 1 
  }

  // Special because in unsigned we want our min to be 1 less because the space is taken by the Nan
  int quantized_exponent_store = (quantized_num << 1 >> 24);
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127 - special_p1; 
  int min_exponent_store = -((1 << (exp_bits - 1)) - 1) + 127 + spec_exp;

  if (saturation_mode == SaturateMode::NO_OVERFLOW) { // Saturate to max without infinity
    max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  }

  if (quantized_exponent_store > max_exponent_store || ((quantized_exponent_store == max_exponent_store) && (man_val > max_man))) 
  {
    if (saturation_mode == SaturateMode::OVERFLOWS){ // Overflow to infinity
        return quantized_num = old_sign | 0x7F800000; // INF
    } 
    return quantized_num = old_sign | ((uint32_t)max_exponent_store << 23) | max_man;
  }
  if (quantized_exponent_store < min_exponent_store) {
    if (subnormal) {
        int subnormal_shift = min_exponent_store - quantized_exponent_store;
        int min_subnormals_exp = min_exponent_store - man_bits;
        uint32_t min_num = ((uint32_t)min_subnormals_exp << 23);
        uint32_t middle_num = ((uint32_t)(min_subnormals_exp - 1) << 23);
      if (subnormal_shift <= man_bits) {
        // quantized_num stays the same in this case
      } else if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    } 
    if(!subnormal) {
      uint32_t min_num = (uint32_t)min_exponent_store << 23;
      uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23);
      if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    }
  }
  return quantized_num;
}

float cast_binary8_signed_nearest_cpu(float origin_float, int P, SaturateMode saturation_mode, bool subnormals) {

    int exp_bits = 8-P;
    int man_bits = P-1;    
    int subnormal_shift = 0;
    uint32_t uval32, uval8;
    float fval8;

    uval32 = FLOAT_TO_BITS(&origin_float);

    int exp_val = (uval32 << 1 >> 24) - 127;
    uint32_t man_val = uval32 & 0x7FFFFF;

    if (exp_val == 128 && !(saturation_mode == SaturateMode::NO_OVERFLOW && man_val == 0)) {             // inf/Nan case
        return origin_float;
    }

    if (subnormals){
        int spec_exp = (P == 1) ? 1 : 0;
        int max_exp = (1 << (exp_bits -1)) - 1;    // minimal and maximal exponent value in binary8
        int min_exp = spec_exp - max_exp;

        if(((min_exp - exp_val) <= man_bits) && (exp_val < min_exp) && (subnormals)){ 
            subnormal_shift = min_exp - exp_val;
        }
    }

    uval8 = round_bitwise_nearest_cpu(uval32, man_bits - subnormal_shift);
    uval8 = binary8_clip_exponent_cpu(exp_bits, man_bits, uval32, uval8, saturation_mode, subnormals);
    fval8 = BITS_TO_FLOAT(&uval8);

    return fval8;
}

void binary8_signed_nearest_cpu(float *o, float *a, int N, int P, SaturateMode saturation_mode, bool subnormals) {
  for (int i = 0; i < N; ++i){
    o[i] = cast_binary8_signed_nearest_cpu(a[i], P, saturation_mode, subnormals);
  }
}


// ---------------------------------------------------------------------------------------
// GPU kernels
__device__ __forceinline__ uint32_t
round_bitwise_nearest(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}

__device__ __forceinline__ uint32_t
binary8_clip_exponent(int exp_bits, int man_bits, uint32_t old_num, uint32_t quantized_num, SaturateMode saturation_mode, bool subnormal) {

  if (quantized_num == 0){
    return quantized_num;
  }
  
  uint32_t man_val = quantized_num & 0x7FFFFF;
  uint32_t old_sign = old_num >> 31 << 31;
  uint32_t max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  
  int spec_exp = (man_bits == 0) ? 1 : 0; // if P = 1
  int special_p1 = 0;
  
  if(exp_bits == 8 && saturation_mode != SaturateMode::NO_OVERFLOW){ // unsigned and p=1
      special_p1 = 1; // 0 bit of mantissa so the max value 0xfd = max_exp - 1 | mantissa = 0
  }else if (exp_bits == 7 &&  man_bits == 1 && saturation_mode != SaturateMode::NO_OVERFLOW){ // unsigned and p=2 
      special_p1 = 1;
  }else if(exp_bits + man_bits == 8){ // unsigned
      max_man = ((1u << man_bits) - 3u) << (23 - man_bits); // 2+ bit of mantissa so the max value 0xfd = mACax_exp | max_mantissa - 1 
  }

  // Special because in unsigned we want our min to be 1 less because the space is taken by the Nan
  int quantized_exponent_store = (quantized_num << 1 >> 24);
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127 - special_p1; 
  int min_exponent_store = -((1 << (exp_bits - 1)) - 1) + 127 + spec_exp;

  if (saturation_mode == SaturateMode::NO_OVERFLOW) { // Saturate to max without infinity
    max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  }

  if (quantized_exponent_store > max_exponent_store || ((quantized_exponent_store == max_exponent_store) && (man_val > max_man))) 
  {
    if (saturation_mode == SaturateMode::OVERFLOWS){ // Overflow to infinity
        return quantized_num = old_sign | 0x7F800000; // INF
    } 
    return quantized_num = old_sign | ((uint32_t)max_exponent_store << 23) | max_man;
  }
  if (quantized_exponent_store < min_exponent_store) {
    if (subnormal) {
        int subnormal_shift = min_exponent_store - quantized_exponent_store;
        int min_subnormals_exp = min_exponent_store - man_bits;
        uint32_t min_num = ((uint32_t)min_subnormals_exp << 23);
        uint32_t middle_num = ((uint32_t)(min_subnormals_exp - 1) << 23);
      if (subnormal_shift <= man_bits) {
        // quantized_num stays the same in this case
      } else if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    } 
    if(!subnormal) {
      uint32_t min_num = (uint32_t)min_exponent_store << 23;
      uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23);
      if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    }
  }
  return quantized_num;
}

__device__ float cast_binary8_signed_nearest(float origin_float, int P, SaturateMode saturation_mode, bool subnormals) {

    int exp_bits = 8-P;
    int man_bits = P-1;    
    int subnormal_shift = 0;
    uint32_t uval32, uval8;
    float fval8;

    uval32 = FLOAT_TO_BITS(&origin_float);

    int exp_val = (uval32 << 1 >> 24) - 127;
    uint32_t man_val = uval32 & 0x7FFFFF;

    if (exp_val == 128 && !(saturation_mode == SaturateMode::NO_OVERFLOW && man_val == 0)) {             // inf/Nan case
        return origin_float;
    }

    if (subnormals){
        int spec_exp = (P == 1) ? 1 : 0;
        int max_exp = (1 << (exp_bits -1)) - 1;    // minimal and maximal exponent value in binary8
        int min_exp = spec_exp - max_exp;

        if(((min_exp - exp_val) <= man_bits) && (exp_val < min_exp) && (subnormals)){ 
            subnormal_shift = min_exp - exp_val;
        }
    }

    uval8 = round_bitwise_nearest(uval32, man_bits - subnormal_shift);
    uval8 = binary8_clip_exponent(exp_bits, man_bits, uval32, uval8, saturation_mode, subnormals);
    fval8 = BITS_TO_FLOAT(&uval8);

    return fval8;
}

__device__ __forceinline__ uint32_t clip_subnormal_range_exponent(int exp_bits, int man_bits,
            uint32_t old_num, uint32_t quantized_num)
{
    if (quantized_num == 0)
        return quantized_num;

    int quantized_exp_store = quantized_num << 1 >> 24;
    int min_exp_store = -((1 << (exp_bits - 1)) - 1) - man_bits + 127 + (man_bits == 0);

    uint32_t old_sign = old_num & 0x80000000;
    if (quantized_exp_store < min_exp_store)
    {
        int offset = (quantized_exp_store == (min_exp_store - 1));
        quantized_num += offset * (1u << 23);
        quantized_num |= old_sign;
        quantized_num *= offset;
    }
    return quantized_num;
}

__device__ __forceinline__ uint32_t clip_normal_range_exponent(int exp_bits, int man_bits,
            uint32_t old_num, uint32_t quantized_num, SaturateMode smode)
{
    if (quantized_num == 0)
        return quantized_num;

    int old_exponent_store = old_num << 1 >> 24;
    int max_exponent_store = ((1 << (exp_bits - 1)) - 1) + 127;
    int min_exponent_store = -((1 << (exp_bits -1)) - 1) + 127 + (man_bits == 0);

    uint32_t old_sign = old_num & 0x80000000;
    uint32_t max_man = 0x007FFFFF >> (24 - man_bits - (int)smode) << (24 - man_bits - (int)smode);
    uint32_t max_num = ((uint32_t)max_exponent_store << 23) | max_man;
    // saturate or overflow
    if ((quantized_num >= max_num) && (old_exponent_store >= max_exponent_store))
    {
        if (smode != SaturateMode::OVERFLOWS)
        {
            quantized_num = old_sign | max_num;
        } else {
            quantized_num = 0x7FFFFFFF;
            quantized_num = old_sign | quantized_num;
        }
    } // underflow or round to smallest nonzero normal value
    else if (old_exponent_store < min_exponent_store) 
    {
        uint32_t offset = (old_exponent_store == (min_exponent_store - 1)) && ((old_num << 9 >> 9) > 0);
        quantized_num = offset * (min_exponent_store << 23);
        quantized_num |= old_sign;        
    }
    return quantized_num;
}

__device__ float cast_binary8_nearest(float origin_float, int man_bits, int exp_bits,
                            bool subnormals = true, SaturateMode smode = SaturateMode::OVERFLOWS)
{
    uint32_t target, quantize_bits;
    target = FLOAT_TO_BITS(&origin_float);
    float quantized;

    int target_exp = (target << 1 >> 1 >> 23) - 127;
    int min_exp = -((1 << (exp_bits - 1)) - 1) + (man_bits == 0);
    bool is_subnormal = (target_exp < min_exp);
    bool noquantize = (man_bits >= 23);

    if (noquantize)
    {
        quantized = origin_float;
    }
    else
    {
        // handle subnormal inputs
        if (is_subnormal && subnormals)
        {
            int exp_diff = man_bits - (min_exp - target_exp);
            int not_uflow = exp_diff > -1 || ((exp_diff == -1) && ((target << 9) > 0));
            quantize_bits = not_uflow * round_bitwise_nearest(target, exp_diff);
            quantize_bits =
                clip_subnormal_range_exponent(exp_bits, man_bits, target, quantize_bits);
            quantized = BITS_TO_FLOAT(&quantize_bits);
        }
        // handle NaN/inf inputs
        else if (target_exp == 128)
        {
            quantized = origin_float;
        }
        // normal value range or overflow
        else
        {
            quantize_bits = round_bitwise_nearest(target, man_bits);
            quantize_bits = clip_normal_range_exponent(exp_bits, man_bits, 
                                                target, quantize_bits, smode);
            quantized = BITS_TO_FLOAT(&quantize_bits);
        }
    }

    return quantized;
}

__global__ void binary8_signed_nearest_gpu1(float *o, float *__restrict__ a, int N, int P, SaturateMode saturation_mode, bool subnormals)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        o[index] = cast_binary8_signed_nearest(a[index], P, saturation_mode, subnormals);
    }
}

__global__ void binary8_signed_nearest_gpu2(float *o, float *__restrict__ a, int N, int P, SaturateMode saturation_mode, bool subnormals)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < N)
    {
        o[index] = cast_binary8_nearest(a[index], P - 1, 8 - P, subnormals, saturation_mode);
    }
}

// ---------------------------------------------------------------------------------------
// Kernel launchers
void binary8_signed_nearest1(float *o, float *a, int N, int P, const int block_size, SaturateMode saturation_mode, bool subnormals)
{
    const int grid_size = ceil_div(N, block_size);
    binary8_signed_nearest_gpu1<<<grid_size, block_size>>>(o, a, N, P, saturation_mode, subnormals);
    cudaCheck(hipGetLastError());
}

void binary8_signed_nearest2(float *o, float *a, int N, int P, const int block_size, SaturateMode saturation_mode, bool subnormals)
{
    const int grid_size = ceil_div(N, block_size);
    binary8_signed_nearest_gpu2<<<grid_size, block_size>>>(o, a, N, P, saturation_mode, subnormals);
    cudaCheck(hipGetLastError());
}

// kernel version dispatch
void binary8_signed_nearest(int kernel_num, float *o, float *a, int N, int P, const int block_size, SaturateMode saturation_mode, bool subnormals)
{
    switch (kernel_num)
    {
    case 1:
        binary8_signed_nearest1(o, a, N, P, block_size, saturation_mode, subnormals);
        break;
    case 2:
        binary8_signed_nearest2(o, a, N, P, block_size, saturation_mode, subnormals);
        break;
    default:
        printf("Invalid kernel number\n");
        exit(EXIT_FAILURE);
    }
}

// ---------------------------------------------------------------------------------------
int main(int argc, const char **argv)
{
    setup_main();

    // read the kernel number from the command line
    int kernel_num = 1;
    if (argc > 1)
    {
        kernel_num = atoi(argv[1]);
    }

    SaturateMode saturation_mode = SaturateMode::SATURATE;
    bool subnormals = true;


    int N = 1 << 24;
    int P = 3;

    float *x = make_random_float(N);
    float *y = (float *)malloc(N * sizeof(float));

    printf("Using kernel %d\n", kernel_num);

    // compute reference CPU solution
    binary8_signed_nearest_cpu(y, x, N, P, saturation_mode, subnormals);

    // move data to the GPU
    float *d_x, *d_y;
    cudaCheck(hipMalloc(&d_x, N * sizeof(float)));
    cudaCheck(hipMalloc(&d_y, N * sizeof(float)));

    cudaCheck(hipMemcpy(d_x, x, N * sizeof(float), hipMemcpyHostToDevice));

    // time the kernel at different block sizes
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];
        printf("Checking block size %d.\n", block_size);
        binary8_signed_nearest(kernel_num, d_y, d_x, N, P, block_size, saturation_mode, subnormals);

        float tol = 0.0f;
        validate_result(d_y, y, "y", N, tol);
    }

    printf("All results match. Starting benchmarks.\n\n");

    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); ++j)
    {
        int block_size = block_sizes[j];

        int repeat_times = 1000;

        float elapsed_time = benchmark_kernel(repeat_times, binary8_signed_nearest, 
                kernel_num, d_y, d_x, N, P, block_size, saturation_mode, subnormals);

        // estimate memory bandwidth achieved
        // for each output element, we do 1 read and 1 write, 4 bytes each
        long memory_ops = N * 2 * (int)sizeof(float);
        float memory_bandwidth = memory_ops / elapsed_time / 1e6;

        printf("block_size %4d | time %.4f ms | bandwidth %.2f GB/s\n", block_size, elapsed_time, memory_bandwidth);
    }

    // free memory
    free(x);
    free(y);

    cudaCheck(hipFree(d_x));
    cudaCheck(hipFree(d_y));

    return 0;
}