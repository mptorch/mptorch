#include "hip/hip_runtime.h"
#include "quant.h"
#include "quant_kernel.h"
#include "cublas_helper.h"
#include <ATen/ATen.h>
#include <climits>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <cmath>
#include <cstdint>
#include <tuple>
#include <stdexcept>
#include <cassert>

using namespace at;

Tensor get_max_entry(Tensor a, int dim)
{
  Tensor max_entry;
  if (dim == -1)
  {
    max_entry = at::max(at::abs(a)).expand_as(a).contiguous();
  }
  else if (dim == 0)
  {
    Tensor input_view = a.view({a.size(0), -1});
    max_entry = std::get<0>(input_view.abs().max(1, true))
                    .expand_as(input_view)
                    .view_as(a)
                    .contiguous();
  }
  else
  {
    Tensor input_transpose = a.transpose(0, dim);
    Tensor input_view =
        input_transpose.contiguous().view({input_transpose.size(0), -1});
    Tensor max_transpose = std::get<0>(input_view.abs().max(1, true))
                               .expand_as(input_view)
                               .view_as(input_transpose);
    max_entry = max_transpose.transpose(dim, 0).contiguous();
  }
  return max_entry;
}

Tensor block_quantize_stochastic_cuda(Tensor a, int wl, int dim)
{
  auto o = at::zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int64_t size = a.numel();

  Tensor max_entry = get_max_entry(a, dim);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_ints.data_ptr<int>(), o.data_ptr<float>(), size,
      max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor block_quantize_nearest_cuda(Tensor a, int wl, int dim)
{
  auto o = at::zeros_like(a);
  int64_t size = a.numel();

  Tensor max_entry = get_max_entry(a, dim);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size,
      max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor block_quantize_sim_stochastic_cuda(Tensor a, int wl)
{
  auto o = at::zeros_like(a);
  auto rand_probs = rand_like(a);
  int64_t size = a.numel();

  Tensor max_entry = at::max(at::abs(a));
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_sim_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_probs.data_ptr<float>(), o.data_ptr<float>(),
      size, max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor block_quantize_sim_nearest_cuda(Tensor a, int wl)
{
  auto o = at::zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int64_t size = a.numel();

  Tensor max_entry = at::max(at::abs(a));
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_sim_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size,
      max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor float_quantize_stochastic_cuda(Tensor a, int man_bits, int exp_bits,
                                      bool subnormals, bool saturate)
{
  // use external random number right now
  auto o = zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int size = a.numel();
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  float_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_ints.data_ptr<int>(), o.data_ptr<float>(), size,
      man_bits, exp_bits, subnormals, saturate);
  return o;
}

Tensor float_quantize_nearest_cuda(Tensor a, int man_bits, int exp_bits,
                                   bool subnormals, bool saturate)
{
  auto o = zeros_like(a);
  int size = a.numel();
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  float_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size, man_bits, exp_bits,
      subnormals, saturate);
  return o;
}

Tensor superfp_quantize_nearest_cuda(Tensor a, int man_bits, int exp_bits,
                                    bool saturate) 
{
  auto o = zeros_like(a);
  int size = a.numel();
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  superfp_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size, man_bits, exp_bits, saturate);
  return o;

}

Tensor p3109_quantize_nearest_cuda(Tensor a, int P, bool is_signed, bool subnormals)
{
  auto o = zeros_like(a);
  // TODO
  return o;
}

Tensor p3109_quantize_stochastic_cuda(Tensor a, int P, int prng_bits, bool is_signed, bool subnormals)
{
  auto o = zeros_like(a);
  // TODO
  return o;
}

void fixed_min_max(int wl, int fl, bool symmetric, float *t_min, float *t_max)
{
  int sigma = -fl;
  *t_min = -ldexp(1.0, wl - fl - 1);
  *t_max = -*t_min - ldexp(1.0, sigma);
  if (symmetric)
    *t_min = *t_min + ldexp(1.0, sigma);
}

Tensor fixed_point_quantize_stochastic_cuda(Tensor a, int wl, int fl,
                                            bool use_clamp, bool symmetric)
{
  // use external random number right now
  auto o = at::zeros_like(a);
  auto rand_probs = rand_like(a);
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_probs.data_ptr<float>(), o.data_ptr<float>(),
      size, sigma, use_clamp, t_min, t_max);
  return o;
}

Tensor fixed_point_quantize_nearest_cuda(Tensor a, int wl, int fl,
                                         bool use_clamp, bool symmetric)
{
  // use external random number right now
  auto o = at::zeros_like(a);
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size, sigma, use_clamp, t_min,
      t_max);
  return o;
}

std::tuple<Tensor, Tensor>
fixed_point_quantize_stochastic_mask_cuda(Tensor a, int wl, int fl,
                                          bool symmetric)
{
  // use external random number right now
  auto o = zeros_like(a);
  auto rand_probs = rand_like(a);
  auto m = zeros_like(a, a.options().dtype(kByte));
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_mask_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_probs.data_ptr<float>(), o.data_ptr<float>(),
      m.data_ptr<uint8_t>(), size, sigma, t_min, t_max);
  return std::make_tuple(o, m);
}

std::tuple<Tensor, Tensor>
fixed_point_quantize_nearest_mask_cuda(Tensor a, int wl, int fl,
                                       bool symmetric)
{
  // use external random number right now
  auto o = at::zeros_like(a);
  auto m = zeros_like(a, a.options().dtype(kByte));
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_mask_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), m.data_ptr<uint8_t>(), size,
      sigma, t_min, t_max);
  return std::make_tuple(o, m);
}

void float_quantize_nearest_mm_cuda(Tensor a, Tensor b, Tensor c, int M, int N,
                                    int K, int man_add, int exp_add,
                                    int man_mul, int exp_mul, bool subnormals,
                                    bool saturate)
{
  mm_fp_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                M, K, N, man_add, exp_add, man_mul, exp_mul, subnormals,
                saturate);
  return;
}

void float_quantize_nearest_mm_fma_cuda(Tensor a, Tensor b, Tensor c, int M,
                                        int N, int K, int man_fma, int exp_fma,
                                        bool subnormals, bool saturate)
{
  mm_fp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                    c.data_ptr<float>(), M, K, N, man_fma, exp_fma, subnormals,
                    saturate);
  return;
}

void float_quantize_nearest_bmm_cuda(Tensor a, Tensor b, Tensor c, int M, int N,
                                     int K, int man_add, int exp_add,
                                     int man_mul, int exp_mul, bool subnormals,
                                     bool saturate)
{
  if (a.sizes().size() > 2)
    bmm_fp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                   c.data_ptr<float>(), a.sizes()[0], M, K, N, man_add, exp_add,
                   man_mul, man_add, subnormals, saturate);
  else
    bmm_fp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                   c.data_ptr<float>(), 1, M, K, N, man_add, exp_add, man_mul,
                   man_add, subnormals, saturate);
  return;
}

void float_quantize_nearest_bmm_fma_cuda(Tensor a, Tensor b, Tensor c, int M,
                                         int N, int K, int man_fma, int exp_fma,
                                         bool subnormals, bool saturate)
{
  if (a.sizes().size() > 2)
    bmm_fp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), a.sizes()[0], M, K, N, man_fma,
                       exp_fma, subnormals, saturate);
  else
    bmm_fp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), 1, M, K, N, man_fma, exp_fma,
                       subnormals, saturate);
  return;
}

void superfp_quantize_nearest_mm_cuda(Tensor a, Tensor b, Tensor c, int M, int N,
                                    int K, int man_add, int exp_add,
                                    int man_mul, int exp_mul,
                                    bool saturate) 
{
  mm_superfp_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                M, K, N, man_add, exp_add, man_mul, exp_mul, saturate);  
}

void superfp_quantize_nearest_mm_fma_cuda(Tensor a, Tensor b, Tensor c, int M,
                                        int N, int K, int man_fma, int exp_fma,
                                        bool saturate)
{
  mm_superfp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                    c.data_ptr<float>(), M, K, N, man_fma, exp_fma,
                    saturate);
  return;
}

void superfp_quantize_nearest_bmm_cuda(Tensor a, Tensor b, Tensor c, int M, int N,
                                     int K, int man_add, int exp_add,
                                     int man_mul, int exp_mul, bool saturate)
{
  if (a.sizes().size() > 2)
    bmm_superfp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                   c.data_ptr<float>(), a.sizes()[0], M, K, N, man_add, exp_add,
                   man_mul, man_add, saturate);
  else
    bmm_superfp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                   c.data_ptr<float>(), 1, M, K, N, man_add, exp_add, man_mul,
                   man_add, saturate);
  return;
}

void superfp_quantize_nearest_bmm_fma_cuda(Tensor a, Tensor b, Tensor c, int M,
                                         int N, int K, int man_fma, int exp_fma, bool saturate)
{
  if (a.sizes().size() > 2)
    bmm_superfp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), a.sizes()[0], M, K, N, man_fma,
                       exp_fma, saturate);
  else
    bmm_superfp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), 1, M, K, N, man_fma, exp_fma, saturate);
  return;
}

void float_quantize_stochastic_mm_cuda(Tensor a, Tensor b, Tensor c, int M,
                                       int N, int K, int man_add, int exp_add,
                                       int man_mul, int exp_mul,
                                       bool subnormals, bool saturate)
{
  mm_fp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                   c.data_ptr<float>(), M, K, N, man_add, exp_add, man_mul,
                   exp_mul, subnormals, saturate);
  return;
}

void float_quantize_stochastic_mm_fma_cuda(Tensor a, Tensor b, Tensor c, int M,
                                           int N, int K, int man_fma,
                                           int exp_fma, bool subnormals,
                                           bool saturate)
{
  mm_fp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), M, K, N, man_fma, exp_fma,
                       subnormals, saturate);
  return;
}

void float_quantize_stochastic_bmm_cuda(Tensor a, Tensor b, Tensor c, int M,
                                        int N, int K, int man_add, int exp_add,
                                        int man_mul, int exp_mul,
                                        bool subnormals, bool saturate)
{
  if (a.sizes().size() > 2)
    bmm_fp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                      c.data_ptr<float>(), a.sizes()[0], M, K, N, man_add,
                      exp_add, man_mul, man_add, subnormals, saturate);
  else
    bmm_fp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                      c.data_ptr<float>(), 1, M, K, N, man_add, exp_add,
                      man_mul, man_add, subnormals, saturate);
}

void float_quantize_stochastic_bmm_fma_cuda(Tensor a, Tensor b, Tensor c, int M,
                                            int N, int K, int man_fma,
                                            int exp_fma, bool subnormals,
                                            bool saturate)
{

  if (a.sizes().size() > 2)
    bmm_fp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                          c.data_ptr<float>(), a.sizes()[0], M, K, N, man_fma,
                          exp_fma, subnormals, saturate);
  else
    bmm_fp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                          c.data_ptr<float>(), 1, M, K, N, man_fma, exp_fma,
                          subnormals, saturate);
}

void fixed_point_quantize_nearest_mm_cuda(Tensor a, Tensor b, Tensor c, int M,
                                          int N, int K, int wl_add, int fl_add,
                                          int wl_mul, int fl_mul,
                                          bool symmetric)
{
  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  mm_fxp_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                 M, K, N, sigma_add, t_min_add, t_max_add, sigma_mul, t_min_mul,
                 t_max_mul);
  return;
}

void fixed_point_quantize_nearest_bmm_cuda(Tensor a, Tensor b, Tensor c, int M,
                                           int N, int K, int wl_add, int fl_add,
                                           int wl_mul, int fl_mul,
                                           bool symmetric)
{

  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  if (a.sizes().size() > 2)
    bmm_fxp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                    c.data_ptr<float>(), a.sizes()[0], M, K, N, sigma_add,
                    t_min_add, t_max_add, sigma_mul, t_min_mul, t_max_mul);
  else
    bmm_fxp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                    c.data_ptr<float>(), 1, M, K, N, sigma_add, t_min_add,
                    t_max_add, sigma_mul, t_min_mul, t_max_mul);
  return;
}

void fixed_point_quantize_nearest_mm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                              int M, int N, int K, int wl_fma,
                                              int fl_fma, bool symmetric)
{
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  mm_fxp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                     c.data_ptr<float>(), M, K, N, sigma_fma, t_min_fma,
                     t_max_fma);
  return;
}

void fixed_point_quantize_nearest_bmm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                               int M, int N, int K, int wl_fma,
                                               int fl_fma, bool symmetric)
{
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  if (a.sizes().size() > 2)
    bmm_fxp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                        c.data_ptr<float>(), a.sizes()[0], M, K, N, sigma_fma,
                        t_min_fma, t_max_fma);
  else
    bmm_fxp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                        c.data_ptr<float>(), 1, M, K, N, sigma_fma, t_min_fma,
                        t_max_fma);
  return;
}

void fixed_point_quantize_stochastic_mm_cuda(Tensor a, Tensor b, Tensor c,
                                             int M, int N, int K, int wl_add,
                                             int fl_add, int wl_mul, int fl_mul,
                                             bool symmetric)
{
  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  mm_fxp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                    c.data_ptr<float>(), M, K, N, sigma_add, t_min_add,
                    t_max_add, sigma_mul, t_min_mul, t_max_mul);
  return;
}

void fixed_point_quantize_stochastic_bmm_cuda(Tensor a, Tensor b, Tensor c,
                                              int M, int N, int K, int wl_add,
                                              int fl_add, int wl_mul,
                                              int fl_mul, bool symmetric)
{
  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  if (a.sizes().size() > 2)
    bmm_fxp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), a.sizes()[0], M, K, N, sigma_add,
                       t_min_add, t_max_add, sigma_mul, t_min_mul, t_max_mul);
  else
    bmm_fxp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), 1, M, K, N, sigma_add, t_min_add,
                       t_max_add, sigma_mul, t_min_mul, t_max_mul);
  return;
}

void fixed_point_quantize_stochastic_mm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                                 int M, int N, int K,
                                                 int wl_fma, int fl_fma,
                                                 bool symmetric)
{
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  mm_fxp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                        c.data_ptr<float>(), M, K, N, sigma_fma, t_min_fma,
                        t_max_fma);
  return;
}

void fixed_point_quantize_stochastic_bmm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                                  int M, int N, int K,
                                                  int wl_fma, int fl_fma,
                                                  bool symmetric)
{
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  if (a.sizes().size() > 2)
    bmm_fxp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                           c.data_ptr<float>(), a.sizes()[0], M, K, N,
                           sigma_fma, t_min_fma, t_max_fma);
  else
    bmm_fxp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                           c.data_ptr<float>(), 1, M, K, N, sigma_fma,
                           t_min_fma, t_max_fma);
  return;
}



void floating_point_mm_cublas(Tensor a, Tensor b, Tensor c, int M, int N, int K,
                              cublas_matrix_dt AB_type, cublas_matrix_dt C_type,
                              cublas_compute_dt compute_type, bool pedantic)
{
  // Tensors a, b, and c are assumed to have the right datatype and transposed.
  cublas_config config;
  get_cublas_configuration(AB_type, C_type, compute_type, pedantic, config);

  hipblasMath_t math = pedantic ? HIPBLAS_PEDANTIC_MATH : HIPBLAS_DEFAULT_MATH;
  math = (hipblasMath_t)(math | HIPBLAS_MATH_DISALLOW_REDUCED_PRECISION_REDUCTION);
  hipblasSetMathMode(get_cublas_handle(), math);

  // special case for scalar types: https://docs.nvidia.com/cuda/cublas/index.html#cublasgemmex
  switch (config.scalar) {
  case HIP_R_16F:
    {
    half alpha = __float2half(1.f);
    half beta = __float2half(0.f);
    hipblasGemmEx(get_cublas_handle(),
                  HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha,
                  a.data_ptr(), config.matrix_a, M,
                  b.data_ptr(), config.matrix_b, K, &beta,
                  c.data_ptr(), config.matrix_c, M,
                  config.compute,
                  HIPBLAS_GEMM_DEFAULT);
    }
    break;
  default:
    {
    float alpha = 1.f;
    float beta = 0.f;
    hipblasGemmEx(get_cublas_handle(),
                  HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha,
                  a.data_ptr(), config.matrix_a, M,
                  b.data_ptr(), config.matrix_b, K, &beta,
                  c.data_ptr(), config.matrix_c, M,
                  config.compute,
                  HIPBLAS_GEMM_DEFAULT);
    }
    break;
  }
}


void floating_point_bmm_cublas(Tensor a, Tensor b, Tensor c, int M, int N, int K,
                               cublas_matrix_dt AB_type, cublas_matrix_dt C_type,
                               cublas_compute_dt compute_type, bool pedantic)
{
  // Tensors a, b, and c are assumed to have the right datatype and transposed.
  cublas_config config;
  get_cublas_configuration(AB_type, C_type, compute_type, pedantic, config);

  hipblasMath_t math = pedantic ? HIPBLAS_PEDANTIC_MATH : HIPBLAS_DEFAULT_MATH;
  math = (hipblasMath_t)(math | HIPBLAS_MATH_DISALLOW_REDUCED_PRECISION_REDUCTION);
  hipblasSetMathMode(get_cublas_handle(), math);

  int B = a.sizes().size() > 2 ? a.size(0) : 1; // batch count

  // Allocate the array of pointers to each matrices  
  auto copy_ptrs = [B](void** arr, Tensor a, hipDataType t, int stride) {
    switch (t) {
    case HIP_R_32F:
      {
      float *p = a.data_ptr<float>();
      for (int i = 0; i < B; i++) {
        arr[i] = p + i * stride;
      }
      }
      break;
    case HIP_R_16F:
    case HIP_R_16BF:
      {
      at::Half *p = a.data_ptr<at::Half>();
      for (int i = 0; i < B; i++) {
        arr[i] = p + i * stride;
      }
      }
      break;
    default:
      throw std::invalid_argument("Invalid datatype.");
    }
  };
  void *a_array[B];
  void *b_array[B];
  void *c_array[B];
  copy_ptrs(a_array, a, config.matrix_a, M*K);
  copy_ptrs(b_array, b, config.matrix_b, K*N);
  copy_ptrs(c_array, c, config.matrix_c, M*N);

  // special case for scalar types: https://docs.nvidia.com/cuda/cublas/index.html#cublasgemmbatchedex
  // TODO: Fix, it crashes, memory alignement issues apparently
  switch (config.scalar) {
  case HIP_R_16F:
    // {
    // half alpha = __float2half(1.f);
    // half beta = __float2half(0.f);
    // hipblasGemmBatchedEx(get_cublas_handle(),
    //               HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha,
    //               a_array, config.matrix_a, M,
    //               b_array, config.matrix_b, K, &beta,
    //               c_array, config.matrix_c, M, B,
    //               config.compute,
    //               config.algo);
    // }
    break;
  default:
    // {
    // float alpha = 1.f;
    // float beta = 0.f;
    // hipblasGemmBatchedEx(get_cublas_handle(),
    //               HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K, &alpha,
    //               (void**)a_array, config.matrix_a, M,
    //               (void**)b_array, config.matrix_b, K, &beta,
    //               (void**)c_array, config.matrix_c, M, B,
    //               config.compute,
    //               config.algo);
    // }
    break;
  }
}