#include "hip/hip_runtime.h"
#include "quant_cuda.h"
#include "quant_kernel.h"
#include <ATen/ATen.h>
#include <climits>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdint.h>
#include <tuple>

using namespace at;

Tensor get_max_entry(Tensor a, int dim) {
  Tensor max_entry;
  if (dim == -1) {
    max_entry = at::max(at::abs(a)).expand_as(a).contiguous();
  } else if (dim == 0) {
    Tensor input_view = a.view({a.size(0), -1});
    max_entry = std::get<0>(input_view.abs().max(1, true))
                    .expand_as(input_view)
                    .view_as(a)
                    .contiguous();
  } else {
    Tensor input_transpose = a.transpose(0, dim);
    Tensor input_view =
        input_transpose.contiguous().view({input_transpose.size(0), -1});
    Tensor max_transpose = std::get<0>(input_view.abs().max(1, true))
                               .expand_as(input_view)
                               .view_as(input_transpose);
    max_entry = max_transpose.transpose(dim, 0).contiguous();
  }
  return max_entry;
}

Tensor block_quantize_stochastic_cuda(Tensor a, int wl, int dim) {
  auto o = at::zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int64_t size = a.numel();

  Tensor max_entry = get_max_entry(a, dim);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_ints.data_ptr<int>(), o.data_ptr<float>(), size,
      max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor block_quantize_nearest_cuda(Tensor a, int wl, int dim) {
  auto o = at::zeros_like(a);
  int64_t size = a.numel();

  Tensor max_entry = get_max_entry(a, dim);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size,
      max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor block_quantize_sim_stochastic_cuda(Tensor a, int wl) {
  auto o = at::zeros_like(a);
  auto rand_probs = rand_like(a);
  int64_t size = a.numel();

  Tensor max_entry = at::max(at::abs(a));
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_sim_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_probs.data_ptr<float>(), o.data_ptr<float>(),
      size, max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor block_quantize_sim_nearest_cuda(Tensor a, int wl) {
  auto o = at::zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int64_t size = a.numel();

  Tensor max_entry = at::max(at::abs(a));
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_sim_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size,
      max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor float_quantize_stochastic_cuda(Tensor a, int man_bits, int exp_bits,
                                      bool subnormals, bool saturate) {
  // use external random number right now
  auto o = zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int size = a.numel();
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  float_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_ints.data_ptr<int>(), o.data_ptr<float>(), size,
      man_bits, exp_bits, subnormals, saturate);
  return o;
}

Tensor float_quantize_nearest_cuda(Tensor a, int man_bits, int exp_bits,
                                   bool subnormals, bool saturate) {
  // use external random number right now
  auto o = zeros_like(a);
  int size = a.numel();
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  float_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size, man_bits, exp_bits,
      subnormals, saturate);
  return o;
}

void fixed_min_max(int wl, int fl, bool symmetric, float *t_min, float *t_max) {
  int sigma = -fl;
  *t_min = -ldexp(1.0, wl - fl - 1);
  *t_max = -*t_min - ldexp(1.0, sigma);
  if (symmetric)
    *t_min = *t_min + ldexp(1.0, sigma);
}

Tensor fixed_point_quantize_stochastic_cuda(Tensor a, int wl, int fl,
                                            bool use_clamp, bool symmetric) {
  // use external random number right now
  auto o = at::zeros_like(a);
  auto rand_probs = rand_like(a);
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_probs.data_ptr<float>(), o.data_ptr<float>(),
      size, sigma, use_clamp, t_min, t_max);
  return o;
}

Tensor fixed_point_quantize_nearest_cuda(Tensor a, int wl, int fl,
                                         bool use_clamp, bool symmetric) {
  // use external random number right now
  auto o = at::zeros_like(a);
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size, sigma, use_clamp, t_min,
      t_max);
  return o;
}

std::tuple<Tensor, Tensor>
fixed_point_quantize_stochastic_mask_cuda(Tensor a, int wl, int fl,
                                          bool symmetric) {
  // use external random number right now
  auto o = zeros_like(a);
  auto rand_probs = rand_like(a);
  auto m = zeros_like(a, a.options().dtype(kByte));
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_mask_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_probs.data_ptr<float>(), o.data_ptr<float>(),
      m.data_ptr<uint8_t>(), size, sigma, t_min, t_max);
  return std::make_tuple(o, m);
}

std::tuple<Tensor, Tensor>
fixed_point_quantize_nearest_mask_cuda(Tensor a, int wl, int fl,
                                       bool symmetric) {
  // use external random number right now
  auto o = at::zeros_like(a);
  auto m = zeros_like(a, a.options().dtype(kByte));
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_mask_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), m.data_ptr<uint8_t>(), size,
      sigma, t_min, t_max);
  return std::make_tuple(o, m);
}

void float_quantize_nearest_mm_cuda(Tensor a, Tensor b, Tensor c, int M, int N,
                                    int K, int man_add, int exp_add,
                                    int man_mul, int exp_mul, bool subnormals,
                                    bool saturate) {
  mm_fp_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                M, K, N, man_add, exp_add, man_mul, exp_mul, subnormals,
                saturate);
  return;
}

void float_quantize_nearest_mm_fma_cuda(Tensor a, Tensor b, Tensor c, int M,
                                        int N, int K, int man_fma, int exp_fma,
                                        bool subnormals, bool saturate) {
  mm_fp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                    c.data_ptr<float>(), M, K, N, man_fma, exp_fma, subnormals,
                    saturate);
  return;
}

void float_quantize_nearest_bmm_cuda(Tensor a, Tensor b, Tensor c, int M, int N,
                                     int K, int man_add, int exp_add,
                                     int man_mul, int exp_mul, bool subnormals,
                                     bool saturate) {
  if (a.sizes().size() > 2)
    bmm_fp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                   c.data_ptr<float>(), a.sizes()[0], M, K, N, man_add, exp_add,
                   man_mul, man_add, subnormals, saturate);
  else
    bmm_fp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                   c.data_ptr<float>(), 1, M, K, N, man_add, exp_add, man_mul,
                   man_add, subnormals, saturate);
  return;
}

void float_quantize_nearest_bmm_fma_cuda(Tensor a, Tensor b, Tensor c, int M,
                                         int N, int K, int man_fma, int exp_fma,
                                         bool subnormals, bool saturate) {
  if (a.sizes().size() > 2)
    bmm_fp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), a.sizes()[0], M, K, N, man_fma,
                       exp_fma, subnormals, saturate);
  else
    bmm_fp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), 1, M, K, N, man_fma, exp_fma,
                       subnormals, saturate);
  return;
}

void float_quantize_stochastic_mm_cuda(Tensor a, Tensor b, Tensor c, int M,
                                       int N, int K, int man_add, int exp_add,
                                       int man_mul, int exp_mul,
                                       bool subnormals, bool saturate) {
  mm_fp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                   c.data_ptr<float>(), M, K, N, man_add, exp_add, man_mul,
                   exp_mul, subnormals, saturate);
  return;
}

void float_quantize_stochastic_mm_fma_cuda(Tensor a, Tensor b, Tensor c, int M,
                                           int N, int K, int man_fma,
                                           int exp_fma, bool subnormals,
                                           bool saturate) {
  mm_fp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), M, K, N, man_fma, exp_fma,
                       subnormals, saturate);
  return;
}

void float_quantize_stochastic_bmm_cuda(Tensor a, Tensor b, Tensor c, int M,
                                        int N, int K, int man_add, int exp_add,
                                        int man_mul, int exp_mul,
                                        bool subnormals, bool saturate) {
  if (a.sizes().size() > 2)
    bmm_fp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                      c.data_ptr<float>(), a.sizes()[0], M, K, N, man_add,
                      exp_add, man_mul, man_add, subnormals, saturate);
  else
    bmm_fp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                      c.data_ptr<float>(), 1, M, K, N, man_add, exp_add,
                      man_mul, man_add, subnormals, saturate);
}

void float_quantize_stochastic_bmm_fma_cuda(Tensor a, Tensor b, Tensor c, int M,
                                            int N, int K, int man_fma,
                                            int exp_fma, bool subnormals,
                                            bool saturate) {

  if (a.sizes().size() > 2)
    bmm_fp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                          c.data_ptr<float>(), a.sizes()[0], M, K, N, man_fma,
                          exp_fma, subnormals, saturate);
  else
    bmm_fp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                          c.data_ptr<float>(), 1, M, K, N, man_fma, exp_fma,
                          subnormals, saturate);
}

void fixed_point_quantize_nearest_mm_cuda(Tensor a, Tensor b, Tensor c, int M,
                                          int N, int K, int wl_add, int fl_add,
                                          int wl_mul, int fl_mul,
                                          bool symmetric) {
  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  mm_fxp_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                 M, K, N, sigma_add, t_min_add, t_max_add, sigma_mul, t_min_mul,
                 t_max_mul);
  return;
}

void fixed_point_quantize_nearest_bmm_cuda(Tensor a, Tensor b, Tensor c, int M,
                                           int N, int K, int wl_add, int fl_add,
                                           int wl_mul, int fl_mul,
                                           bool symmetric) {

  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  if (a.sizes().size() > 2)
    bmm_fxp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                    c.data_ptr<float>(), a.sizes()[0], M, K, N, sigma_add,
                    t_min_add, t_max_add, sigma_mul, t_min_mul, t_max_mul);
  else
    bmm_fxp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                    c.data_ptr<float>(), 1, M, K, N, sigma_add, t_min_add,
                    t_max_add, sigma_mul, t_min_mul, t_max_mul);
  return;
}

void fixed_point_quantize_nearest_mm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                              int M, int N, int K, int wl_fma,
                                              int fl_fma, bool symmetric) {
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  mm_fxp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                     c.data_ptr<float>(), M, K, N, sigma_fma, t_min_fma,
                     t_max_fma);
  return;
}

void fixed_point_quantize_nearest_bmm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                               int M, int N, int K, int wl_fma,
                                               int fl_fma, bool symmetric) {
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  if (a.sizes().size() > 2)
    bmm_fxp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                        c.data_ptr<float>(), a.sizes()[0], M, K, N, sigma_fma,
                        t_min_fma, t_max_fma);
  else
    bmm_fxp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                        c.data_ptr<float>(), 1, M, K, N, sigma_fma, t_min_fma,
                        t_max_fma);
  return;
}

void fixed_point_quantize_stochastic_mm_cuda(Tensor a, Tensor b, Tensor c,
                                             int M, int N, int K, int wl_add,
                                             int fl_add, int wl_mul, int fl_mul,
                                             bool symmetric) {
  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  mm_fxp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                    c.data_ptr<float>(), M, K, N, sigma_add, t_min_add,
                    t_max_add, sigma_mul, t_min_mul, t_max_mul);
  return;
}

void fixed_point_quantize_stochastic_bmm_cuda(Tensor a, Tensor b, Tensor c,
                                              int M, int N, int K, int wl_add,
                                              int fl_add, int wl_mul,
                                              int fl_mul, bool symmetric) {
  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  if (a.sizes().size() > 2)
    bmm_fxp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), a.sizes()[0], M, K, N, sigma_add,
                       t_min_add, t_max_add, sigma_mul, t_min_mul, t_max_mul);
  else
    bmm_fxp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), 1, M, K, N, sigma_add, t_min_add,
                       t_max_add, sigma_mul, t_min_mul, t_max_mul);
  return;
}

void fixed_point_quantize_stochastic_mm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                                 int M, int N, int K,
                                                 int wl_fma, int fl_fma,
                                                 bool symmetric) {
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  mm_fxp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                        c.data_ptr<float>(), M, K, N, sigma_fma, t_min_fma,
                        t_max_fma);
  return;
}

void fixed_point_quantize_stochastic_bmm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                                  int M, int N, int K,
                                                  int wl_fma, int fl_fma,
                                                  bool symmetric) {
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  if (a.sizes().size() > 2)
    bmm_fxp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                           c.data_ptr<float>(), a.sizes()[0], M, K, N,
                           sigma_fma, t_min_fma, t_max_fma);
  else
    bmm_fxp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                           c.data_ptr<float>(), 1, M, K, N, sigma_fma,
                           t_min_fma, t_max_fma);
  return;
}