#include "hip/hip_runtime.h"
#include "quant.h"
#include "quant_kernel.h"
#include "binary8_kernel.h"
#include <ATen/ATen.h>
#include <climits>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdint>
#include <tuple>
#include <stdexcept>
#include <cassert>
#include <vector>

using namespace at;

Tensor get_max_entry(Tensor a, int dim)
{
  Tensor max_entry;
  if (dim == -1)
  {
    max_entry = at::max(at::abs(a)).expand_as(a).contiguous();
  }
  else if (dim == 0)
  {
    Tensor input_view = a.view({a.size(0), -1});
    max_entry = std::get<0>(input_view.abs().max(1, true))
                    .expand_as(input_view)
                    .view_as(a)
                    .contiguous();
  }
  else
  {
    Tensor input_transpose = a.transpose(0, dim);
    Tensor input_view =
        input_transpose.contiguous().view({input_transpose.size(0), -1});
    Tensor max_transpose = std::get<0>(input_view.abs().max(1, true))
                               .expand_as(input_view)
                               .view_as(input_transpose);
    max_entry = max_transpose.transpose(dim, 0).contiguous();
  }
  return max_entry;
}

Tensor block_quantize_stochastic_cuda(Tensor a, int wl, int dim)
{
  auto o = at::zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int64_t size = a.numel();

  Tensor max_entry = get_max_entry(a, dim);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_ints.data_ptr<int>(), o.data_ptr<float>(), size,
      max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor block_quantize_nearest_cuda(Tensor a, int wl, int dim)
{
  auto o = at::zeros_like(a);
  int64_t size = a.numel();

  Tensor max_entry = get_max_entry(a, dim);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size,
      max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor block_quantize_sim_stochastic_cuda(Tensor a, int wl)
{
  auto o = at::zeros_like(a);
  auto rand_probs = rand_like(a);
  int64_t size = a.numel();

  Tensor max_entry = at::max(at::abs(a));
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_sim_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_probs.data_ptr<float>(), o.data_ptr<float>(),
      size, max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor block_quantize_sim_nearest_cuda(Tensor a, int wl)
{
  auto o = at::zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int64_t size = a.numel();

  Tensor max_entry = at::max(at::abs(a));
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  block_kernel_sim_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size,
      max_entry.data_ptr<float>(), wl);
  return o;
}

Tensor float_quantize_stochastic_cuda(Tensor a,
                                      int man_bits, int exp_bits,
                                      bool subnormals, bool saturate)
{
  // use external random number right now
  auto o = zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int size = a.numel();
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  float_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_ints.data_ptr<int>(), o.data_ptr<float>(), size,
      man_bits, exp_bits, subnormals, saturate);
  return o;
}

Tensor float_quantize_stochastic_cuda(Tensor a,
                                      int man_bits, int exp_bits, int prng_bits,
                                      bool subnormals, bool saturate)
{
  // use external random number right now
  auto o = zeros_like(a);
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int size = a.numel();
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  float_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_ints.data_ptr<int>(), o.data_ptr<float>(), size,
      man_bits, exp_bits, prng_bits, subnormals, saturate);
  return o;
}

Tensor float_quantize_nearest_cuda(Tensor a,
                                   int man_bits, int exp_bits,
                                   bool subnormals, bool saturate)
{
  auto o = zeros_like(a);
  int size = a.numel();
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  float_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size, man_bits, exp_bits,
      subnormals, saturate);
  return o;
}

Tensor superfp_quantize_nearest_cuda(Tensor a,
                                     int man_bits, int exp_bits,
                                     int binades_l, int binades_u,
                                     bool saturate)
{
  auto o = zeros_like(a);
  int size = a.numel();
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  superfp_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size, man_bits, exp_bits, binades_l, binades_u, saturate);
  return o;
}

Tensor binary8_quantize_nearest_cuda(Tensor a,
                                     int P, bool is_signed, OverflowPolicy overflow_policy,
                                     bool subnormals)
{
  auto o = zeros_like(a);
  int size = a.numel(); // gets number of elements in tensor a
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  if (is_signed == true)
  { // signed
    binary8_signed_kernel_nearest<<<blockNums, blockSize>>>(
        a.data_ptr<float>(), o.data_ptr<float>(), size, P, overflow_policy, subnormals);
  }
  else
  { // unsigned
    binary8_unsigned_kernel_nearest<<<blockNums, blockSize>>>(
        a.data_ptr<float>(), o.data_ptr<float>(), size, P, overflow_policy, subnormals);
  }

  return o;
}

Tensor binary8_quantize_stochastic_cuda(Tensor a,
                                        int P, int prng_bits, bool is_signed, OverflowPolicy overflow_policy,
                                        bool subnormals)
{
  auto o = zeros_like(a);
  // generate random number on the GPU for the SR operation
  auto rand_ints = randint_like(a, INT_MAX, device(kCUDA).dtype(kInt));
  int size = a.numel(); // gets number of elements in tensor a
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  if (is_signed == true)
  { // signed
    binary8_signed_kernel_stochastic<<<blockNums, blockSize>>>(
        a.data_ptr<float>(), rand_ints.data_ptr<int>(), o.data_ptr<float>(), size, P, prng_bits, overflow_policy, subnormals);
  }
  else
  { // unsigned
    binary8_unsigned_kernel_stochastic<<<blockNums, blockSize>>>(
        a.data_ptr<float>(), rand_ints.data_ptr<int>(), o.data_ptr<float>(), size, P, prng_bits, overflow_policy, subnormals);
  }

  return o;
}

Tensor binary8_quantize_truncate_cuda(Tensor a,
                                      int P, bool is_signed, OverflowPolicy overflow_policy,
                                      bool subnormals)
{
  auto o = zeros_like(a);
  int size = a.numel(); // gets number of elements in tensor a
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  if (is_signed == true)
  { // signed
    binary8_signed_kernel_truncate<<<blockNums, blockSize>>>(
        a.data_ptr<float>(), o.data_ptr<float>(), size, P, overflow_policy, subnormals);
  }
  else
  { // unsigned
    binary8_unsigned_kernel_truncate<<<blockNums, blockSize>>>(
        a.data_ptr<float>(), o.data_ptr<float>(), size, P, overflow_policy, subnormals);
  }

  return o;
}

void fixed_min_max(int wl, int fl, bool symmetric, float *t_min, float *t_max)
{
  int sigma = -fl;
  *t_min = -ldexp(1.0, wl - fl - 1);
  *t_max = -*t_min - ldexp(1.0, sigma);
  if (symmetric)
    *t_min = *t_min + ldexp(1.0, sigma);
}

Tensor fixed_point_quantize_stochastic_cuda(Tensor a,
                                            int wl, int fl,
                                            bool use_clamp, bool symmetric)
{
  // use external random number right now
  auto o = at::zeros_like(a);
  auto rand_probs = rand_like(a);
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_probs.data_ptr<float>(), o.data_ptr<float>(),
      size, sigma, use_clamp, t_min, t_max);
  return o;
}

Tensor fixed_point_quantize_nearest_cuda(Tensor a,
                                         int wl, int fl,
                                         bool use_clamp, bool symmetric)
{
  // use external random number right now
  auto o = at::zeros_like(a);
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), size, sigma, use_clamp, t_min,
      t_max);
  return o;
}

std::tuple<Tensor, Tensor>
fixed_point_quantize_stochastic_mask_cuda(Tensor a,
                                          int wl, int fl,
                                          bool symmetric)
{
  // use external random number right now
  auto o = zeros_like(a);
  auto rand_probs = rand_like(a);
  auto m = zeros_like(a, a.options().dtype(kByte));
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_mask_stochastic<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), rand_probs.data_ptr<float>(), o.data_ptr<float>(),
      m.data_ptr<uint8_t>(), size, sigma, t_min, t_max);
  return std::make_tuple(o, m);
}

std::tuple<Tensor, Tensor>
fixed_point_quantize_nearest_mask_cuda(Tensor a,
                                       int wl, int fl,
                                       bool symmetric)
{
  // use external random number right now
  auto o = at::zeros_like(a);
  auto m = zeros_like(a, a.options().dtype(kByte));
  int64_t size = a.numel();
  int sigma = -fl;
  float t_min, t_max;
  fixed_min_max(wl, fl, symmetric, &t_min, &t_max);
  int blockSize = 1024;
  int blockNums = (size + blockSize - 1) / blockSize;

  fixed_point_quantize_kernel_mask_nearest<<<blockNums, blockSize>>>(
      a.data_ptr<float>(), o.data_ptr<float>(), m.data_ptr<uint8_t>(), size,
      sigma, t_min, t_max);
  return std::make_tuple(o, m);
}

void float_quantize_nearest_mm_cuda(Tensor a, Tensor b, Tensor c,
                                    int M, int N, int K,
                                    int man_add, int exp_add,
                                    int man_mul, int exp_mul,
                                    bool subnormals,
                                    bool saturate,
                                    bool compensated)
{
  mm_fp_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                M, K, N, man_add, exp_add, man_mul, exp_mul,
                subnormals, saturate, compensated);
  return;
}

void float_quantize_nearest_mm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                        int M, int N, int K,
                                        int man_fma, int exp_fma,
                                        bool subnormals,
                                        bool saturate,
                                        bool compensated)
{
  mm_fp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                    M, K, N, man_fma, exp_fma,
                    subnormals, saturate, compensated);
  return;
}

void float_quantize_nearest_bmm_cuda(Tensor a, Tensor b, Tensor c,
                                     int M, int N, int K,
                                     int man_add, int exp_add,
                                     int man_mul, int exp_mul,
                                     bool subnormals,
                                     bool saturate,
                                     bool compensated)
{
  if (a.sizes().size() > 2)
    bmm_fp_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                   a.sizes()[0], M, K, N,
                   man_add, exp_add, man_mul, exp_mul,
                   subnormals, saturate, compensated);
  else
    bmm_fp_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                   1, M, K, N,
                   man_add, exp_add, man_mul, exp_mul,
                   subnormals, saturate, compensated);
  return;
}

void float_quantize_nearest_bmm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                         int M, int N, int K,
                                         int man_fma, int exp_fma,
                                         bool subnormals,
                                         bool saturate,
                                         bool compensated)
{
  if (a.sizes().size() > 2)
    bmm_fp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                       a.sizes()[0], M, K, N,
                       man_fma, exp_fma,
                       subnormals, saturate, compensated);
  else
    bmm_fp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                       1, M, K, N,
                       man_fma, exp_fma,
                       subnormals, saturate, compensated);
  return;
}

void superfp_quantize_nearest_mm_cuda(Tensor a, Tensor b, Tensor c,
                                      int M, int N, int K,
                                      int man_add, int exp_add,
                                      int man_mul, int exp_mul,
                                      int binades_add_l, int binades_add_u,
                                      int binades_mul_l, int binades_mul_u,
                                      bool saturate)
{
  mm_superfp_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                     M, K, N, man_add, exp_add, man_mul, exp_mul,
                     binades_add_l, binades_add_u,
                     binades_mul_l, binades_mul_u,
                     saturate);
}

void superfp_quantize_nearest_mm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                          int M, int N, int K,
                                          int man_fma, int exp_fma,
                                          int binades_fma_l, int binades_fma_u,
                                          bool saturate)
{
  mm_superfp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                         c.data_ptr<float>(), M, K, N, man_fma, exp_fma,
                         binades_fma_l, binades_fma_u,
                         saturate);
  return;
}

void superfp_quantize_nearest_bmm_cuda(Tensor a, Tensor b, Tensor c,
                                       int M, int N, int K,
                                       int man_add, int exp_add,
                                       int man_mul, int exp_mul,
                                       int binades_add_l, int binades_add_u,
                                       int binades_mul_l, int binades_mul_u,
                                       bool saturate)
{
  if (a.sizes().size() > 2)
    bmm_superfp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                        c.data_ptr<float>(), a.sizes()[0], M, K, N, man_add, exp_add,
                        man_mul, exp_mul, binades_add_l, binades_add_u,
                        binades_mul_l, binades_mul_u, saturate);
  else
    bmm_superfp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                        c.data_ptr<float>(), 1, M, K, N, man_add, exp_add, man_mul,
                        exp_mul, binades_add_l, binades_add_u,
                        binades_mul_l, binades_mul_u, saturate);
  return;
}

void superfp_quantize_nearest_bmm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                           int M, int N, int K,
                                           int man_fma, int exp_fma,
                                           int binades_fma_l, int binades_fma_u,
                                           bool saturate)
{
  if (a.sizes().size() > 2)
    bmm_superfp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                            c.data_ptr<float>(), a.sizes()[0], M, K, N, man_fma,
                            exp_fma, binades_fma_l, binades_fma_u, saturate);
  else
    bmm_superfp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                            c.data_ptr<float>(), 1, M, K, N, man_fma, exp_fma,
                            binades_fma_l, binades_fma_u, saturate);
  return;
}

void float_quantize_stochastic_mm_cuda(Tensor a, Tensor b, Tensor c,
                                       int M, int N, int K,
                                       int man_add, int exp_add, int rbits_add,
                                       int man_mul, int exp_mul, int rbits_mul,
                                       bool subnormals, bool saturate)
{
  mm_fp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                   M, K, N,
                   man_add, exp_add, rbits_add,
                   man_mul, exp_mul, rbits_mul,
                   subnormals, saturate);
  return;
}

void float_quantize_stochastic_mm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                           int M, int N, int K,
                                           int man_fma, int exp_fma, int rbits_fma,
                                           bool subnormals, bool saturate)
{
  mm_fp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                       M, K, N,
                       man_fma, exp_fma, rbits_fma,
                       subnormals, saturate);
  return;
}

void float_quantize_stochastic_bmm_cuda(Tensor a, Tensor b, Tensor c,
                                        int M, int N, int K,
                                        int man_add, int exp_add, int rbits_add,
                                        int man_mul, int exp_mul, int rbits_mul,
                                        bool subnormals, bool saturate)
{
  if (a.sizes().size() > 2)
    bmm_fp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                      a.sizes()[0], M, K, N,
                      man_add, exp_add, rbits_add,
                      man_mul, exp_mul, rbits_mul,
                      subnormals, saturate);
  else
    bmm_fp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                      1, M, K, N,
                      man_add, exp_add, rbits_add,
                      man_mul, exp_mul, rbits_mul,
                      subnormals, saturate);
}

void float_quantize_stochastic_bmm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                            int M, int N, int K,
                                            int man_fma, int exp_fma, int rbits_fma,
                                            bool subnormals, bool saturate)
{

  if (a.sizes().size() > 2)
    bmm_fp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                          a.sizes()[0], M, K, N,
                          man_fma, exp_fma, rbits_fma,
                          subnormals, saturate);
  else
    bmm_fp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                          1, M, K, N,
                          man_fma, exp_fma, rbits_fma,
                          subnormals, saturate);
}

void fixed_point_quantize_nearest_mm_cuda(Tensor a, Tensor b, Tensor c,
                                          int M, int N, int K,
                                          int wl_add, int fl_add,
                                          int wl_mul, int fl_mul,
                                          bool symmetric)
{
  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  mm_fxp_nearest(a.data_ptr<float>(), b.data_ptr<float>(), c.data_ptr<float>(),
                 M, K, N, sigma_add, t_min_add, t_max_add, sigma_mul, t_min_mul,
                 t_max_mul);
  return;
}

void fixed_point_quantize_nearest_bmm_cuda(Tensor a, Tensor b, Tensor c,
                                           int M, int N, int K,
                                           int wl_add, int fl_add,
                                           int wl_mul, int fl_mul,
                                           bool symmetric)
{

  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  if (a.sizes().size() > 2)
    bmm_fxp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                    c.data_ptr<float>(), a.sizes()[0], M, K, N, sigma_add,
                    t_min_add, t_max_add, sigma_mul, t_min_mul, t_max_mul);
  else
    bmm_fxp_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                    c.data_ptr<float>(), 1, M, K, N, sigma_add, t_min_add,
                    t_max_add, sigma_mul, t_min_mul, t_max_mul);
  return;
}

void fixed_point_quantize_nearest_mm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                              int M, int N, int K,
                                              int wl_fma, int fl_fma,
                                              bool symmetric)
{
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  mm_fxp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                     c.data_ptr<float>(), M, K, N, sigma_fma, t_min_fma,
                     t_max_fma);
  return;
}

void fixed_point_quantize_nearest_bmm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                               int M, int N, int K,
                                               int wl_fma, int fl_fma,
                                               bool symmetric)
{
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  if (a.sizes().size() > 2)
    bmm_fxp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                        c.data_ptr<float>(), a.sizes()[0], M, K, N, sigma_fma,
                        t_min_fma, t_max_fma);
  else
    bmm_fxp_fma_nearest(a.data_ptr<float>(), b.data_ptr<float>(),
                        c.data_ptr<float>(), 1, M, K, N, sigma_fma, t_min_fma,
                        t_max_fma);
  return;
}

void fixed_point_quantize_stochastic_mm_cuda(Tensor a, Tensor b, Tensor c,
                                             int M, int N, int K,
                                             int wl_add, int fl_add,
                                             int wl_mul, int fl_mul,
                                             bool symmetric)
{
  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  mm_fxp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                    c.data_ptr<float>(), M, K, N, sigma_add, t_min_add,
                    t_max_add, sigma_mul, t_min_mul, t_max_mul);
  return;
}

void fixed_point_quantize_stochastic_bmm_cuda(Tensor a, Tensor b, Tensor c,
                                              int M, int N, int K,
                                              int wl_add, int fl_add,
                                              int wl_mul, int fl_mul,
                                              bool symmetric)
{
  int sigma_add = -fl_add;
  int sigma_mul = -fl_mul;
  float t_min_add, t_max_add, t_min_mul, t_max_mul;
  fixed_min_max(wl_add, fl_add, symmetric, &t_min_add, &t_max_add);
  fixed_min_max(wl_mul, fl_mul, symmetric, &t_min_mul, &t_max_mul);
  if (a.sizes().size() > 2)
    bmm_fxp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), a.sizes()[0], M, K, N, sigma_add,
                       t_min_add, t_max_add, sigma_mul, t_min_mul, t_max_mul);
  else
    bmm_fxp_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                       c.data_ptr<float>(), 1, M, K, N, sigma_add, t_min_add,
                       t_max_add, sigma_mul, t_min_mul, t_max_mul);
  return;
}

void fixed_point_quantize_stochastic_mm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                                 int M, int N, int K,
                                                 int wl_fma, int fl_fma,
                                                 bool symmetric)
{
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  mm_fxp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                        c.data_ptr<float>(), M, K, N, sigma_fma, t_min_fma,
                        t_max_fma);
  return;
}

void fixed_point_quantize_stochastic_bmm_fma_cuda(Tensor a, Tensor b, Tensor c,
                                                  int M, int N, int K,
                                                  int wl_fma, int fl_fma,
                                                  bool symmetric)
{
  int sigma_fma = -fl_fma;
  float t_min_fma, t_max_fma;
  fixed_min_max(wl_fma, fl_fma, symmetric, &t_min_fma, &t_max_fma);
  if (a.sizes().size() > 2)
    bmm_fxp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                           c.data_ptr<float>(), a.sizes()[0], M, K, N,
                           sigma_fma, t_min_fma, t_max_fma);
  else
    bmm_fxp_fma_stochastic(a.data_ptr<float>(), b.data_ptr<float>(),
                           c.data_ptr<float>(), 1, M, K, N, sigma_fma,
                           t_min_fma, t_max_fma);
  return;
}

static DimSizes partition_tensor(Tensor input, std::vector<int> &dims)
{
  DimSizes sizes;
  std::vector<int> real_dims(dims.size());
  for (int i = 0; i < dims.size(); i++)
  {
    real_dims[i] = (input.dim() + (dims[i] % input.dim())) % input.dim();
  }

  sizes.channel = 1;
  for (int dim : real_dims)
  {
    sizes.channel *= input.size(dim);
  }

  int min_dim = real_dims.back();
  int max_dim = real_dims.front();

  sizes.outer = 1;
  for (int i = 0; i < min_dim; i++)
  {
    sizes.outer *= input.size(i);
  }

  sizes.inner = 1;
  for (int i = max_dim + 1; i < input.dim(); i++)
  {
    sizes.inner *= input.size(i);
  }
  return sizes;
}

static DimSizes partition_tensor(Tensor a, int dim)
{
  DimSizes sizes;
  int real_dim = (a.dim() + (dim % a.dim())) % a.dim();
  sizes.outer = 1;
  sizes.channel = a.size(real_dim);
  sizes.inner = 1;
  for (int i = 0; i < real_dim; ++i)
  {
    sizes.outer *= a.size(i);
  }
  for (int i = real_dim + 1; i < a.dim(); ++i)
  {
    sizes.inner *= a.size(i);
  }
  return sizes;
}

void float_quantize_nearest_layernorm_forward_cuda(Tensor input, Tensor weight, Tensor bias,
                                                   Tensor output, Tensor mean, Tensor rstd,
                                                   float eps, std::vector<int> &dims,
                                                   int man_acc, int exp_acc,
                                                   int man_mul, int exp_mul,
                                                   int man_div, int exp_div,
                                                   int man_sqrt, int exp_sqrt,
                                                   bool subnormals, bool saturate)
{
  auto sizes = partition_tensor(input, dims);
  layernorm_forward_fp_nearest(input.data_ptr<float>(), weight.data_ptr<float>(), bias.data_ptr<float>(),
                               output.data_ptr<float>(), mean.data_ptr<float>(), rstd.data_ptr<float>(),
                               eps, sizes,
                               man_acc, exp_acc,
                               man_mul, exp_mul,
                               man_div, exp_div,
                               man_sqrt, exp_sqrt,
                               subnormals, saturate);
}

void float_quantize_nearest_layernorm_backward_cuda(Tensor input, Tensor grad_output,
                                                    Tensor weight, Tensor bias,
                                                    Tensor mean, Tensor rstd,
                                                    Tensor grad_input, Tensor grad_gamma, Tensor grad_beta,
                                                    std::vector<int> &dims,
                                                    int man_acc, int exp_acc,
                                                    int man_mul, int exp_mul,
                                                    int man_div, int exp_div,
                                                    bool subnormals, bool saturate)
{
  auto sizes = partition_tensor(input, dims);
  layernorm_backward_fp_nearest(input.data_ptr<float>(), grad_output.data_ptr<float>(),
                                weight.data_ptr<float>(), bias.data_ptr<float>(),
                                mean.data_ptr<float>(), rstd.data_ptr<float>(),
                                grad_input.data_ptr<float>(), grad_gamma.data_ptr<float>(), grad_beta.data_ptr<float>(),
                                sizes,
                                man_acc, exp_acc,
                                man_mul, exp_mul,
                                man_div, exp_div,
                                subnormals, saturate);
}

void superfp_quantize_nearest_layernorm_forward_cuda(Tensor input, Tensor weight, Tensor bias,
                                                     Tensor output, Tensor mean, Tensor rstd,
                                                     float eps, std::vector<int> &dims,
                                                     int man_acc, int exp_acc, int binades_acc_l, int binades_acc_u,
                                                     int man_mul, int exp_mul, int binades_mul_l, int binades_mul_u,
                                                     int man_div, int exp_div, int binades_div_l, int binades_div_u,
                                                     int man_sqrt, int exp_sqrt, int binades_sqrt_l, int binades_sqrt_u,
                                                     bool saturate)
{
  auto sizes = partition_tensor(input, dims);
  layernorm_forward_superfp_nearest(input.data_ptr<float>(), weight.data_ptr<float>(), bias.data_ptr<float>(),
                                    output.data_ptr<float>(), mean.data_ptr<float>(), rstd.data_ptr<float>(),
                                    eps, sizes,
                                    man_acc, exp_acc, binades_acc_l, binades_acc_u,
                                    man_mul, exp_mul, binades_mul_l, binades_mul_u,
                                    man_div, exp_div, binades_div_l, binades_div_u,
                                    man_sqrt, exp_sqrt, binades_sqrt_l, binades_sqrt_u,
                                    saturate);
}

void superfp_quantize_nearest_layernorm_backward_cuda(Tensor input, Tensor grad_output,
                                                      Tensor weight, Tensor bias,
                                                      Tensor mean, Tensor rstd,
                                                      Tensor grad_input, Tensor grad_gamma, Tensor grad_beta,
                                                      std::vector<int> &dims,
                                                      int man_acc, int exp_acc, int binades_acc_l, int binades_acc_u,
                                                      int man_mul, int exp_mul, int binades_mul_l, int binades_mul_u,
                                                      int man_div, int exp_div, int binades_div_l, int binades_div_u,
                                                      bool saturate)
{
  auto sizes = partition_tensor(input, dims);
  layernorm_backward_superfp_nearest(input.data_ptr<float>(), grad_output.data_ptr<float>(),
                                     weight.data_ptr<float>(), bias.data_ptr<float>(),
                                     mean.data_ptr<float>(), rstd.data_ptr<float>(),
                                     grad_input.data_ptr<float>(), grad_gamma.data_ptr<float>(), grad_beta.data_ptr<float>(),
                                     sizes,
                                     man_acc, exp_acc, binades_acc_l, binades_acc_u,
                                     man_mul, exp_mul, binades_mul_l, binades_mul_u,
                                     man_div, exp_div, binades_div_l, binades_div_u,
                                     saturate);
}

void binary8_quantize_nearest_layernorm_forward_cuda(Tensor input, Tensor weight, Tensor bias,
                                                     Tensor output, Tensor mean, Tensor rstd,
                                                     float eps, std::vector<int> &dims,
                                                     int P_acc, OverflowPolicy op_acc, bool signed_acc,
                                                     int P_mul, OverflowPolicy op_mul, bool signed_mul,
                                                     int P_div, OverflowPolicy op_div, bool signed_div,
                                                     int P_sqrt, OverflowPolicy op_sqrt, bool signed_sqrt,
                                                     bool subnormals)
{
  auto sizes = partition_tensor(input, dims);
  layernorm_forward_binary8_nearest(input.data_ptr<float>(), weight.data_ptr<float>(), bias.data_ptr<float>(),
                                    output.data_ptr<float>(), mean.data_ptr<float>(), rstd.data_ptr<float>(),
                                    eps, sizes,
                                    P_acc, op_acc, signed_acc,
                                    P_mul, op_mul, signed_mul,
                                    P_div, op_div, signed_div,
                                    P_sqrt, op_sqrt, signed_sqrt,
                                    subnormals);
}

void binary8_quantize_nearest_layernorm_backward_cuda(Tensor input, Tensor grad_output,
                                                      Tensor weight, Tensor bias,
                                                      Tensor mean, Tensor rstd,
                                                      Tensor grad_input, Tensor grad_gamma, Tensor grad_beta,
                                                      std::vector<int> &dims,
                                                      int P_acc, OverflowPolicy op_acc, bool signed_acc,
                                                      int P_mul, OverflowPolicy op_mul, bool signed_mul,
                                                      int P_div, OverflowPolicy op_div, bool signed_div,
                                                      bool subnormals)
{
  auto sizes = partition_tensor(input, dims);
  layernorm_backward_binary8_nearest(input.data_ptr<float>(), grad_output.data_ptr<float>(),
                                     weight.data_ptr<float>(), bias.data_ptr<float>(),
                                     mean.data_ptr<float>(), rstd.data_ptr<float>(),
                                     grad_input.data_ptr<float>(), grad_gamma.data_ptr<float>(), grad_beta.data_ptr<float>(),
                                     sizes,
                                     P_acc, op_acc, signed_acc,
                                     P_mul, op_mul, signed_mul,
                                     P_div, op_div, signed_div,
                                     subnormals);
}

void float_quantize_nearest_softmax_forward_cuda(Tensor a, Tensor o, int dim,
                                                 int man_exp, int exp_exp,
                                                 int man_off, int exp_off,
                                                 int man_acc, int exp_acc,
                                                 bool subnormals, bool saturate)
{
  auto sizes = partition_tensor(a, dim);
  softmax_forward_fp_nearest(a.data_ptr<float>(), o.data_ptr<float>(), sizes,
                             man_exp, exp_exp,
                             man_off, exp_off,
                             man_acc, exp_acc,
                             subnormals, saturate);
}

void float_quantize_nearest_softmax_lse_forward_cuda(Tensor a, Tensor o, int dim,
                                                     int man_off, int exp_off,
                                                     int man_lse, int exp_lse,
                                                     bool subnormals, bool saturate)
{
  auto sizes = partition_tensor(a, dim);
  softmax_lse_forward_fp_nearest(a.data_ptr<float>(), o.data_ptr<float>(), sizes,
                                 man_off, exp_off,
                                 man_lse, exp_lse,
                                 subnormals, saturate);
}

void float_quantize_nearest_softmax_backward_cuda(Tensor a, Tensor g, Tensor o, int dim,
                                                  int man_add, int exp_add,
                                                  int man_mul, int exp_mul,
                                                  bool subnormals, bool saturate)
{
  auto sizes = partition_tensor(a, dim);
  softmax_backward_fp_nearest(a.data_ptr<float>(), g.data_ptr<float>(), o.data_ptr<float>(), sizes,
                              man_add, exp_add,
                              man_mul, exp_mul,
                              subnormals, saturate);
}

void superfp_quantize_nearest_softmax_forward_cuda(Tensor a, Tensor o, int dim,
                                                   int man_exp, int exp_exp, int binades_exp_l, int binades_exp_u,
                                                   int man_off, int exp_off, int binades_off_l, int binades_off_u,
                                                   int man_acc, int exp_acc, int binades_acc_l, int binades_acc_u,
                                                   bool saturate)
{
  auto sizes = partition_tensor(a, dim);
  softmax_forward_superfp_nearest(a.data_ptr<float>(), o.data_ptr<float>(), sizes,
                                  man_exp, exp_exp, binades_exp_l, binades_exp_u,
                                  man_off, exp_off, binades_off_l, binades_off_u,
                                  man_acc, exp_acc, binades_acc_l, binades_acc_u,
                                  saturate);
}

void superfp_quantize_nearest_softmax_lse_forward_cuda(Tensor a, Tensor o, int dim,
                                                       int man_off, int exp_off, int binades_off_l, int binades_off_u,
                                                       int man_lse, int exp_lse, int binades_lse_l, int binades_lse_u,
                                                       bool saturate)
{
  auto sizes = partition_tensor(a, dim);
  softmax_lse_forward_superfp_nearest(a.data_ptr<float>(), o.data_ptr<float>(), sizes,
                                      man_off, exp_off, binades_off_l, binades_off_u,
                                      man_lse, exp_lse, binades_lse_l, binades_lse_u,
                                      saturate);
}

void superfp_quantize_nearest_softmax_backward_cuda(Tensor a, Tensor g, Tensor o, int dim,
                                                    int man_add, int exp_add, int binades_add_l, int binades_add_u,
                                                    int man_mul, int exp_mul, int binades_mul_l, int binades_mul_u,
                                                    bool saturate)
{
  auto sizes = partition_tensor(a, dim);
  softmax_backward_superfp_nearest(a.data_ptr<float>(), g.data_ptr<float>(), o.data_ptr<float>(), sizes,
                                   man_add, exp_add, binades_add_l, binades_add_u,
                                   man_mul, exp_mul, binades_mul_l, binades_mul_u,
                                   saturate);
}

void binary8_quantize_nearest_softmax_forward_cuda(Tensor a, Tensor o, int dim,
                                                   int P_exp, OverflowPolicy op_exp, bool signed_exp,
                                                   int P_off, OverflowPolicy op_off, bool signed_off,
                                                   int P_acc, OverflowPolicy op_acc, bool signed_acc,
                                                   bool subnormals)
{
  auto sizes = partition_tensor(a, dim);
  softmax_forward_binary8_nearest(a.data_ptr<float>(), o.data_ptr<float>(), sizes,
                                  P_exp, op_exp, signed_exp,
                                  P_off, op_off, signed_off,
                                  P_acc, op_acc, signed_acc,
                                  subnormals);
}

void binary8_quantize_nearest_softmax_lse_forward_cuda(Tensor a, Tensor o, int dim,
                                                       int P_off, OverflowPolicy op_off, bool signed_off,
                                                       int P_lse, OverflowPolicy op_lse, bool signed_lse,
                                                       bool subnormals)
{
  auto sizes = partition_tensor(a, dim);
  softmax_lse_forward_binary8_nearest(a.data_ptr<float>(), o.data_ptr<float>(), sizes,
                                      P_off, op_off, signed_off,
                                      P_lse, op_lse, signed_lse,
                                      subnormals);
}

void binary8_quantize_nearest_softmax_backward_cuda(Tensor a, Tensor g, Tensor o, int dim,
                                                    int P_add, OverflowPolicy op_add, bool signed_add,
                                                    int P_mul, OverflowPolicy op_mul, bool signed_mul,
                                                    bool subnormals)
{
  auto sizes = partition_tensor(a, dim);
  softmax_backward_binary8_nearest(a.data_ptr<float>(), g.data_ptr<float>(), o.data_ptr<float>(), sizes,
                                   P_add, op_add, signed_add,
                                   P_mul, op_mul, signed_mul,
                                   subnormals);
}