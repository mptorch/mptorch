#include "hip/hip_runtime.h"
#include "bit_helper.cu"
#include "quant_kernel.h"
#include "sim_helper.cu"
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

template <bool subnormals>
__device__ float cast_p3109_signed_nearest(float origin_float, int P) {
    // P range from 1 to 7 in signed
    int exp_bits = 8-P;
    int man_bits = P-1; //P in unsigned
    
    int spec_exp = (P == 1) ? 1 : 0;

    uint32_t uval32, uval8;
    float fval8;
    uval32 = FLOAT_TO_BITS(&origin_float);

    int exp_val = (uval32 << 1 >> 24) - 127;

    // minimal and maximal exponent value in binary8
    int max_exp = (1 << (exp_bits -1)) - 1;
    int min_exp = spec_exp - max_exp;

    if (exp_val == 128) {             // inf/Nan case
        return origin_float;
    }
    
    int subnormal_shift = 0;
    if(((min_exp - exp_val) <= man_bits) && (exp_val < min_exp) && (subnormals)){ 
      subnormal_shift = min_exp - exp_val;
    }

    uval8 = round_bitwise_nearest(uval32, man_bits - subnormal_shift);
    uval8 = p3109_clip_exponent(exp_bits, man_bits, uval32, uval8, true, subnormals);
    fval8 = BITS_TO_FLOAT(&uval8);

    return fval8;
}

template <>
__device__ float cast_p3109_signed_nearest<false>(float origin_float, int P) {
    // P range from 1 to 7 in signed
    int exp_bits = 8-P;
    int man_bits = P-1; //P in unsigned
    
    int spec_exp = (P == 1) ? 1 : 0;

    uint32_t uval32, uval8;
    float fval8;
    uval32 = FLOAT_TO_BITS(&origin_float);

    int exp_val = (uval32 << 1 >> 24) - 127;

    // minimal and maximal exponent value in binary8
    int max_exp = (1 << (exp_bits -1)) - 1;
    int min_exp = spec_exp - max_exp;

    if (exp_val == 128) {             // inf/Nan case
        return origin_float;
    }

    uval8 = round_bitwise_nearest(uval32,man_bits - subnormal_shift);
    uval8 = p3109_clip_exponent(exp_bits, man_bits, uval32, uval8, true, subnormals);
    fval8 = BITS_TO_FLOAT(&uval8);

    return fval8;
}

template <bool subnormals>
__device__ float cast_p3109_signed_stochastic(float origin_float, int P, int prng_bits) {
  // TODO:
  return 0.0f;
}

template <>
__device__ float cast_p3109_signed_stochastic<false>(float origin_float, int P, int prng_bits) {
  // TODO:
  return 0.0f;
}

template <bool subnormals>
__device__ float cast_p3109_unsigned_nearest(float origin_float, int P) {
    // P range from 1 to 7 in signed
    int exp_bits = 8 - P + 1;
    int man_bits = P - 1; 
    
    int spec_exp = (P == 1) ? 1 : 0;

    uint32_t uval32, uval8;
    float fval8;
    uval32 = FLOAT_TO_BITS(&origin_float);


    int sign = uval32 >> 31;
    int exp_val = (uval32 << 1 >> 24) - 127;
    
    bool subnormals = true;

    // minimal and maximal exponent value in binary8
    int max_exp = (1 << (exp_bits -1)) - 1;
    int min_exp = spec_exp - max_exp;
    
    if(sign == 1){
        return NAN;
    }

    if (exp_val == 128) {             // inf/Nan case
        return origin_float;
    }
    
    int subnormal_shift = 0;
    if((min_exp - exp_val) <= man_bits && exp_val < min_exp && subnormals){ 
      subnormal_shift = min_exp - exp_val;
    }

    uval8 = round_bitwise_nearest(uval32, man_bits - subnormal_shift);
    uval8 = p3109_clip_exponent(exp_bits, man_bits, uval32, uval8, true, subnormals);
    fval8 = BITS_TO_FLOAT(&uval8);

    return fval8;
}

template <>
__device__ float cast_p3109_unsigned_nearest<false>(float origin_float, int P) {
    // P range from 1 to 7 in signed
    int exp_bits = 8 - P + 1;
    int man_bits = P - 1; 
    
    int spec_exp = (P == 1) ? 1 : 0;

    uint32_t uval32, uval8;
    float fval8;
    uval32 = FLOAT_TO_BITS(&origin_float);


    int sign = uval32 >> 31;
    int exp_val = (uval32 << 1 >> 24) - 127;
    
    bool subnormals = false;

    // minimal and maximal exponent value in binary8
    int max_exp = (1 << (exp_bits -1)) - 1;
    int min_exp = spec_exp - max_exp;
    
    if(sign == 1){
        return NAN;
    }

    if (exp_val == 128) {             // inf/Nan case
        return origin_float;
    }
    
    int subnormal_shift = 0;
    if((min_exp - exp_val) <= man_bits && exp_val < min_exp && subnormals){ 
      subnormal_shift = min_exp - exp_val;
    }

    uval8 = round_bitwise_nearest(uval32, man_bits - subnormal_shift);
    uval8 = p3109_clip_exponent(exp_bits, man_bits, uval32, uval8, true, subnormals);
    fval8 = BITS_TO_FLOAT(&uval8);

    return fval8;
}

template <bool subnormals>
__device__ float cast_p3109_unsigned_stochastic(float origin_float, int P, int prng_bits) {
  // TODO:
  return 0.0f;
}

template <>
__device__ float cast_p3109_unsigned_stochastic<false>(float origin_float, int P, int prng_bits) {
  // TODO:
  return 0.0f;
}

__global__ void p3109_signed_kernel_nearest(float *__restrict__ a, float *o, int size, int P, bool subnormals) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < size) {
      if (subnormals) {
          o[idx] = cast_p3109_signed_nearest<true>(a[idx], P);
      } else {
          o[idx] = cast_p3109_signed_nearest<false>(a[idx], P);
      }
  }
}

__global__ void p3109_unsigned_kernel_nearest(float *__restrict__ a, float *o, int size,
                                      int P, bool subnormals) {
  // TODO
}

__global__ void p3109_signed_kernel_stochastic(float *__restrict__ a, float *o, int size,
                                      int P, int prng_bits, bool subnormals) {
  // TODO
}

__global__ void p3109_unsigned_kernel_stochastic(float *__restrict__ a, float *o, int size,
                                      int P, int prng_bits, bool subnormals) {
  // TODO
}