#include "hip/hip_runtime.h"
#include "binary8_kernel.h"
#include <cmath>
#include <cstdint>

#define FLOAT_TO_BITS(x) (*reinterpret_cast<uint32_t *>(x))
#define BITS_TO_FLOAT(x) (*reinterpret_cast<float *>(x))

__host__ __device__ __forceinline__ uint32_t extract_exponent(float *a) {
  uint32_t temp = *(reinterpret_cast<uint32_t *>(a));
  temp = (temp << 1 >> 24); // single precision, 1 sign bit, 23 mantissa bits
  return temp - 127 + 1;    // exponent offset and virtual bit
}

__host__ __device__ __forceinline__ uint32_t
round_bitwise_stochastic(uint32_t target, uint32_t rand_prob, int man_bits) { // passing number of random bits as second parameter (all the bits after the least significant bit which is based on prng); target is the original number
  uint32_t mask = (1 << (23 - man_bits)) - 1; 
  uint32_t add_r = target + (rand_prob & mask); // adding random bits to target (which is not masked)
  uint32_t quantized = add_r & ~mask; // masking out bits on the right hand side of the significant bits (truncating)
  return quantized;
}

__host__ __device__ __forceinline__ uint32_t
round_bitwise_nearest_p1(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  // tie breaking rule offset
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}

__host__ __device__ __forceinline__ uint32_t
round_bitwise_nearest(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  // tie breaking rule offset
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  return add_r & ~((1 << std::min((23 - man_bits + offset),23)) - 1);
}

__host__ __device__ __forceinline__ uint32_t round_bitwise_up(uint32_t target, int man_bits) {
  uint32_t mask = (1 << (23 - man_bits)) - 1;
  uint32_t nexact = ((target << 1 >> 1) & mask) > 0u ? 1u : 0u;
  uint32_t sign = target >> 31;
  uint32_t rand_prob = (nexact & ~sign) << (23 - man_bits);
  uint32_t add_r = target + rand_prob;
  uint32_t quantized = add_r & ~mask;
  return quantized;
}

__host__ __device__ __forceinline__ uint32_t round_bitwise_down(uint32_t target, int man_bits) {
  uint32_t mask = (1 << (23 - man_bits)) - 1;
  uint32_t nexact = ((target << 1 >> 1) & mask) > 0u ? 1u : 0u;
  uint32_t sign = target >> 31;
  uint32_t rand_prob = (nexact & sign) << (23 - man_bits);
  uint32_t add_r = target + rand_prob;
  uint32_t quantized = add_r & ~mask;
  return quantized;
}

__host__ __device__ __forceinline__ uint32_t
clip_exponent(int exp_bits, int man_bits, uint32_t old_num,
              uint32_t quantized_num, bool saturate) {
  if (quantized_num == 0)
    return quantized_num;

  int quantized_exponent_store = quantized_num << 1 >> 24;
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127;
  int min_exponent_store = -((1 << (exp_bits - 1)) - 2) + 127;

  uint32_t old_sign = old_num >> 31 << 31;
  // saturate or overflow
  if (quantized_exponent_store > max_exponent_store) {
    if (saturate) {
      uint32_t max_man =
          (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits);
      uint32_t max_num = ((uint32_t)max_exponent_store << 23) | max_man;
      quantized_num = old_sign | max_num;
    } else {
      quantized_num =
          ((((uint32_t)1 << 31) - 1) ^ (((uint32_t)1 << 23) - 1));
      quantized_num = quantized_num | old_sign;
    }
  } else if (quantized_exponent_store < min_exponent_store) {
    uint32_t min_num = ((uint32_t)min_exponent_store << 23);
    uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23);
    uint32_t unsigned_quantized_num = quantized_num << 1 >> 1;
    if (unsigned_quantized_num > middle_num) {
      uint32_t old_sign = old_num >> 31 << 31;
      quantized_num = old_sign | min_num;
    } else {
      quantized_num = 0;
    }
  }
  return quantized_num;
}

__host__ __device__ __forceinline__ uint32_t
clip_max_exponent(int man_bits, uint32_t max_exponent,  uint32_t quantized_num) {
  uint32_t quantized_exponent = quantized_num << 1 >> 24 << 23; // 1 sign bit, 23 mantissa bits
  if (quantized_exponent > max_exponent) {
    uint32_t max_man = (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits); // 1 sign bit, 8 exponent bits
    uint32_t max_num = max_exponent | max_man;
    uint32_t old_sign = quantized_num >> 31 << 31;
    quantized_num = old_sign | max_num;
  }
  return quantized_num;
}

__device__ __forceinline__ uint32_t clip_exponent_with_subnormals(int exp_bits, int man_bits, uint32_t old_num,
                                                  uint32_t quantized_num, bool saturate = false)
{
    if (quantized_num == 0)
        return quantized_num;

    int quantized_exponent_store = quantized_num << 1 >> 24;
    int min_exponent_store = -((1 << (exp_bits - 1)) - 2) - man_bits + 127;

    uint32_t old_sign = old_num >> 31 << 31;
    // underflow or round to smallest non zero subnormal value
    if (quantized_exponent_store < min_exponent_store)
    {
        int offset = (quantized_exponent_store == (min_exponent_store - 1));
        quantized_num += offset * (1u << 23);
        quantized_num = quantized_num | old_sign;
        quantized_num = offset * quantized_num;
    }
    return quantized_num;
}

__device__ __forceinline__ uint32_t clip_exponent_without_subnormals(int exp_bits, int man_bits, uint32_t old_num,
                                                    uint32_t quantized_num, bool saturate = false)
{
    if (quantized_num == 0)
        return quantized_num;

    int quantized_exponent_store = quantized_num << 1 >> 24;
    int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127;
    int min_exponent_store = -((1 << (exp_bits - 1)) - 2) + 127;

    uint32_t old_sign = old_num >> 31 << 31;
    // saturate or overflow
    if (quantized_exponent_store > max_exponent_store)
    {
        if (saturate)
        {
            uint32_t max_man =
                (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits);
            uint32_t max_num = ((uint32_t)max_exponent_store << 23) | max_man;
            quantized_num = old_sign | max_num;
        }
        else
        {
            quantized_num = ((((uint32_t)1 << 31) - 1) ^ (((uint32_t)1 << 23) - 1));
            quantized_num = quantized_num | old_sign;
        }
    } // underflow or round to smallest nonzero normal value
    else if (quantized_exponent_store < min_exponent_store)
    {
        uint32_t offset = (quantized_exponent_store == (min_exponent_store - 1)) && ((old_num << 9 >> 9) > (1 << 22));
        quantized_num = offset * (min_exponent_store << 23);
        quantized_num |= old_sign;
    }
    return quantized_num;
}

__host__ __device__ __forceinline__ uint32_t 
binary8_clip_exponent(int exp_bits, int man_bits, uint32_t old_num, uint32_t quantized_num, OverflowPolicy overflow_policy, bool subnormal)
{
  if (quantized_num == 0){
    return quantized_num;
  }
  
  uint32_t man_val = quantized_num & 0x7FFFFF;
  uint32_t old_sign = old_num >> 31 << 31;
  uint32_t max_man;

  int spec_exp = (man_bits == 0) ? 1 : 0; // if P = 1
  int special_unsigned_exp = 0;

  max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits); // max mantissa = 0xfe in the normal case

  if (exp_bits + man_bits == 7 && overflow_policy == OverflowPolicy::SATURATE_RE){  // if signed and policy maxfloat_real then max mantissa = 0xff   
    max_man = ((1u << man_bits) - 1u) << (23 - man_bits);
  }
  
  if(overflow_policy != OverflowPolicy::SATURATE_RE){ // if we are not in OVERFLOW_MAXFLOAT_REALS policy :
    if(exp_bits == 8){ // unsigned and p=1
        special_unsigned_exp = 1; // 0 bit of mantissa so the max value 0xfd = max_exp - 1 | mantissa = 0
    }else if (exp_bits == 7 && man_bits == 1){ // unsigned and p=2 
        special_unsigned_exp = 1; // 1 bit of mantissa so the max value 0xfd = max_exp - 1 | mantissa = 1 
        max_man = ((1u << man_bits) - 1u) << (23 - man_bits);
    }else if(exp_bits + man_bits == 8){ // unsigned
        max_man = ((1u << man_bits) - 3u) << (23 - man_bits); // 2+ bit of mantissa so the max value is 0xfd = max_exp | max_mantissa - 1 
    }
  } 

  // if(exp_bits == 8 && overflow_policy == OverflowPolicy::OVERFLOW_INFTY){ // unsigned and p=1
  //     special_unsigned_exp = 1; // 0 bit of mantissa so the max value 0xfd = max_exp - 1 | mantissa = 0
  // }else if (exp_bits == 7 &&  man_bits == 1 && overflow_policy == OverflowPolicy::OVERFLOW_INFTY){ // unsigned and p=2 
  //     special_unsigned_exp = 1; // 1 bit of mantissa so the max value 0xfd = max_exp - 1 | mantissa = 1 
  // }else if(exp_bits + man_bits == 8){ // unsigned
  //     max_man = ((1u << man_bits) - 3u) << (23 - man_bits); // 2+ bit of mantissa so the max value is 0xfd = max_exp | max_mantissa - 1 
  // }

  // Special because in unsigned we want our min to be 1 less because the space is taken by the Nan
  int quantized_exponent_store = (quantized_num << 1 >> 24);
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127 - special_unsigned_exp; 
  int min_exponent_store = -((1 << (exp_bits - 1)) - 1) + 127 + spec_exp;

  if(!subnormal){
    min_exponent_store--;
  }

  if (quantized_exponent_store > max_exponent_store || ((quantized_exponent_store == max_exponent_store) && (man_val > max_man))) 
  {
    if (overflow_policy == OverflowPolicy::SATURATE_INFTY){ // Overflow to infinity (exceeds the max value 0xfe or 0xfd if signed or unsigned)
      return quantized_num = old_sign | 0x7F800000; // INF
    } 
    // Otherwise saturate to the max float value permitted by the policy reprensented by the max_man and max_exponent_store
    quantized_num = old_sign | ((uint32_t)max_exponent_store << 23) | max_man; 
  }
    if (quantized_exponent_store < min_exponent_store) {
      if (subnormal) {
        int subnormal_shift = min_exponent_store - quantized_exponent_store;
        int min_subnormals_exp = min_exponent_store - man_bits;
        uint32_t min_num = ((uint32_t)min_subnormals_exp << 23);
        uint32_t middle_num = ((uint32_t)(min_subnormals_exp - 1) << 23);
        if (subnormal_shift <= man_bits) {
          // quantized_num stays the same in this case
        } else if ((old_num & 0x7FFFFFFF) > middle_num) {
          quantized_num = old_sign | min_num;
        } else {
          quantized_num = 0;
        }
      } else {  // no subnormal case; normalizing subnormal values
          uint32_t min_num = ((uint32_t)min_exponent_store<< 23) | 1 << (23-man_bits);
          uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23 | 1 << (23-man_bits));
          if ((old_num & 0x7FFFFFFF) > middle_num){
            return quantized_num = old_sign | min_num;
          } else {
            return quantized_num = 0;
          }
      }
    }
    return quantized_num;
  }
