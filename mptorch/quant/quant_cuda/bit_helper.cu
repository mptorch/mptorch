#include "hip/hip_runtime.h"
#include "binary8_kernel.h"
#include <cmath>
#include <cstdint>

#define FLOAT_TO_BITS(x) (*reinterpret_cast<uint32_t *>(x))
#define BITS_TO_FLOAT(x) (*reinterpret_cast<float *>(x))

__host__ __device__ __forceinline__ uint32_t extract_exponent(float *a) {
  uint32_t temp = *(reinterpret_cast<uint32_t *>(a));
  temp = (temp << 1 >> 24); // single precision, 1 sign bit, 23 mantissa bits
  return temp - 127 + 1;    // exponent offset and virtual bit
}

__host__ __device__ __forceinline__ uint32_t
round_bitwise_stochastic(uint32_t target, uint32_t rand_prob, int man_bits) { // passing number of random bits as second parameter (all the bits after the least significant bit which is based on prng); target is the original number
  uint32_t mask = (1 << (23 - man_bits)) - 1; 
  uint32_t add_r = target + (rand_prob & mask); // adding random bits to target (which is not masked)
  uint32_t quantized = add_r & ~mask; // masking out bits on the right hand side of the significant bits (truncating)
  return quantized;
}

__host__ __device__ __forceinline__ uint32_t
round_bitwise_nearest_p1(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  // tie breaking rule offset
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}

__host__ __device__ __forceinline__ uint32_t
round_bitwise_nearest(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  // tie breaking rule offset
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  return add_r & ~((1 << std::min((23 - man_bits + offset),23)) - 1);
}

__host__ __device__ __forceinline__ uint32_t round_bitwise_up(uint32_t target, int man_bits) {
  uint32_t mask = (1 << (23 - man_bits)) - 1;
  uint32_t nexact = ((target << 1 >> 1) & mask) > 0u ? 1u : 0u;
  uint32_t sign = target >> 31;
  uint32_t rand_prob = (nexact & ~sign) << (23 - man_bits);
  uint32_t add_r = target + rand_prob;
  uint32_t quantized = add_r & ~mask;
  return quantized;
}

__host__ __device__ __forceinline__ uint32_t round_bitwise_down(uint32_t target, int man_bits) {
  uint32_t mask = (1 << (23 - man_bits)) - 1;
  uint32_t nexact = ((target << 1 >> 1) & mask) > 0u ? 1u : 0u;
  uint32_t sign = target >> 31;
  uint32_t rand_prob = (nexact & sign) << (23 - man_bits);
  uint32_t add_r = target + rand_prob;
  uint32_t quantized = add_r & ~mask;
  return quantized;
}

__host__ __device__ __forceinline__ uint32_t
clip_exponent(int exp_bits, int man_bits, uint32_t old_num,
              uint32_t quantized_num, bool saturate) {
  if (quantized_num == 0)
    return quantized_num;

  int quantized_exponent_store = quantized_num << 1 >> 24;
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127;
  int min_exponent_store = -((1 << (exp_bits - 1)) - 2) + 127;

  uint32_t old_sign = old_num >> 31 << 31;
  // saturate or overflow
  if (quantized_exponent_store > max_exponent_store) {
    if (saturate) {
      uint32_t max_man =
          (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits);
      uint32_t max_num = ((uint32_t)max_exponent_store << 23) | max_man;
      quantized_num = old_sign | max_num;
    } else {
      quantized_num =
          ((((uint32_t)1 << 31) - 1) ^ (((uint32_t)1 << 23) - 1));
      quantized_num = quantized_num | old_sign;
    }
  } else if (quantized_exponent_store < min_exponent_store) {
    uint32_t min_num = ((uint32_t)min_exponent_store << 23);
    uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23);
    uint32_t unsigned_quantized_num = quantized_num << 1 >> 1;
    if (unsigned_quantized_num > middle_num) {
      uint32_t old_sign = old_num >> 31 << 31;
      quantized_num = old_sign | min_num;
    } else {
      quantized_num = 0;
    }
  }
  return quantized_num;
}

__host__ __device__ __forceinline__ uint32_t
clip_max_exponent(int man_bits, uint32_t max_exponent,  uint32_t quantized_num) {
  uint32_t quantized_exponent = quantized_num << 1 >> 24 << 23; // 1 sign bit, 23 mantissa bits
  if (quantized_exponent > max_exponent) {
    uint32_t max_man = (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits); // 1 sign bit, 8 exponent bits
    uint32_t max_num = max_exponent | max_man;
    uint32_t old_sign = quantized_num >> 31 << 31;
    quantized_num = old_sign | max_num;
  }
  return quantized_num;
}

__host__ __device__ __forceinline__ uint32_t 
binary8_clip_exponent(int exp_bits, int man_bits, uint32_t old_num, uint32_t quantized_num, SaturationMode saturation_mode, bool subnormal) {

  if (quantized_num == 0){
    return quantized_num;
  }
  
  uint32_t man_val = quantized_num & 0x7FFFFF;
  uint32_t old_sign = old_num >> 31 << 31;
  uint32_t max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  
  int spec_exp = (man_bits == 0) ? 1 : 0; // if P = 1
  int special_unsigned_exp = 0;
  
  if(exp_bits == 8 && saturation_mode != SaturationMode::NO_OVERFLOW){ // unsigned and p=1
      special_unsigned_exp = 1; // 0 bit of mantissa so the max value 0xfd = max_exp - 1 | mantissa = 0
  }else if (exp_bits == 7 &&  man_bits == 1 && saturation_mode != SaturationMode::NO_OVERFLOW){ // unsigned and p=2 
      special_unsigned_exp = 1; // 1 bit of mantissa so the max value 0xfd = max_exp - 1 | mantissa = 1 
  }else if(exp_bits + man_bits == 8){ // unsigned
      max_man = ((1u << man_bits) - 3u) << (23 - man_bits); // 2+ bit of mantissa so the max value 0xfd = mACax_exp | max_mantissa - 1 
  }

  // Special because in unsigned we want our min to be 1 less because the space is taken by the Nan
  int quantized_exponent_store = (quantized_num << 1 >> 24);
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127 - special_unsigned_exp; 
  int min_exponent_store = -((1 << (exp_bits - 1)) - 1) + 127 + spec_exp;

  if (saturation_mode == SaturationMode::NO_OVERFLOW) { // Saturate to max without infinity
    max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  }

  if (quantized_exponent_store > max_exponent_store || ((quantized_exponent_store == max_exponent_store) && (man_val > max_man))) 
  {
    if (saturation_mode == SaturationMode::OVERFLOWS){ // Overflow to infinity
      return quantized_num = old_sign | 0x7F800000; // INF
    } 
    return quantized_num = old_sign | ((uint32_t)max_exponent_store << 23) | max_man;
  }
  if (quantized_exponent_store < min_exponent_store) {
    if (subnormal) {
        int subnormal_shift = min_exponent_store - quantized_exponent_store;
        int min_subnormals_exp = min_exponent_store - man_bits;
        uint32_t min_num = ((uint32_t)min_subnormals_exp << 23);
        uint32_t middle_num = ((uint32_t)(min_subnormals_exp - 1) << 23);
      if (subnormal_shift <= man_bits) {
        // quantized_num stays the same in this case
      } else if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    } 
    if(!subnormal) {
      uint32_t min_num = (uint32_t)min_exponent_store << 23;
      uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23);
      if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    }
  }
  return quantized_num;
}