
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdint>

#define FLOAT_TO_BITS(x) (*reinterpret_cast<uint32_t *>(x))
#define BITS_TO_FLOAT(x) (*reinterpret_cast<float *>(x))

__device__ __forceinline__ uint32_t extract_exponent(float *a) {
  uint32_t temp = *(reinterpret_cast<uint32_t *>(a));
  temp = (temp << 1 >> 24); // single precision, 1 sign bit, 23 mantissa bits
  return temp - 127 + 1;    // exponent offset and virtual bit
}

__device__ __forceinline__ uint32_t
round_bitwise_stochastic(uint32_t target, uint32_t rand_prob,
                         int man_bits) {
  uint32_t mask = (1 << (23 - man_bits)) - 1;
  uint32_t add_r = target + (rand_prob & mask);
  uint32_t quantized = add_r & ~mask;
  return quantized;
}

__device__ __forceinline__ uint32_t
round_bitwise_nearest(uint32_t target, int man_bits) {
    uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
    uint32_t machine_eps = 1 << (22 - man_bits);
    // tie breaking rule offset
    int offset = (down == machine_eps);
    uint32_t add_r = target + machine_eps;
    // apply the mask
    // this is the analogue of how you would do round 
    // to nearest integer using the floor function: 
    // round(x) = floor(x + 0.5)
    return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}

__device__ __forceinline__ uint32_t round_bitwise_up(uint32_t target,
                                                         int man_bits) {
  uint32_t mask = (1 << (23 - man_bits)) - 1;

  uint32_t nexact = ((target << 1 >> 1) & mask) > 0u ? 1u : 0u;
  uint32_t sign = target >> 31;
  uint32_t rand_prob = (nexact & ~sign) << (23 - man_bits);
  uint32_t add_r = target + rand_prob;
  uint32_t quantized = add_r & ~mask;
  return quantized;
}

__device__ __forceinline__ uint32_t round_bitwise_down(uint32_t target,
                                                           int man_bits) {
  uint32_t mask = (1 << (23 - man_bits)) - 1;

  uint32_t nexact = ((target << 1 >> 1) & mask) > 0u ? 1u : 0u;
  uint32_t sign = target >> 31;
  uint32_t rand_prob = (nexact & sign) << (23 - man_bits);
  uint32_t add_r = target + rand_prob;
  uint32_t quantized = add_r & ~mask;
  return quantized;
}

__device__ __forceinline__ uint32_t
clip_exponent(int exp_bits, int man_bits, uint32_t old_num,
              uint32_t quantized_num, bool saturate = false) {
  if (quantized_num == 0)
    return quantized_num;

  int quantized_exponent_store = quantized_num << 1 >> 24;
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127;
  int min_exponent_store = -((1 << (exp_bits - 1)) - 2) + 127;

  uint32_t old_sign = old_num >> 31 << 31;
  // saturate or overflow
  if (quantized_exponent_store > max_exponent_store) {
    if (saturate) {
      uint32_t max_man =
          (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits);
      uint32_t max_num = ((uint32_t)max_exponent_store << 23) | max_man;
      quantized_num = old_sign | max_num;
    } else {
      quantized_num =
          ((((uint32_t)1 << 31) - 1) ^ (((uint32_t)1 << 23) - 1));
      quantized_num = quantized_num | old_sign;
    }
  } else if (quantized_exponent_store < min_exponent_store) {
    uint32_t min_num = ((uint32_t)min_exponent_store << 23);
    uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23);
    uint32_t unsigned_quantized_num = quantized_num << 1 >> 1;
    if (unsigned_quantized_num > middle_num) {
      uint32_t old_sign = old_num >> 31 << 31;
      quantized_num = old_sign | min_num;
    } else {
      quantized_num = 0;
    }
  }
  return quantized_num;
}

__device__ uint32_t clip_exponent_with_subnormals(int exp_bits, int man_bits, uint32_t old_num,
                                                  uint32_t quantized_num, bool saturate = false)
{
    if (quantized_num == 0)
        return quantized_num;

    int quantized_exponent_store = quantized_num << 1 >> 24;
    int min_exponent_store = -((1 << (exp_bits - 1)) - 2) - man_bits + 127;

    uint32_t old_sign = old_num >> 31 << 31;
    // underflow or round to smallest non zero subnormal value
    if (quantized_exponent_store < min_exponent_store)
    {
        int offset = (quantized_exponent_store == (min_exponent_store - 1));
        quantized_num += offset * (1u << 23);
        quantized_num = quantized_num | old_sign;
        quantized_num = offset * quantized_num;
    }
    return quantized_num;
}

__device__ uint32_t clip_exponent_without_subnormals(int exp_bits, int man_bits, uint32_t old_num,
                                                    uint32_t quantized_num, bool saturate = false)
{
    if (quantized_num == 0)
        return quantized_num;

    int quantized_exponent_store = quantized_num << 1 >> 24;
    int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127;
    int min_exponent_store = -((1 << (exp_bits - 1)) - 2) + 127;

    uint32_t old_sign = old_num >> 31 << 31;
    // saturate or overflow
    if (quantized_exponent_store > max_exponent_store)
    {
        if (saturate)
        {
            uint32_t max_man =
                (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits);
            uint32_t max_num = ((uint32_t)max_exponent_store << 23) | max_man;
            quantized_num = old_sign | max_num;
        }
        else
        {
            quantized_num = ((((uint32_t)1 << 31) - 1) ^ (((uint32_t)1 << 23) - 1));
            quantized_num = quantized_num | old_sign;
        }
    } // underflow or round to smallest nonzero normal value
    else if (quantized_exponent_store < min_exponent_store)
    {
        uint32_t offset = (quantized_exponent_store == (min_exponent_store - 1)) && ((old_num << 9 >> 9) > (1 << 22));
        quantized_num = offset * (min_exponent_store << 23);
        quantized_num |= old_sign;
    }
    return quantized_num;
}

__device__ __forceinline__ uint32_t
clip_max_exponent(int man_bits, uint32_t max_exponent,
                  uint32_t quantized_num) {
  uint32_t quantized_exponent =
      quantized_num << 1 >> 24 << 23; // 1 sign bit, 23 mantissa bits
  if (quantized_exponent > max_exponent) {
    uint32_t max_man =
        (uint32_t)-1 << 9 >> 9 >>
        (23 - man_bits) << (23 - man_bits); // 1 sign bit, 8 exponent bits
    uint32_t max_num = max_exponent | max_man;
    uint32_t old_sign = quantized_num >> 31 << 31;
    quantized_num = old_sign | max_num;
  }
  return quantized_num;
}
