
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdint>

#define FLOAT_TO_BITS(x) (*reinterpret_cast<uint32_t *>(x))
#define BITS_TO_FLOAT(x) (*reinterpret_cast<float *>(x))

__device__ __forceinline__ uint32_t extract_exponent(float *a) {
  uint32_t temp = *(reinterpret_cast<uint32_t *>(a));
  temp = (temp << 1 >> 24); // single precision, 1 sign bit, 23 mantissa bits
  return temp - 127 + 1;    // exponent offset and virtual bit
}

__device__ __forceinline__ uint32_t
round_bitwise_stochastic(uint32_t target, uint32_t rand_prob, int man_bits) { // passing number of random bits as second parameter (all the bits after the least significant bit which is based on prng); target is the original number
  uint32_t mask = (1 << (23 - man_bits)) - 1; 
  uint32_t add_r = target + (rand_prob & mask); // adding random bits to target (which is not masked)
  uint32_t quantized = add_r & ~mask; // masking out bits on the right hand side of the significant bits (truncating)
  return quantized;
}

// examples with man_bits = 2   
// 0 00000000 0x|xxxxxxxxxxxxxxxxxxxxx -> x = 0100000000000000000000 -> remove the (8+man_bits) first bit
// 0 00000000 00|100000000000000000000 -> m = x0100000000000000000000 ->  offset equal to 1 if the mantissa = x0100...
// 0 00000000 00|111111111111111111111 or 01|111111111111111111111
// 0 00000000 00|100000000000000000000          
// every case we add 00|10... and "undo" it with the mask if special case -> the offset = 1              
// set to zero the end that we don't want unless we are in the special case with x0|100...
// in this case we round down because we are in the middle with the nearest even being the down

__device__ __forceinline__ uint32_t
round_bitwise_nearest(uint32_t target, int man_bits) {
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  // tie breaking rule offset
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  // apply the mask
  // this is the analogue of how you would do round 
  // to nearest integer using the floor function: 
  // round(x) = floor(x + 0.5)
  return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}


__device__ __forceinline__ uint32_t round_bitwise_up(uint32_t target, int man_bits) {
  uint32_t mask = (1 << (23 - man_bits)) - 1;
  uint32_t nexact = ((target << 1 >> 1) & mask) > 0u ? 1u : 0u;
  uint32_t sign = target >> 31;
  uint32_t rand_prob = (nexact & ~sign) << (23 - man_bits);
  uint32_t add_r = target + rand_prob;
  uint32_t quantized = add_r & ~mask;
  return quantized;
}

__device__ __forceinline__ uint32_t round_bitwise_down(uint32_t target, int man_bits) {
  uint32_t mask = (1 << (23 - man_bits)) - 1;
  uint32_t nexact = ((target << 1 >> 1) & mask) > 0u ? 1u : 0u;
  uint32_t sign = target >> 31;
  uint32_t rand_prob = (nexact & sign) << (23 - man_bits);
  uint32_t add_r = target + rand_prob;
  uint32_t quantized = add_r & ~mask;
  return quantized;
}

__device__ __forceinline__ uint32_t
clip_exponent(int exp_bits, int man_bits, uint32_t old_num,
              uint32_t quantized_num, bool saturate) {
  if (quantized_num == 0)
    return quantized_num;

  int quantized_exponent_store = quantized_num << 1 >> 24;
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127;
  int min_exponent_store = -((1 << (exp_bits - 1)) - 2) + 127;

  uint32_t old_sign = old_num >> 31 << 31;
  // saturate or overflow
  if (quantized_exponent_store > max_exponent_store) {
    if (saturate) {
      uint32_t max_man =
          (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits);
      uint32_t max_num = ((uint32_t)max_exponent_store << 23) | max_man;
      quantized_num = old_sign | max_num;
    } else {
      quantized_num =
          ((((uint32_t)1 << 31) - 1) ^ (((uint32_t)1 << 23) - 1));
      quantized_num = quantized_num | old_sign;
    }
  } else if (quantized_exponent_store < min_exponent_store) {
    uint32_t min_num = ((uint32_t)min_exponent_store << 23);
    uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23);
    uint32_t unsigned_quantized_num = quantized_num << 1 >> 1;
    if (unsigned_quantized_num > middle_num) {
      uint32_t old_sign = old_num >> 31 << 31;
      quantized_num = old_sign | min_num;
    } else {
      quantized_num = 0;
    }
  }
  return quantized_num;
}

__device__ __forceinline__ uint32_t
clip_max_exponent(int man_bits, uint32_t max_exponent,  uint32_t quantized_num) {
  uint32_t quantized_exponent = quantized_num << 1 >> 24 << 23; // 1 sign bit, 23 mantissa bits
  if (quantized_exponent > max_exponent) {
    uint32_t max_man = (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits); // 1 sign bit, 8 exponent bits
    uint32_t max_num = max_exponent | max_man;
    uint32_t old_sign = quantized_num >> 31 << 31;
    quantized_num = old_sign | max_num;
  }
  return quantized_num;
}

__device__ __forceinline__ uint32_t
p3109_clip_exponent(int exp_bits, int man_bits, uint32_t old_num, uint32_t quantized_num, bool saturate, bool subnormal) {  // currently sets max to FE; talks of possibly setting max to FD were mentioned for unsigned P = 1
  if (quantized_num == 0) 
    return quantized_num;
  
  int spec_exp = (man_bits == 0) ? 1 : 0; // special exponent case at P = 1
  int quantized_exponent_store = (quantized_num >> 23) & 0xFF;
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127;
  int min_exponent_store = -((1 << (exp_bits - 1)) - 1) + 127 + spec_exp; // adding special exponent (1 for P = 1 and 0 for all other precision)

  // the following values were calculated prior to entering the conditionals (which defers from clip_exponent) 
  uint32_t max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  uint32_t man_val = quantized_num & 0x7FFFFF;  // max mantissa val is all 1s
  uint32_t old_sign = old_num & 0x80000000;

  if (quantized_exponent_store > max_exponent_store) {
    if (saturate) { 
      quantized_num = old_sign | ((uint32_t)max_exponent_store << 23) | max_man;
    } else {
      quantized_num = old_sign | 0x7F800000; // INF
    }
  } else if (quantized_exponent_store < min_exponent_store) {
    if (subnormal) {
        int subnormal_shift = min_exponent_store - quantized_exponent_store;
        int min_subnormals_exp = min_exponent_store - man_bits;
        uint32_t min_num = ((uint32_t)min_subnormals_exp << 23);
        uint32_t middle_num = ((uint32_t)(min_subnormals_exp - 1) << 23);
      if (subnormal_shift <= man_bits) {
        // quantized_num stays the same in this case
      } else if ((old_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    } else {
      uint32_t min_num = (uint32_t)min_exponent_store << 23;
      uint32_t middle_num = ((uint32_t)(min_exponent_store - 1) << 23);
      if ((quantized_num & 0x7FFFFFFF) > middle_num) {
        quantized_num = old_sign | min_num;
      } else {
        quantized_num = 0;
      }
    }
  }

  return quantized_num;
}
