
#include <hip/hip_runtime.h>
#include <cmath>
#include <cstdint>

#define FLOAT_TO_BITS(x) (*reinterpret_cast<uint32_t *>(x))
#define BITS_TO_FLOAT(x) (*reinterpret_cast<float *>(x))

/* __device__ uint32_t rn_prob[24] = {
    4194304, 2097152, 1048576, 524288, 262144, 131072, 65536, 32768,
    16384,   8192,    4096,    2048,   1024,   512,    256,   128,
    64,      32,      16,      8,      4,      2,      1,     0}; */

__device__ __forceinline__ uint32_t extract_exponent(float *a) {
  uint32_t temp = *(reinterpret_cast<uint32_t *>(a));
  temp = (temp << 1 >> 24); // single precision, 1 sign bit, 23 mantissa bits
  return temp - 127 + 1;    // exponent offset and virtual bit
}

__device__ __forceinline__ uint32_t
round_bitwise_stochastic(uint32_t target, uint32_t rand_prob,
                         int man_bits) {
  uint32_t mask = (1 << (23 - man_bits)) - 1;
  uint32_t add_r = target + (rand_prob & mask);
  uint32_t quantized = add_r & ~mask;
  return quantized;
}

// examples with man_bits = 2   
// 0 00000000 0x|xxxxxxxxxxxxxxxxxxxxx -> x = 0100000000000000000000 -> remove the (8+man_bits) first bit
// 0 00000000 00|100000000000000000000 -> m = x0100000000000000000000 ->  offset equal to 1 if the mantissa = x0100...
// 0 00000000 00|111111111111111111111 or 01|111111111111111111111
// 0 00000000 00|100000000000000000000          
// every case we add 00|10... and "undo" it with the mask if special case -> the offset = 1              
// set to zero the end that we don't want unless we are in the special case with x0|100...
// in this case we round down because we are in the middle with the nearest even being the down

__device__ __forceinline__ uint32_t
round_bitwise_nearest(uint32_t target, int old_man_bits, int subnormal_shift) {
  int man_bits = old_man_bits - subnormal_shift;
  uint32_t down = target << (8 + man_bits) >> (8 + man_bits);
  uint32_t machine_eps = 1 << (22 - man_bits);
  // tie breaking rule offset
  int offset = (down == machine_eps);
  uint32_t add_r = target + machine_eps;
  // apply the mask
  // this is the analogue of how you would do round 
  // to nearest integer using the floor function: 
  // round(x) = floor(x + 0.5)
  return add_r & ~((1 << (23 - man_bits + offset)) - 1);
}


__device__ __forceinline__ uint32_t round_bitwise_up(uint32_t target, int man_bits) {
  uint32_t mask = (1 << (23 - man_bits)) - 1;
  uint32_t nexact = ((target << 1 >> 1) & mask) > 0u ? 1u : 0u;
  uint32_t sign = target >> 31;
  uint32_t rand_prob = (nexact & ~sign) << (23 - man_bits);
  uint32_t add_r = target + rand_prob;
  uint32_t quantized = add_r & ~mask;
  return quantized;
}

__device__ __forceinline__ uint32_t round_bitwise_down(uint32_t target, int man_bits) {
  uint32_t mask = (1 << (23 - man_bits)) - 1;
  uint32_t nexact = ((target << 1 >> 1) & mask) > 0u ? 1u : 0u;
  uint32_t sign = target >> 31;
  uint32_t rand_prob = (nexact & sign) << (23 - man_bits);
  uint32_t add_r = target + rand_prob;
  uint32_t quantized = add_r & ~mask;
  return quantized;
}

__device__ __forceinline__ uint32_t
clip_exponent(int exp_bits, int man_bits, uint32_t old_num, uint32_t quantized_num, bool saturate) {

  if (quantized_num == 0){
    return quantized_num;
  }

  int quantized_exponent_store = quantized_num << 1 >> 24;
  int max_exponent_store = (1 << (exp_bits - 1)) - 1 + 127;
  int min_exponent_store = -((1 << (exp_bits - 1)) - 1) + 127;
  
  uint32_t max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
  uint32_t man_val = quantized_num & 0x7FFFFF;

  uint32_t old_sign = old_num >> 31 << 31;
  // saturate or overflow
  if ((quantized_exponent_store > max_exponent_store) || ((quantized_exponent_store == max_exponent_store) && (man_val > max_man))) {
    if (saturate) {
      uint32_t max_man = (((1u << man_bits) - 1u) & ~1u) << (23 - man_bits);
      //uint32_t max_man = (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits);
      uint32_t max_num = ((uint32_t)max_exponent_store << 23) | max_man;
      quantized_num = old_sign | max_num;
    } else {
      quantized_num = ((((uint32_t)1 << 31) - 1) ^ (((uint32_t)1 << 23) - 1));
      quantized_num = quantized_num | old_sign;
    }
  } else if (quantized_exponent_store < min_exponent_store) {
    int subnormal_shift = min_exponent_store - quantized_exponent_store;
    int min_subnormals_exp = min_exponent_store - man_bits;
    uint32_t min_num = ((uint32_t)min_subnormals_exp << 23);
    uint32_t middle_num = ((uint32_t)(min_subnormals_exp - 1) << 23);
    uint32_t unsigned_quantized_num = old_num << 1 >> 1;
    //print_uint(unsigned_quantized_num);
    //print_uint(middle_num);
    if (subnormal_shift <= man_bits)  {
      quantized_num = quantized_num;
      //cout << "1" << endl;
    } 
    else if (unsigned_quantized_num > middle_num) {
      uint32_t old_sign = old_num >> 31 << 31;
      quantized_num = old_sign | min_num;
      //cout << "2" << endl;
    }else {
      quantized_num = 0;
      //cout << "3 " << endl;
    }
  }
  return quantized_num;
}

__device__ __forceinline__ uint32_t
clip_max_exponent(int man_bits, uint32_t max_exponent,  uint32_t quantized_num) {
  uint32_t quantized_exponent = quantized_num << 1 >> 24 << 23; // 1 sign bit, 23 mantissa bits
  if (quantized_exponent > max_exponent) {
    uint32_t max_man = (uint32_t)-1 << 9 >> 9 >> (23 - man_bits) << (23 - man_bits); // 1 sign bit, 8 exponent bits
    uint32_t max_num = max_exponent | max_man;
    uint32_t old_sign = quantized_num >> 31 << 31;
    quantized_num = old_sign | max_num;
  }
  return quantized_num;
}
